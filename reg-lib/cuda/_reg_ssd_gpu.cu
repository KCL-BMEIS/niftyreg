#include "hip/hip_runtime.h"
/*
 * @file _reg_ssd_gpu.cu
 * @author Marc Modat
 * @date 14/11/2012
 *
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 * See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _REG_SSD_GPU_CU
#define _REG_SSD_GPU_CU

#include "_reg_ssd_gpu.h"
#include "_reg_ssd_kernels.cu"

/* *************************************************************** */
/* *************************************************************** */
reg_ssd_gpu::reg_ssd_gpu()
	: reg_ssd::reg_ssd()
{
#ifndef NDEBUG
		printf("[NiftyReg DEBUG] reg_ssd_gpu constructor called\n");
#endif
}
/* *************************************************************** */
/* *************************************************************** */
void reg_ssd_gpu::InitialiseMeasure(nifti_image *refImgPtr,
									nifti_image *floImgPtr,
									int *maskRefPtr,
									int activeVoxNum,
									nifti_image *warFloImgPtr,
									nifti_image *warFloGraPtr,
									nifti_image *forVoxBasedGraPtr,
									hipArray **refDevicePtr,
									hipArray **floDevicePtr,
									int **refMskDevicePtr,
									float **warFloDevicePtr,
									float4 **warFloGradDevicePtr,
									float4 **forVoxBasedGraDevicePtr)
{
	reg_ssd::InitialiseMeasure(refImgPtr,
							   floImgPtr,
							   maskRefPtr,
							   warFloImgPtr,
							   warFloGraPtr,
							   forVoxBasedGraPtr);
	// Check if a symmetric measure is required
	if(this->isSymmetric){
		fprintf(stderr,"[NiftyReg ERROR] reg_nmi_gpu::InitialiseMeasure\n");
		fprintf(stderr,"[NiftyReg ERROR] Symmetric scheme is not yet supported on the GPU\n");
		reg_exit(1);
	}
	// Check that the input image are of type float
	if(this->referenceImagePointer->datatype!=NIFTI_TYPE_FLOAT32 ||
	   this->warpedFloatingImagePointer->datatype!=NIFTI_TYPE_FLOAT32){
		fprintf(stderr,"[NiftyReg ERROR] reg_nmi_gpu::InitialiseMeasure\n");
		fprintf(stderr,"[NiftyReg ERROR] The input images are expected to be float\n");
		reg_exit(1);
	}
	// Check that the input images have only one time point
	if(this->referenceImagePointer->nt>1 || this->floatingImagePointer->nt>1){
		fprintf(stderr,"[NiftyReg ERROR] reg_nmi_gpu::InitialiseMeasure\n");
		fprintf(stderr,"[NiftyReg ERROR] Both input images should have only one time point\n");
		reg_exit(1);
	}
	// Bind the required pointers
	this->referenceDevicePointer = *refDevicePtr;
	this->floatingDevicePointer = *floDevicePtr;
	this->referenceMaskDevicePointer = *refMskDevicePtr;
	this->activeVoxeNumber=activeVoxNum;
	this->warpedFloatingDevicePointer = *warFloDevicePtr;
	this->warpedFloatingGradientDevicePointer = *warFloGradDevicePtr;
	this->forwardVoxelBasedGradientDevicePointer = *forVoxBasedGraDevicePtr;
#ifndef NDEBUG
		printf("[NiftyReg DEBUG] reg_ssd_gpu::InitialiseMeasure()\n");
#endif
}
/* *************************************************************** */
float reg_getSSDValue_gpu(nifti_image *referenceImage,
						  hipArray **reference_d,
						  float **warped_d,
						  int **mask_d,
						  int activeVoxelNumber
						  )
{
    // Get the BlockSize - The values have been set in _reg_common_gpu.h - cudaCommon_setCUDACard
    NiftyReg_CudaBlock100 *NR_BLOCK = NiftyReg_CudaBlock::getInstance(0);

	// Copy the constant memory variables
	int3 referenceDim = make_int3(referenceImage->nx, referenceImage->ny, referenceImage->nz);
	int voxelNumber = referenceImage->nx * referenceImage->ny * referenceImage->nz;
	NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ReferenceImageDim),&referenceDim,sizeof(int3)))
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ActiveVoxelNumber),&activeVoxelNumber,sizeof(int)))
	// Bind the required textures
	referenceTexture.normalized = true;
	referenceTexture.filterMode = hipFilterModeLinear;
	referenceTexture.addressMode[0] = hipAddressModeWrap;
	referenceTexture.addressMode[1] = hipAddressModeWrap;
	referenceTexture.addressMode[2] = hipAddressModeWrap;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	NR_CUDA_SAFE_CALL(hipBindTextureToArray(referenceTexture, *reference_d, channelDesc))
	NR_CUDA_SAFE_CALL(hipBindTexture(0, warpedTexture, *warped_d, voxelNumber*sizeof(float)))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, maskTexture, *mask_d, activeVoxelNumber*sizeof(int)))
	// Create an array on the device to store the absolute difference values
	float *absoluteValues_d;
    NR_CUDA_SAFE_CALL(hipMalloc(&absoluteValues_d, activeVoxelNumber*sizeof(float)))
	// Compute the absolute values
	const unsigned int Grid_reg_getSquaredDifference =
            (unsigned int)ceil(sqrtf((float)activeVoxelNumber/(float)NR_BLOCK->Block_reg_getSquaredDifference));
    dim3 B1(NR_BLOCK->Block_reg_getSquaredDifference,1,1);
	dim3 G1(Grid_reg_getSquaredDifference,Grid_reg_getSquaredDifference,1);
	if(referenceDim.z>1)
		reg_getSquaredDifference3D_kernel <<< G1, B1 >>> (absoluteValues_d);
	else reg_getSquaredDifference2D_kernel <<< G1, B1 >>> (absoluteValues_d);
	NR_CUDA_CHECK_KERNEL(G1,B1)
	// Unbind the textures
	NR_CUDA_SAFE_CALL(hipUnbindTexture(referenceTexture))
	NR_CUDA_SAFE_CALL(hipUnbindTexture(warpedTexture))
	NR_CUDA_SAFE_CALL(hipUnbindTexture(maskTexture))
	// Perform a reduction on the absolute values
    float ssd = (float)((double)reg_sumReduction_gpu(absoluteValues_d,activeVoxelNumber) / (double)activeVoxelNumber);
	// Free the absolute value array
	NR_CUDA_SAFE_CALL(hipFree(absoluteValues_d))

	return ssd;
}
/* *************************************************************** */
/* *************************************************************** */
double reg_ssd_gpu::GetSimilarityMeasureValue()
{
	double SSDValue = reg_getSSDValue_gpu(this->referenceImagePointer,
										  &this->referenceDevicePointer,
										  &this->warpedFloatingDevicePointer,
										  &this->referenceMaskDevicePointer,
										  this->activeVoxeNumber
										  );
    return -SSDValue;
}
/* *************************************************************** */
/* *************************************************************** */
void reg_getVoxelBasedSSDGradient_gpu(nifti_image *referenceImage,
									  hipArray **reference_d,
									  float **warped_d,
									  float4 **spaGradient_d,
									  float4 **ssdGradient_d,
									  float maxSD,
									  int **mask_d,
									  int activeVoxelNumber
									  )
{
    // Get the BlockSize - The values have been set in _reg_common_gpu.h - cudaCommon_setCUDACard
    NiftyReg_CudaBlock100 *NR_BLOCK = NiftyReg_CudaBlock::getInstance(0);

	// Copy the constant memory variables
	int3 referenceDim = make_int3(referenceImage->nx, referenceImage->ny, referenceImage->nz);
	int voxelNumber = referenceImage->nx*referenceImage->ny*referenceImage->nz;
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ReferenceImageDim),&referenceDim,sizeof(int3)))
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ActiveVoxelNumber),&activeVoxelNumber,sizeof(int)))
	NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NormalisationNumber),&maxSD,sizeof(float)))
	// Bind the required textures
	referenceTexture.normalized = true;
	referenceTexture.filterMode = hipFilterModeLinear;
	referenceTexture.addressMode[0] = hipAddressModeWrap;
	referenceTexture.addressMode[1] = hipAddressModeWrap;
	referenceTexture.addressMode[2] = hipAddressModeWrap;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	NR_CUDA_SAFE_CALL(hipBindTextureToArray(referenceTexture, *reference_d, channelDesc))
	NR_CUDA_SAFE_CALL(hipBindTexture(0, warpedTexture, *warped_d, voxelNumber*sizeof(float)))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, maskTexture, *mask_d, activeVoxelNumber*sizeof(int)))
	NR_CUDA_SAFE_CALL(hipBindTexture(0, spaGradientTexture, *spaGradient_d, voxelNumber*sizeof(float4)))
	// Set the gradient image to zero
	NR_CUDA_SAFE_CALL(hipMemset(*ssdGradient_d,0,voxelNumber*sizeof(float4)))
	const unsigned int Grid_reg_getSSDGradient =
            (unsigned int)ceil(sqrtf((float)activeVoxelNumber/(float)NR_BLOCK->Block_reg_getSSDGradient));
    dim3 B1(NR_BLOCK->Block_reg_getSSDGradient,1,1);
	dim3 G1(Grid_reg_getSSDGradient,Grid_reg_getSSDGradient,1);
	if(referenceDim.z>1)
		reg_getSSDGradient3D_kernel <<< G1, B1 >>> (*ssdGradient_d);
	else reg_getSSDGradient2D_kernel <<< G1, B1 >>> (*ssdGradient_d);
	NR_CUDA_CHECK_KERNEL(G1,B1)
	// Unbind the textures
	NR_CUDA_SAFE_CALL(hipUnbindTexture(referenceTexture))
	NR_CUDA_SAFE_CALL(hipUnbindTexture(warpedTexture))
	NR_CUDA_SAFE_CALL(hipUnbindTexture(maskTexture))
	NR_CUDA_SAFE_CALL(hipUnbindTexture(spaGradientTexture))
}
/* *************************************************************** */
/* *************************************************************** */
void reg_ssd_gpu::GetVoxelBasedSimilarityMeasureGradient()
{
	reg_getVoxelBasedSSDGradient_gpu(this->referenceImagePointer,
									 &this->referenceDevicePointer,
									 &this->warpedFloatingDevicePointer,
									 &this->warpedFloatingGradientDevicePointer,
									 &this->forwardVoxelBasedGradientDevicePointer,
                                     1.0f,
									 &this->referenceMaskDevicePointer,
									 this->activeVoxeNumber
									 );
	return;
}
/* *************************************************************** */
/* *************************************************************** */
#endif
