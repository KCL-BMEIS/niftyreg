#include "hip/hip_runtime.h"
/*
 * @file _reg_ssd_gpu.cu
 * @author Marc Modat
 * @date 14/11/2012
 *
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 * See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#include "_reg_ssd_gpu.h"
#include "_reg_ssd_kernels.cu"

/* *************************************************************** */
reg_ssd_gpu::reg_ssd_gpu(): reg_ssd::reg_ssd() {
#ifndef NDEBUG
    printf("[NiftyReg DEBUG] reg_ssd_gpu constructor called\n");
#endif
}
/* *************************************************************** */
void reg_ssd_gpu::InitialiseMeasure(nifti_image *refImgPtr,
                                    nifti_image *floImgPtr,
                                    int *maskRefPtr,
                                    size_t activeVoxNum,
                                    nifti_image *warFloImgPtr,
                                    nifti_image *warFloGraPtr,
                                    nifti_image *forVoxBasedGraPtr,
                                    nifti_image *localWeightSimPtr,
                                    hipArray *refDevicePtr,
                                    hipArray *floDevicePtr,
                                    int *refMskDevicePtr,
                                    float *warFloDevicePtr,
                                    float4 *warFloGradDevicePtr,
                                    float4 *forVoxBasedGraDevicePtr) {
    reg_ssd::InitialiseMeasure(refImgPtr,
                               floImgPtr,
                               maskRefPtr,
                               warFloImgPtr,
                               warFloGraPtr,
                               forVoxBasedGraPtr,
                               localWeightSimPtr);
    // Check if a symmetric measure is required
    if (this->isSymmetric) {
        fprintf(stderr, "[NiftyReg ERROR] reg_nmi_gpu::InitialiseMeasure\n");
        fprintf(stderr, "[NiftyReg ERROR] Symmetric scheme is not yet supported on the GPU\n");
        reg_exit();
    }
    // Check that the input image are of type float
    if (this->referenceImagePointer->datatype != NIFTI_TYPE_FLOAT32 ||
        this->warpedFloatingImagePointer->datatype != NIFTI_TYPE_FLOAT32) {
        fprintf(stderr, "[NiftyReg ERROR] reg_nmi_gpu::InitialiseMeasure\n");
        fprintf(stderr, "[NiftyReg ERROR] The input images are expected to be float\n");
        reg_exit();
    }
    // Check that the input images have only one time point
    if (this->referenceImagePointer->nt > 1 || this->floatingImagePointer->nt > 1) {
        fprintf(stderr, "[NiftyReg ERROR] reg_nmi_gpu::InitialiseMeasure\n");
        fprintf(stderr, "[NiftyReg ERROR] Both input images should have only one time point\n");
        reg_exit();
    }
    // Bind the required pointers
    this->referenceDevicePointer = refDevicePtr;
    this->floatingDevicePointer = floDevicePtr;
    this->referenceMaskDevicePointer = refMskDevicePtr;
    this->activeVoxelNumber = activeVoxNum;
    this->warpedFloatingDevicePointer = warFloDevicePtr;
    this->warpedFloatingGradientDevicePointer = warFloGradDevicePtr;
    this->forwardVoxelBasedGradientDevicePointer = forVoxBasedGraDevicePtr;
#ifndef NDEBUG
    printf("[NiftyReg DEBUG] reg_ssd_gpu::InitialiseMeasure()\n");
#endif
}
/* *************************************************************** */
double reg_getSSDValue_gpu(const nifti_image *referenceImage,
                           const hipArray *referenceImageCuda,
                           const float *warpedCuda,
                           const int *maskCuda,
                           const size_t& activeVoxelNumber) {
    // Copy the constant memory variables
    const int3 referenceImageDim = make_int3(referenceImage->nx, referenceImage->ny, referenceImage->nz);
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(referenceImage, 3);

    auto referenceTexture = cudaCommon_createTextureObject(referenceImageCuda, hipResourceTypeArray, 0,
                                                           hipChannelFormatKindNone, 1, hipFilterModePoint, true);
    auto warpedTexture = cudaCommon_createTextureObject(warpedCuda, hipResourceTypeLinear, voxelNumber * sizeof(float),
                                                        hipChannelFormatKindFloat, 1);
    auto maskTexture = cudaCommon_createTextureObject(maskCuda, hipResourceTypeLinear, activeVoxelNumber * sizeof(int),
                                                      hipChannelFormatKindSigned, 1);

    // Create an array on the device to store the absolute difference values
    float *absoluteValuesCuda;
    NR_CUDA_SAFE_CALL(hipMalloc(&absoluteValuesCuda, activeVoxelNumber * sizeof(float)));

    // Compute the absolute values
    const unsigned blocks = NiftyReg::CudaContext::GetBlockSize()->reg_getSquaredDifference;
    const unsigned grids = (unsigned)ceil(sqrtf((float)activeVoxelNumber / (float)blocks));
    const dim3 gridDims(grids, grids, 1);
    const dim3 blockDims(blocks, 1, 1);
    if (referenceImageDim.z > 1)
        reg_getSquaredDifference3D_kernel<<<gridDims, blockDims>>>(absoluteValuesCuda, *referenceTexture, *warpedTexture, *maskTexture,
                                                                   referenceImageDim, (unsigned)activeVoxelNumber);
    else reg_getSquaredDifference2D_kernel<<<gridDims, blockDims>>>(absoluteValuesCuda, *referenceTexture, *warpedTexture, *maskTexture,
                                                                    referenceImageDim, (unsigned)activeVoxelNumber);
    NR_CUDA_CHECK_KERNEL(gridDims, blockDims);

    // Perform a reduction on the absolute values
    const double ssd = (double)reg_sumReduction_gpu(absoluteValuesCuda, activeVoxelNumber) / (double)activeVoxelNumber;

    // Free the absolute value array
    NR_CUDA_SAFE_CALL(hipFree(absoluteValuesCuda));

    return ssd;
}
/* *************************************************************** */
double reg_ssd_gpu::GetSimilarityMeasureValue() {
    const double SSDValue = reg_getSSDValue_gpu(this->referenceImagePointer,
                                                this->referenceDevicePointer,
                                                this->warpedFloatingDevicePointer,
                                                this->referenceMaskDevicePointer,
                                                this->activeVoxelNumber);
    return -SSDValue;
}
/* *************************************************************** */
void reg_getVoxelBasedSSDGradient_gpu(const nifti_image *referenceImage,
                                      const hipArray *referenceImageCuda,
                                      const float *warpedCuda,
                                      const float4 *spaGradientCuda,
                                      float4 *ssdGradientCuda,
                                      const float& maxSD,
                                      const int *maskCuda,
                                      const size_t& activeVoxelNumber) {
    // Copy the constant memory variables
    const int3 referenceImageDim = make_int3(referenceImage->nx, referenceImage->ny, referenceImage->nz);
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(referenceImage, 3);

    auto referenceTexture = cudaCommon_createTextureObject(referenceImageCuda, hipResourceTypeArray, 0,
                                                           hipChannelFormatKindNone, 1, hipFilterModePoint, true);
    auto warpedTexture = cudaCommon_createTextureObject(warpedCuda, hipResourceTypeLinear, voxelNumber * sizeof(float),
                                                        hipChannelFormatKindFloat, 1);
    auto maskTexture = cudaCommon_createTextureObject(maskCuda, hipResourceTypeLinear, activeVoxelNumber * sizeof(int),
                                                      hipChannelFormatKindSigned, 1);
    auto spaGradientTexture = cudaCommon_createTextureObject(spaGradientCuda, hipResourceTypeLinear, voxelNumber * sizeof(float4),
                                                             hipChannelFormatKindFloat, 4);

    // Set the gradient image to zero
    NR_CUDA_SAFE_CALL(hipMemset(ssdGradientCuda, 0, voxelNumber * sizeof(float4)));

    const unsigned blocks = NiftyReg::CudaContext::GetBlockSize()->reg_getSSDGradient;
    const unsigned grids = (unsigned)ceil(sqrtf((float)activeVoxelNumber / (float)blocks));
    const dim3 gridDims(grids, grids, 1);
    const dim3 blockDims(blocks, 1, 1);
    if (referenceImageDim.z > 1)
        reg_getSSDGradient3D_kernel<<<gridDims, blockDims>>>(ssdGradientCuda, *referenceTexture, *warpedTexture, *maskTexture,
                                                             *spaGradientTexture, referenceImageDim, maxSD, (unsigned)activeVoxelNumber);
    else reg_getSSDGradient2D_kernel<<<gridDims, blockDims>>>(ssdGradientCuda, *referenceTexture, *warpedTexture, *maskTexture,
                                                              *spaGradientTexture, referenceImageDim, maxSD, (unsigned)activeVoxelNumber);
    NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
}
/* *************************************************************** */
void reg_ssd_gpu::GetVoxelBasedSimilarityMeasureGradient(int current_timepoint) {
    reg_getVoxelBasedSSDGradient_gpu(this->referenceImagePointer,
                                     this->referenceDevicePointer,
                                     this->warpedFloatingDevicePointer,
                                     this->warpedFloatingGradientDevicePointer,
                                     this->forwardVoxelBasedGradientDevicePointer,
                                     1.f,
                                     this->referenceMaskDevicePointer,
                                     this->activeVoxelNumber);
}
/* *************************************************************** */
