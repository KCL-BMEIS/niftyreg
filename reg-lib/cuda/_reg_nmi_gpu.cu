#include "hip/hip_runtime.h"
/**
 * @file _reg_nmi_gpu.cu
 *
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#include "_reg_nmi_gpu.h"
#include "_reg_nmi_kernels.cu"
#include <thrust/device_vector.h>

/* *************************************************************** */
reg_nmi_gpu::reg_nmi_gpu(): reg_nmi::reg_nmi() {
#ifndef NDEBUG
    reg_print_msg_debug("reg_nmi_gpu constructor called");
#endif
}
/* *************************************************************** */
reg_nmi_gpu::~reg_nmi_gpu() {
#ifndef NDEBUG
    reg_print_msg_debug("reg_nmi_gpu destructor called");
#endif
}
/* *************************************************************** */
void reg_nmi_gpu::InitialiseMeasure(nifti_image *refImg, hipArray *refImgCuda,
                                    nifti_image *floImg, hipArray *floImgCuda,
                                    int *refMask, int *refMaskCuda,
                                    size_t activeVoxNum,
                                    nifti_image *warpedImg, float *warpedImgCuda,
                                    nifti_image *warpedGrad, float4 *warpedGradCuda,
                                    nifti_image *voxelBasedGrad, float4 *voxelBasedGradCuda,
                                    nifti_image *localWeightSim,
                                    int *floMask, int *floMaskCuda,
                                    nifti_image *warpedImgBw, float *warpedImgBwCuda,
                                    nifti_image *warpedGradBw, float4 *warpedGradBwCuda,
                                    nifti_image *voxelBasedGradBw, float4 *voxelBasedGradBwCuda) {
    this->DeallocateHistogram();
    reg_nmi::InitialiseMeasure(refImg, floImg, refMask, warpedImg, warpedGrad, voxelBasedGrad,
                               localWeightSim, floMask, warpedImgBw, warpedGradBw, voxelBasedGradBw);
    reg_measure_gpu::InitialiseMeasure(refImg, refImgCuda, floImg, floImgCuda, refMask, refMaskCuda, activeVoxNum, warpedImg, warpedImgCuda,
                                       warpedGrad, warpedGradCuda, voxelBasedGrad, voxelBasedGradCuda, localWeightSim, floMask, floMaskCuda,
                                       warpedImgBw, warpedImgBwCuda, warpedGradBw, warpedGradBwCuda, voxelBasedGradBw, voxelBasedGradBwCuda);
    // Check if the input images have multiple timepoints
    if (this->referenceTimePoint > 1 || this->floatingImage->nt > 1) {
        reg_print_fct_error("reg_nmi_gpu::InitialiseMeasure");
        reg_print_msg_error("Multiple timepoints are not yet supported");
        reg_exit();
    }
    // The reference and floating images have to be updated on the device
    if (cudaCommon_transferNiftiToArrayOnDevice<float>(this->referenceImageCuda, this->referenceImage) ||
        cudaCommon_transferNiftiToArrayOnDevice<float>(this->floatingImageCuda, this->floatingImage)) {
        reg_print_fct_error("reg_nmi_gpu::InitialiseMeasure");
        reg_print_msg_error("Error when transferring the reference or floating image");
        reg_exit();
    }
#ifndef NDEBUG
    reg_print_msg_debug("reg_nmi_gpu::InitialiseMeasure called");
#endif
}
/* *************************************************************** */
double reg_nmi_gpu::GetSimilarityMeasureValue() {
    // The NMI computation is performed into the host for now
    // The relevant images have to be transferred from the device to the host
    cudaCommon_transferFromDeviceToNifti<float>(this->warpedImage, this->warpedImageCuda);
    reg_getNMIValue<float>(this->referenceImage,
                           this->warpedImage,
                           this->timePointWeight,
                           this->referenceBinNumber,
                           this->floatingBinNumber,
                           this->totalBinNumber,
                           this->jointHistogramLog,
                           this->jointHistogramPro,
                           this->entropyValues,
                           this->referenceMask);

    if (this->isSymmetric) {
        cudaCommon_transferFromDeviceToNifti<float>(this->warpedImageBw, this->warpedImageBwCuda);
        reg_getNMIValue<float>(this->floatingImage,
                               this->warpedImageBw,
                               this->timePointWeight,
                               this->floatingBinNumber,
                               this->referenceBinNumber,
                               this->totalBinNumber,
                               this->jointHistogramLogBw,
                               this->jointHistogramProBw,
                               this->entropyValuesBw,
                               this->floatingMask);
    }

    double nmiFw = 0, nmiBw = 0;
    for (int t = 0; t < this->referenceTimePoint; ++t) {
        if (this->timePointWeight[t] > 0) {
            nmiFw += timePointWeight[t] * (this->entropyValues[t][0] + this->entropyValues[t][1]) / this->entropyValues[t][2];
            if (this->isSymmetric)
                nmiBw += timePointWeight[t] * (this->entropyValuesBw[t][0] + this->entropyValuesBw[t][1]) / this->entropyValuesBw[t][2];
        }
    }

#ifndef NDEBUG
    reg_print_msg_debug("reg_nmi_gpu::GetSimilarityMeasureValue called");
#endif
    return nmiFw + nmiBw;
}
/* *************************************************************** */
/// Called when we only have one target and one source image
void reg_getVoxelBasedNMIGradient_gpu(const nifti_image *referenceImage,
                                      const hipArray *referenceImageCuda,
                                      const float *warpedImageCuda,
                                      const float4 *warpedGradientCuda,
                                      const float *logJointHistogramCuda,
                                      float4 *voxelBasedGradientCuda,
                                      const int *maskCuda,
                                      const size_t& activeVoxelNumber,
                                      const double *entropies,
                                      const int& refBinning,
                                      const int& floBinning) {
    auto blockSize = NiftyReg::CudaContext::GetBlockSize();
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(referenceImage, 3);
    const int3 imageSize = make_int3(referenceImage->nx, referenceImage->ny, referenceImage->nz);
    const int binNumber = refBinning * floBinning + refBinning + floBinning;
    const float normalisedJE = (float)(entropies[2] * entropies[3]);
    const float nmi = (float)((entropies[0] + entropies[1]) / entropies[2]);

    auto referenceImageTexture = cudaCommon_createTextureObject(referenceImageCuda, hipResourceTypeArray, 0,
                                                                hipChannelFormatKindNone, 1, hipFilterModePoint, true);
    auto warpedImageTexture = cudaCommon_createTextureObject(warpedImageCuda, hipResourceTypeLinear, voxelNumber * sizeof(float),
                                                             hipChannelFormatKindFloat, 1);
    auto warpedGradientTexture = cudaCommon_createTextureObject(warpedGradientCuda, hipResourceTypeLinear, voxelNumber * sizeof(float4),
                                                                hipChannelFormatKindFloat, 4);
    auto histogramTexture = cudaCommon_createTextureObject(logJointHistogramCuda, hipResourceTypeLinear, binNumber * sizeof(float),
                                                           hipChannelFormatKindFloat, 1);
    auto maskTexture = cudaCommon_createTextureObject(maskCuda, hipResourceTypeLinear, activeVoxelNumber * sizeof(int),
                                                      hipChannelFormatKindSigned, 1);
    NR_CUDA_SAFE_CALL(hipMemset(voxelBasedGradientCuda, 0, voxelNumber * sizeof(float4)));

    if (referenceImage->nz > 1) {
        const unsigned blocks = blockSize->reg_getVoxelBasedNMIGradientUsingPW3D;
        const unsigned grids = (unsigned)ceil(sqrtf((float)activeVoxelNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_getVoxelBasedNMIGradientUsingPW3D_kernel<<<gridDims, blockDims>>>(voxelBasedGradientCuda, *referenceImageTexture, *warpedImageTexture,
                                                                              *warpedGradientTexture, *histogramTexture, *maskTexture,
                                                                              imageSize, refBinning, floBinning, normalisedJE, nmi,
                                                                              (unsigned)activeVoxelNumber);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    } else {
        const unsigned blocks = blockSize->reg_getVoxelBasedNMIGradientUsingPW2D;
        const unsigned grids = (unsigned)ceil(sqrtf((float)activeVoxelNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_getVoxelBasedNMIGradientUsingPW2D_kernel<<<gridDims, blockDims>>>(voxelBasedGradientCuda, *referenceImageTexture, *warpedImageTexture,
                                                                              *warpedGradientTexture, *histogramTexture, *maskTexture,
                                                                              imageSize, refBinning, floBinning, normalisedJE, nmi,
                                                                              (unsigned)activeVoxelNumber);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }
}
/* *************************************************************** */
void reg_nmi_gpu::GetVoxelBasedSimilarityMeasureGradient(int currentTimepoint) {
    // Check if the specified time point exists and is active
    reg_measure::GetVoxelBasedSimilarityMeasureGradient(currentTimepoint);
    if (this->timePointWeight[currentTimepoint] == 0)
        return;

    // Call compute similarity measure to calculate joint histogram
    this->GetSimilarityMeasureValue();

    // The latest joint histogram is transferred onto the GPU
    thrust::device_vector<float> jointHistogramLogCuda(this->jointHistogramLog[0], this->jointHistogramLog[0] + this->totalBinNumber[0]);

    // The gradient of the NMI is computed on the GPU
    reg_getVoxelBasedNMIGradient_gpu(this->referenceImage,
                                     this->referenceImageCuda,
                                     this->warpedImageCuda,
                                     this->warpedGradientCuda,
                                     jointHistogramLogCuda.data().get(),
                                     this->voxelBasedGradientCuda,
                                     this->referenceMaskCuda,
                                     this->activeVoxelNumber,
                                     this->entropyValues[0],
                                     this->referenceBinNumber[0],
                                     this->floatingBinNumber[0]);

    if (this->isSymmetric) {
        thrust::device_vector<float> jointHistogramLogCudaBw(this->jointHistogramLogBw[0], this->jointHistogramLogBw[0] + this->totalBinNumber[0]);
        reg_getVoxelBasedNMIGradient_gpu(this->floatingImage,
                                         this->floatingImageCuda,
                                         this->warpedImageBwCuda,
                                         this->warpedGradientBwCuda,
                                         jointHistogramLogCudaBw.data().get(),
                                         this->voxelBasedGradientBwCuda,
                                         this->floatingMaskCuda,
                                         this->activeVoxelNumber,
                                         this->entropyValuesBw[0],
                                         this->floatingBinNumber[0],
                                         this->referenceBinNumber[0]);
    }
#ifndef NDEBUG
    reg_print_msg_debug("reg_nmi_gpu::GetVoxelBasedSimilarityMeasureGradient called\n");
#endif
}
/* *************************************************************** */
