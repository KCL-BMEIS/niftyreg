#include "hip/hip_runtime.h"
/**
 * @file _reg_nmi_gpu.cu
 *
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#include "_reg_nmi_gpu.h"
#include "_reg_nmi_kernels.cu"
#include <thrust/device_vector.h>

/* *************************************************************** */
reg_nmi_gpu::reg_nmi_gpu(): reg_nmi::reg_nmi() {
    NR_FUNC_CALLED();
}
/* *************************************************************** */
reg_nmi_gpu::~reg_nmi_gpu() {
    NR_FUNC_CALLED();
}
/* *************************************************************** */
void reg_nmi_gpu::InitialiseMeasure(nifti_image *refImg, hipArray *refImgCuda,
                                    nifti_image *floImg, hipArray *floImgCuda,
                                    int *refMask, int *refMaskCuda,
                                    size_t activeVoxNum,
                                    nifti_image *warpedImg, float *warpedImgCuda,
                                    nifti_image *warpedGrad, float4 *warpedGradCuda,
                                    nifti_image *voxelBasedGrad, float4 *voxelBasedGradCuda,
                                    nifti_image *localWeightSim,
                                    int *floMask, int *floMaskCuda,
                                    nifti_image *warpedImgBw, float *warpedImgBwCuda,
                                    nifti_image *warpedGradBw, float4 *warpedGradBwCuda,
                                    nifti_image *voxelBasedGradBw, float4 *voxelBasedGradBwCuda) {
    this->DeallocateHistogram();
    reg_nmi::InitialiseMeasure(refImg, floImg, refMask, warpedImg, warpedGrad, voxelBasedGrad,
                               localWeightSim, floMask, warpedImgBw, warpedGradBw, voxelBasedGradBw);
    reg_measure_gpu::InitialiseMeasure(refImg, refImgCuda, floImg, floImgCuda, refMask, refMaskCuda, activeVoxNum, warpedImg, warpedImgCuda,
                                       warpedGrad, warpedGradCuda, voxelBasedGrad, voxelBasedGradCuda, localWeightSim, floMask, floMaskCuda,
                                       warpedImgBw, warpedImgBwCuda, warpedGradBw, warpedGradBwCuda, voxelBasedGradBw, voxelBasedGradBwCuda);
    // Check if the input images have multiple timepoints
    if (this->referenceTimePoint > 1 || this->floatingImage->nt > 1)
        NR_FATAL_ERROR("Multiple timepoints are not yet supported");
    // The reference and floating images have to be updated on the device
    Cuda::TransferNiftiToDevice<float>(this->referenceImageCuda, this->referenceImage);
    Cuda::TransferNiftiToDevice<float>(this->floatingImageCuda, this->floatingImage);
    NR_FUNC_CALLED();
}
/* *************************************************************** */
double GetSimilarityMeasureValue(const nifti_image *referenceImage,
                                 nifti_image *warpedImage,
                                 const float *warpedImageCuda,
                                 const double *timePointWeight,
                                 const unsigned short *referenceBinNumber,
                                 const unsigned short *floatingBinNumber,
                                 const unsigned short *totalBinNumber,
                                 double **jointHistogramLog,
                                 double **jointHistogramPro,
                                 double **entropyValues,
                                 const int *referenceMask,
                                 const int& referenceTimePoint) {
    // The NMI computation is performed on the host for now
    Cuda::TransferFromDeviceToNifti<float>(warpedImage, warpedImageCuda);
    reg_getNMIValue<float>(referenceImage,
                           warpedImage,
                           timePointWeight,
                           referenceBinNumber,
                           floatingBinNumber,
                           totalBinNumber,
                           jointHistogramLog,
                           jointHistogramPro,
                           entropyValues,
                           referenceMask);

    double nmi = 0;
    for (int t = 0; t < referenceTimePoint; ++t) {
        if (timePointWeight[t] > 0)
            nmi += timePointWeight[t] * (entropyValues[t][0] + entropyValues[t][1]) / entropyValues[t][2];
    }
    return nmi;
}
/* *************************************************************** */
double reg_nmi_gpu::GetSimilarityMeasureValueFw() {
    return ::GetSimilarityMeasureValue(this->referenceImage,
                                       this->warpedImage,
                                       this->warpedImageCuda,
                                       this->timePointWeight,
                                       this->referenceBinNumber,
                                       this->floatingBinNumber,
                                       this->totalBinNumber,
                                       this->jointHistogramLog,
                                       this->jointHistogramPro,
                                       this->entropyValues,
                                       this->referenceMask,
                                       this->referenceTimePoint);
}
/* *************************************************************** */
double reg_nmi_gpu::GetSimilarityMeasureValueBw() {
    return ::GetSimilarityMeasureValue(this->floatingImage,
                                       this->warpedImageBw,
                                       this->warpedImageBwCuda,
                                       this->timePointWeight,
                                       this->floatingBinNumber,
                                       this->referenceBinNumber,
                                       this->totalBinNumber,
                                       this->jointHistogramLogBw,
                                       this->jointHistogramProBw,
                                       this->entropyValuesBw,
                                       this->floatingMask,
                                       this->referenceTimePoint);
}
/* *************************************************************** */
/// Called when we only have one target and one source image
void reg_getVoxelBasedNMIGradient_gpu(const nifti_image *referenceImage,
                                      const hipArray *referenceImageCuda,
                                      const float *warpedImageCuda,
                                      const float4 *warpedGradientCuda,
                                      const float *logJointHistogramCuda,
                                      float4 *voxelBasedGradientCuda,
                                      const int *maskCuda,
                                      const size_t& activeVoxelNumber,
                                      const double *entropies,
                                      const int& refBinning,
                                      const int& floBinning) {
    auto blockSize = CudaContext::GetBlockSize();
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(referenceImage, 3);
    const int3 imageSize = make_int3(referenceImage->nx, referenceImage->ny, referenceImage->nz);
    const int binNumber = refBinning * floBinning + refBinning + floBinning;
    const float normalisedJE = (float)(entropies[2] * entropies[3]);
    const float nmi = (float)((entropies[0] + entropies[1]) / entropies[2]);

    auto referenceImageTexture = Cuda::CreateTextureObject(referenceImageCuda, hipResourceTypeArray, 0,
                                                           hipChannelFormatKindNone, 1, hipFilterModePoint, true);
    auto warpedImageTexture = Cuda::CreateTextureObject(warpedImageCuda, hipResourceTypeLinear, voxelNumber * sizeof(float),
                                                        hipChannelFormatKindFloat, 1);
    auto warpedGradientTexture = Cuda::CreateTextureObject(warpedGradientCuda, hipResourceTypeLinear, voxelNumber * sizeof(float4),
                                                           hipChannelFormatKindFloat, 4);
    auto histogramTexture = Cuda::CreateTextureObject(logJointHistogramCuda, hipResourceTypeLinear, binNumber * sizeof(float),
                                                      hipChannelFormatKindFloat, 1);
    auto maskTexture = Cuda::CreateTextureObject(maskCuda, hipResourceTypeLinear, activeVoxelNumber * sizeof(int),
                                                 hipChannelFormatKindSigned, 1);
    NR_CUDA_SAFE_CALL(hipMemset(voxelBasedGradientCuda, 0, voxelNumber * sizeof(float4)));

    if (referenceImage->nz > 1) {
        const unsigned blocks = blockSize->reg_getVoxelBasedNMIGradientUsingPW3D;
        const unsigned grids = (unsigned)ceil(sqrtf((float)activeVoxelNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_getVoxelBasedNMIGradientUsingPW3D_kernel<<<gridDims, blockDims>>>(voxelBasedGradientCuda, *referenceImageTexture, *warpedImageTexture,
                                                                              *warpedGradientTexture, *histogramTexture, *maskTexture,
                                                                              imageSize, refBinning, floBinning, normalisedJE, nmi,
                                                                              (unsigned)activeVoxelNumber);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    } else {
        const unsigned blocks = blockSize->reg_getVoxelBasedNMIGradientUsingPW2D;
        const unsigned grids = (unsigned)ceil(sqrtf((float)activeVoxelNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_getVoxelBasedNMIGradientUsingPW2D_kernel<<<gridDims, blockDims>>>(voxelBasedGradientCuda, *referenceImageTexture, *warpedImageTexture,
                                                                              *warpedGradientTexture, *histogramTexture, *maskTexture,
                                                                              imageSize, refBinning, floBinning, normalisedJE, nmi,
                                                                              (unsigned)activeVoxelNumber);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }
}
/* *************************************************************** */
void reg_nmi_gpu::GetVoxelBasedSimilarityMeasureGradientFw(int currentTimepoint) {
    // Call compute similarity measure to calculate joint histogram
    this->GetSimilarityMeasureValue();

    // The latest joint histogram is transferred onto the GPU
    thrust::device_vector<float> jointHistogramLogCuda(this->jointHistogramLog[0], this->jointHistogramLog[0] + this->totalBinNumber[0]);

    // The gradient of the NMI is computed on the GPU
    reg_getVoxelBasedNMIGradient_gpu(this->referenceImage,
                                     this->referenceImageCuda,
                                     this->warpedImageCuda,
                                     this->warpedGradientCuda,
                                     jointHistogramLogCuda.data().get(),
                                     this->voxelBasedGradientCuda,
                                     this->referenceMaskCuda,
                                     this->activeVoxelNumber,
                                     this->entropyValues[0],
                                     this->referenceBinNumber[0],
                                     this->floatingBinNumber[0]);
}
/* *************************************************************** */
void reg_nmi_gpu::GetVoxelBasedSimilarityMeasureGradientBw(int currentTimepoint) {
    // The latest joint histogram is transferred onto the GPU
    thrust::device_vector<float> jointHistogramLogCudaBw(this->jointHistogramLogBw[0], this->jointHistogramLogBw[0] + this->totalBinNumber[0]);

    // The gradient of the NMI is computed on the GPU
    reg_getVoxelBasedNMIGradient_gpu(this->floatingImage,
                                     this->floatingImageCuda,
                                     this->warpedImageBwCuda,
                                     this->warpedGradientBwCuda,
                                     jointHistogramLogCudaBw.data().get(),
                                     this->voxelBasedGradientBwCuda,
                                     this->floatingMaskCuda,
                                     this->activeVoxelNumber,
                                     this->entropyValuesBw[0],
                                     this->floatingBinNumber[0],
                                     this->referenceBinNumber[0]);
}
/* *************************************************************** */
