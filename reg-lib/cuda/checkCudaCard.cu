#include <hip/hip_runtime.h>

#include <iostream>
#include <algorithm>

int main() {
    int deviceCount = 0, output = 0;
    const hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);

    if (cudaResultCode != hipSuccess) {
        std::cerr << hipGetErrorString(cudaResultCode) << " (CUDA Error Code=" << cudaResultCode << ")" << std::endl;
        return EXIT_FAILURE;
    }

    if (deviceCount == 0) {
        std::cerr << "No device detected" << std::endl;
        return EXIT_FAILURE;
    }

    // Detect device capability and pick the best
    for (int i = 0; i < deviceCount; i++) {
        hipSetDevice(i);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        output = std::max(output, deviceProp.major * 10 + deviceProp.minor);
    }

    // Output for device capability
    std::cout << output / 10 << "." << output % 10;

    return EXIT_SUCCESS;
}
