#include "hip/hip_runtime.h"
/*
 *  _reg_affineTransformation_gpu.cu
 *
 *
 *  Created by Marc Modat on 25/03/2009.
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _REG_AFFINETRANSFORMATION_GPU_CU
#define _REG_AFFINETRANSFORMATION_GPU_CU

#include "_reg_globalTransformation_gpu.h"
#include "_reg_globalTransformation_kernels.cu"

/* *************************************************************** */
/* *************************************************************** */
void reg_affine_positionField_gpu(	mat44 *affineMatrix,
					nifti_image *targetImage,
					float4 **array_d)
{
    // Get the BlockSize - The values have been set in _reg_common_gpu.h - cudaCommon_setCUDACard
    NiftyReg_CudaBlock100 *NR_BLOCK = NiftyReg_CudaBlock::getInstance(0);

    int3 imageSize = make_int3(targetImage->nx,targetImage->ny,targetImage->nz);
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ImageSize),&imageSize,sizeof(int3)));
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNumber),&(targetImage->nvox),sizeof(int)));

    // If the target sform is defined, it is used. The qform is used otherwise
    mat44 *targetMatrix;
    if(targetImage->sform_code>0)
            targetMatrix=&(targetImage->sto_xyz);
    else targetMatrix=&(targetImage->qto_xyz);

    // We here performed Affine * TargetMat * voxelIndex
    // Affine * TargetMat is constant
    mat44 transformationMatrix = reg_mat44_mul(affineMatrix, targetMatrix);

    // The transformation matrix is binded to a texture
    float4 *transformationMatrix_h;
    NR_CUDA_SAFE_CALL(hipHostMalloc(&transformationMatrix_h, 3*sizeof(float4)));
	float4 *transformationMatrix_d;
    NR_CUDA_SAFE_CALL(hipMalloc(&transformationMatrix_d, 3*sizeof(float4)));
    for(int i=0; i<3; i++){
            transformationMatrix_h[i].x=transformationMatrix.m[i][0];
            transformationMatrix_h[i].y=transformationMatrix.m[i][1];
            transformationMatrix_h[i].z=transformationMatrix.m[i][2];
            transformationMatrix_h[i].w=transformationMatrix.m[i][3];
    }
    NR_CUDA_SAFE_CALL(hipMemcpy(transformationMatrix_d, transformationMatrix_h, 3*sizeof(float4), hipMemcpyHostToDevice));
	hipBindTexture(0,txAffineTransformation,transformationMatrix_d,3*sizeof(float4));
    NR_CUDA_SAFE_CALL(hipHostFree((void *)transformationMatrix_h));
	
        const unsigned int Grid_reg_affine_deformationField = (unsigned int)ceil(sqrtf((float)targetImage->nvox/(float)NR_BLOCK->Block_reg_affine_deformationField));
    dim3 B1(NR_BLOCK->Block_reg_affine_deformationField,1,1);
        dim3 G1(Grid_reg_affine_deformationField,Grid_reg_affine_deformationField,1);

    reg_affine_deformationField_kernel <<< G1, B1 >>> (*array_d);
        NR_CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[NiftyReg CUDA DEBUG] reg_affine_deformationField_kernel kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif

    NR_CUDA_SAFE_CALL(hipUnbindTexture(txAffineTransformation));
    NR_CUDA_SAFE_CALL(hipFree(transformationMatrix_d));
}
/* *************************************************************** */
/* *************************************************************** */

#endif
