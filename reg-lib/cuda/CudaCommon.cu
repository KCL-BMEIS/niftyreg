#include "hip/hip_runtime.h"
/**
 * @file CudaCommon.cu
 * @author Marc Modat
 * @date 25/03/2009
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 * See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#include "CudaCommon.hpp"

/* *************************************************************** */
namespace NiftyReg::Cuda {
/* *************************************************************** */
template<class DataType>
void Allocate(DataType **arrayCuda, const size_t nVoxels) {
    NR_CUDA_SAFE_CALL(hipMalloc(arrayCuda, nVoxels * sizeof(DataType)));
}
template void Allocate<int>(int**, const size_t);
template void Allocate<float>(float**, const size_t);
template void Allocate<double>(double**, const size_t);
template void Allocate<float4>(float4**, const size_t);
/* *************************************************************** */
template<class DataType>
void Allocate(DataType **arrayCuda, const int *dim) {
    const size_t memSize = (size_t)std::abs(dim[1]) * (size_t)std::abs(dim[2]) * (size_t)std::abs(dim[3]) * sizeof(DataType);
    NR_CUDA_SAFE_CALL(hipMalloc(arrayCuda, memSize));
}
template void Allocate<int>(int**, const int*);
template void Allocate<float>(float**, const int*);
template void Allocate<double>(double**, const int*);
template void Allocate<float4>(float4**, const int*);
/* *************************************************************** */
template<class DataType>
void Allocate(DataType **array1Cuda, DataType **array2Cuda, const int *dim) {
    const size_t memSize = (size_t)std::abs(dim[1]) * (size_t)std::abs(dim[2]) * (size_t)std::abs(dim[3]) * sizeof(DataType);
    NR_CUDA_SAFE_CALL(hipMalloc(array1Cuda, memSize));
    NR_CUDA_SAFE_CALL(hipMalloc(array2Cuda, memSize));
}
template void Allocate<float>(float**, float**, const int*);
template void Allocate<double>(double**, double**, const int*);
template void Allocate<float4>(float4**, float4**, const int*);
/* *************************************************************** */
template<class DataType, class NiftiType>
void TransferNiftiToDevice(DataType *arrayCuda, const nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType))
        NR_FATAL_ERROR("The host and device arrays are of different types");
    NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, img->data, img->nvox * sizeof(NiftiType), hipMemcpyHostToDevice));
}
/* *************************************************************** */
template<class DataType>
void TransferNiftiToDevice(DataType *arrayCuda, const nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if (img->datatype != NIFTI_TYPE_FLOAT32)
            NR_FATAL_ERROR("The specified image is not a single precision image");
        const float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        const auto timePointCount = img->dim[4] * img->dim[5];
        unique_ptr<float4[]> array(new float4[voxelNumber]());
        for (size_t i = 0; i < voxelNumber; i++)
            array[i].x = *niftiImgValues++;
        if (timePointCount >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].y = *niftiImgValues++;
        }
        if (timePointCount >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].z = *niftiImgValues++;
        }
        if (timePointCount >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].w = *niftiImgValues++;
        }
        NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, array.get(), voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            TransferNiftiToDevice<DataType, float>(arrayCuda, img);
            break;
        default:
            NR_FATAL_ERROR("The image data type is not supported");
        }
    }
}
template void TransferNiftiToDevice<int>(int*, const nifti_image*);
template void TransferNiftiToDevice<float>(float*, const nifti_image*);
template void TransferNiftiToDevice<double>(double*, const nifti_image*);
template void TransferNiftiToDevice<float4>(float4*, const nifti_image*);
/* *************************************************************** */
template<class DataType, class NiftiType>
void TransferNiftiToDevice(DataType *array1Cuda, DataType *array2Cuda, const nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType))
        NR_FATAL_ERROR("The host and device arrays are of different types");
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
    const size_t memSize = voxelNumber * sizeof(DataType);
    const NiftiType *array1 = static_cast<NiftiType*>(img->data);
    const NiftiType *array2 = &array1[voxelNumber];
    NR_CUDA_SAFE_CALL(hipMemcpy(array1Cuda, array1, memSize, hipMemcpyHostToDevice));
    NR_CUDA_SAFE_CALL(hipMemcpy(array2Cuda, array2, memSize, hipMemcpyHostToDevice));
}
/* *************************************************************** */
template<class DataType>
void TransferNiftiToDevice(DataType *array1Cuda, DataType *array2Cuda, const nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if (img->datatype != NIFTI_TYPE_FLOAT32)
            NR_FATAL_ERROR("The specified image is not a single precision image");
        const float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        const auto timePointCount = img->dim[4] * img->dim[5];
        unique_ptr<float4[]> array1(new float4[voxelNumber]());
        unique_ptr<float4[]> array2(new float4[voxelNumber]());
        for (size_t i = 0; i < voxelNumber; i++)
            array1[i].x = *niftiImgValues++;
        for (size_t i = 0; i < voxelNumber; i++)
            array2[i].x = *niftiImgValues++;
        if (timePointCount >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].y = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].y = *niftiImgValues++;
        }
        if (timePointCount >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].z = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].z = *niftiImgValues++;
        }
        if (timePointCount >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].w = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].w = *niftiImgValues++;
        }
        NR_CUDA_SAFE_CALL(hipMemcpy(array1Cuda, array1.get(), voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
        NR_CUDA_SAFE_CALL(hipMemcpy(array2Cuda, array2.get(), voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            TransferNiftiToDevice<DataType, float>(array1Cuda, array2Cuda, img);
            break;
        default:
            NR_FATAL_ERROR("The image data type is not supported");
        }
    }
}
template void TransferNiftiToDevice<float>(float*, float*, const nifti_image*);
template void TransferNiftiToDevice<double>(double*, double*, const nifti_image*);
template void TransferNiftiToDevice<float4>(float4*, float4*, const nifti_image*);
/* *************************************************************** */
template<class DataType>
void TransferNiftiToDevice(DataType *arrayCuda, const DataType *img, const size_t nvox) {
    NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, img, nvox * sizeof(DataType), hipMemcpyHostToDevice));
}
template void TransferNiftiToDevice<int>(int*, const int*, const size_t);
template void TransferNiftiToDevice<float>(float*, const float*, const size_t);
template void TransferNiftiToDevice<double>(double*, const double*, const size_t);
/* *************************************************************** */
template<class DataType, class NiftiType>
void TransferFromDeviceToNifti(nifti_image *img, const DataType *arrayCuda) {
    if (sizeof(DataType) != sizeof(NiftiType))
        NR_FATAL_ERROR("The host and device arrays are of different types");
    NR_CUDA_SAFE_CALL(hipMemcpy(img->data, arrayCuda, img->nvox * sizeof(DataType), hipMemcpyDeviceToHost));
}
/* *************************************************************** */
template<class DataType>
void TransferFromDeviceToNifti(nifti_image *img, const DataType *arrayCuda) {
    if (sizeof(DataType) == sizeof(float4)) {
        // A nifti 5D volume is expected
        if (img->datatype != NIFTI_TYPE_FLOAT32)
            NR_FATAL_ERROR("The specified image is not a single precision image");
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        const auto timePointCount = img->dim[4] * img->dim[5];
        thrust::device_ptr<const float4> arrayCudaPtr(reinterpret_cast<const float4*>(arrayCuda));
        const thrust::host_vector<float4> array(arrayCudaPtr, arrayCudaPtr + voxelNumber);
        float *niftiImgValues = static_cast<float*>(img->data);
        for (size_t i = 0; i < voxelNumber; i++)
            *niftiImgValues++ = array[i].x;
        if (timePointCount >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array[i].y;
        }
        if (timePointCount >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array[i].z;
        }
        if (timePointCount >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array[i].w;
        }
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            TransferFromDeviceToNifti<DataType, float>(img, arrayCuda);
            break;
        default:
            NR_FATAL_ERROR("The image data type is not supported");
        }
    }
}
template void TransferFromDeviceToNifti<float>(nifti_image*, const float*);
template void TransferFromDeviceToNifti<double>(nifti_image*, const double*);
template void TransferFromDeviceToNifti<float4>(nifti_image*, const float4*);
/* *************************************************************** */
template<class DataType, class NiftiType>
void TransferFromDeviceToNifti(nifti_image *img, const DataType *array1Cuda, const DataType *array2Cuda) {
    if (sizeof(DataType) != sizeof(NiftiType))
        NR_FATAL_ERROR("The host and device arrays are of different types");
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
    NiftiType *array1 = static_cast<NiftiType*>(img->data);
    NiftiType *array2 = &array1[voxelNumber];
    NR_CUDA_SAFE_CALL(hipMemcpy(array1, array1Cuda, voxelNumber * sizeof(DataType), hipMemcpyDeviceToHost));
    NR_CUDA_SAFE_CALL(hipMemcpy(array2, array2Cuda, voxelNumber * sizeof(DataType), hipMemcpyDeviceToHost));
}
/* *************************************************************** */
template<class DataType>
void TransferFromDeviceToNifti(nifti_image *img, const DataType *array1Cuda, const DataType *array2Cuda) {
    if (sizeof(DataType) == sizeof(float4)) {
        // A nifti 5D volume is expected
        if (img->datatype != NIFTI_TYPE_FLOAT32)
            NR_FATAL_ERROR("The specified image is not a single precision image");
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        const auto timePointCount = img->dim[4] * img->dim[5];
        thrust::device_ptr<const float4> array1CudaPtr(reinterpret_cast<const float4*>(array1Cuda));
        thrust::device_ptr<const float4> array2CudaPtr(reinterpret_cast<const float4*>(array2Cuda));
        const thrust::host_vector<float4> array1(array1CudaPtr, array1CudaPtr + voxelNumber);
        const thrust::host_vector<float4> array2(array2CudaPtr, array2CudaPtr + voxelNumber);
        float *niftiImgValues = static_cast<float*>(img->data);
        for (size_t i = 0; i < voxelNumber; i++)
            *niftiImgValues++ = array1[i].x;
        for (size_t i = 0; i < voxelNumber; i++)
            *niftiImgValues++ = array2[i].x;
        if (timePointCount >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array1[i].y;
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array2[i].y;
        }
        if (timePointCount >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array1[i].z;
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array2[i].z;
        }
        if (timePointCount >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array1[i].w;
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array2[i].w;
        }
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            TransferFromDeviceToNifti<DataType, float>(img, array1Cuda, array2Cuda);
            break;
        default:
            NR_FATAL_ERROR("The image data type is not supported");
        }
    }
}
template void TransferFromDeviceToNifti<float>(nifti_image*, const float*, const float*);
template void TransferFromDeviceToNifti<double>(nifti_image*, const double*, const double*);
template void TransferFromDeviceToNifti<float4>(nifti_image*, const float4*, const float4*);
/* *************************************************************** */
template<class DataType>
void TransferFromDeviceToHost(DataType *array, const DataType *arrayCuda, const size_t nElements) {
    NR_CUDA_SAFE_CALL(hipMemcpy(array, arrayCuda, nElements * sizeof(DataType), hipMemcpyDeviceToHost));
}
template void TransferFromDeviceToHost<float>(float*, const float*, const size_t);
template void TransferFromDeviceToHost<double>(double*, const double*, const size_t);
/* *************************************************************** */
template<class DataType>
void TransferFromHostToDevice(DataType *arrayCuda, const DataType *array, const size_t nElements) {
    NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, array, nElements * sizeof(DataType), hipMemcpyHostToDevice));
}
template void TransferFromHostToDevice<int>(int*, const int*, const size_t);
template void TransferFromHostToDevice<float>(float*, const float*, const size_t);
template void TransferFromHostToDevice<double>(double*, const double*, const size_t);
/* *************************************************************** */
template<class DataType>
void Free(DataType *arrayCuda) {
    if (arrayCuda != nullptr)
        NR_CUDA_SAFE_CALL(hipFree(arrayCuda));
}
template void Free<int>(int*);
template void Free<float>(float*);
template void Free<double>(double*);
template void Free<float4>(float4*);
/* *************************************************************** */
template<>
void Free(hipTextureObject_t *texObj) {
    NR_CUDA_SAFE_CALL(hipDestroyTextureObject(*texObj));
    delete texObj;
}
/* *************************************************************** */
template<class DataType>
UniqueTextureObjectPtr CreateTextureObject(const DataType *devPtr,
                                           const size_t count,
                                           const hipChannelFormatKind channelFormat,
                                           const unsigned channelCount) {
    // Specify texture
    hipResourceDesc resDesc{};
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = const_cast<DataType*>(devPtr);
    resDesc.res.linear.desc.f = channelFormat;
    resDesc.res.linear.desc.x = 32;
    if (channelCount > 1)
        resDesc.res.linear.desc.y = 32;
    if (channelCount > 2)
        resDesc.res.linear.desc.z = 32;
    if (channelCount > 3)
        resDesc.res.linear.desc.w = 32;
    resDesc.res.linear.sizeInBytes = count * sizeof(DataType);

    // Specify texture object parameters
    hipTextureDesc texDesc{};
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.addressMode[2] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = false;

    // Create texture object
    UniqueTextureObjectPtr texObj(new hipTextureObject_t());
    NR_CUDA_SAFE_CALL(hipCreateTextureObject(texObj.get(), &resDesc, &texDesc, nullptr));

    return texObj;
}
template UniqueTextureObjectPtr CreateTextureObject<bool>(const bool*, const size_t, const hipChannelFormatKind, const unsigned);
template UniqueTextureObjectPtr CreateTextureObject<int>(const int*, const size_t, const hipChannelFormatKind, const unsigned);
template UniqueTextureObjectPtr CreateTextureObject<float>(const float*, const size_t, const hipChannelFormatKind, const unsigned);
template UniqueTextureObjectPtr CreateTextureObject<float2>(const float2*, const size_t, const hipChannelFormatKind, const unsigned);
template UniqueTextureObjectPtr CreateTextureObject<float4>(const float4*, const size_t, const hipChannelFormatKind, const unsigned);
template UniqueTextureObjectPtr CreateTextureObject<mat33>(const mat33*, const size_t, const hipChannelFormatKind, const unsigned);
/* *************************************************************** */
} // namespace NiftyReg::Cuda
/* *************************************************************** */
