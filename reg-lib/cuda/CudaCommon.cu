#include "hip/hip_runtime.h"
/**
 * @file CudaCommon.cu
 * @author Marc Modat
 * @date 25/03/2009
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 * See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#include "CudaCommon.hpp"
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>

/* *************************************************************** */
namespace NiftyReg::Cuda {
/* *************************************************************** */
template <class DataType>
void Allocate(hipArray **arrayCuda, const int *dim) {
    const hipExtent volumeSize = make_hipExtent(std::abs(dim[1]), std::abs(dim[2]), std::abs(dim[3]));
    const hipChannelFormatDesc texDesc = hipCreateChannelDesc<DataType>();
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(arrayCuda, &texDesc, volumeSize));
}
template void Allocate<float>(hipArray**, const int*);
template void Allocate<double>(hipArray**, const int*);
template void Allocate<float4>(hipArray**, const int*); // for deformation field
/* *************************************************************** */
template <class DataType>
void Allocate(hipArray **array1Cuda, hipArray **array2Cuda, const int *dim) {
    const hipExtent volumeSize = make_hipExtent(std::abs(dim[1]), std::abs(dim[2]), std::abs(dim[3]));
    const hipChannelFormatDesc texDesc = hipCreateChannelDesc<DataType>();
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(array1Cuda, &texDesc, volumeSize));
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(array2Cuda, &texDesc, volumeSize));
}
template void Allocate<float>(hipArray**, hipArray**, const int*);
template void Allocate<double>(hipArray**, hipArray**, const int*);
template void Allocate<float4>(hipArray**, hipArray**, const int*); // for deformation field
/* *************************************************************** */
template <class DataType>
void Allocate(DataType **arrayCuda, const size_t& nVoxels) {
    NR_CUDA_SAFE_CALL(hipMalloc(arrayCuda, nVoxels * sizeof(DataType)));
}
template void Allocate<int>(int**, const size_t&);
template void Allocate<float>(float**, const size_t&);
template void Allocate<double>(double**, const size_t&);
template void Allocate<float4>(float4**, const size_t&); // for deformation field
/* *************************************************************** */
template <class DataType>
void Allocate(DataType **arrayCuda, const int *dim) {
    const size_t memSize = (size_t)std::abs(dim[1]) * (size_t)std::abs(dim[2]) * (size_t)std::abs(dim[3]) * sizeof(DataType);
    NR_CUDA_SAFE_CALL(hipMalloc(arrayCuda, memSize));
}
template void Allocate<int>(int**, const int*);
template void Allocate<float>(float**, const int*);
template void Allocate<double>(double**, const int*);
template void Allocate<float4>(float4**, const int*); // for deformation field
/* *************************************************************** */
template <class DataType>
void Allocate(DataType **array1Cuda, DataType **array2Cuda, const int *dim) {
    const size_t memSize = (size_t)std::abs(dim[1]) * (size_t)std::abs(dim[2]) * (size_t)std::abs(dim[3]) * sizeof(DataType);
    NR_CUDA_SAFE_CALL(hipMalloc(array1Cuda, memSize));
    NR_CUDA_SAFE_CALL(hipMalloc(array2Cuda, memSize));
}
template void Allocate<float>(float**, float**, const int*);
template void Allocate<double>(double**, double**, const int*);
template void Allocate<float4>(float4**, float4**, const int*); // for deformation field
/* *************************************************************** */
template <class DataType, class NiftiType>
void TransferNiftiToDevice(hipArray *arrayCuda, const nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType))
        NR_FATAL_ERROR("The host and device arrays are of different types");
    hipMemcpy3DParms copyParams{};
    copyParams.extent = make_hipExtent(std::abs(img->dim[1]), std::abs(img->dim[2]), std::abs(img->dim[3]));
    copyParams.srcPtr = make_hipPitchedPtr(img->data,
                                            copyParams.extent.width * sizeof(DataType),
                                            copyParams.extent.width,
                                            copyParams.extent.height);
    copyParams.dstArray = arrayCuda;
    copyParams.kind = hipMemcpyHostToDevice;
    NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
}
/* *************************************************************** */
template <class DataType>
void TransferNiftiToDevice(hipArray *arrayCuda, const nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if (img->datatype != NIFTI_TYPE_FLOAT32 || img->dim[5] < 2 || img->dim[4] > 1)
            NR_FATAL_ERROR("The specified image is not a single precision deformation field image");
        const float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        unique_ptr<float4[]> array(new float4[voxelNumber]());
        for (size_t i = 0; i < voxelNumber; i++)
            array[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].z = *niftiImgValues++;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].w = *niftiImgValues++;
        }
        hipMemcpy3DParms copyParams{};
        copyParams.extent = make_hipExtent(std::abs(img->dim[1]), std::abs(img->dim[2]), std::abs(img->dim[3]));
        copyParams.srcPtr = make_hipPitchedPtr(array.get(),
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = arrayCuda;
        copyParams.kind = hipMemcpyHostToDevice;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            TransferNiftiToDevice<DataType, float>(arrayCuda, img);
            break;
        default:
            NR_FATAL_ERROR("The image data type is not supported");
        }
    }
}
template void TransferNiftiToDevice<int>(hipArray*, const nifti_image*);
template void TransferNiftiToDevice<float>(hipArray*, const nifti_image*);
template void TransferNiftiToDevice<double>(hipArray*, const nifti_image*);
template void TransferNiftiToDevice<float4>(hipArray*, const nifti_image*); // for deformation field
/* *************************************************************** */
template <class DataType, class NiftiType>
void TransferNiftiToDevice(hipArray *array1Cuda, hipArray *array2Cuda, const nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType))
        NR_FATAL_ERROR("The host and device arrays are of different types");
    NiftiType *array1 = static_cast<NiftiType*>(img->data);
    NiftiType *array2 = &array1[NiftiImage::calcVoxelNumber(img, 3)];
    hipMemcpy3DParms copyParams{};
    copyParams.extent = make_hipExtent(std::abs(img->dim[1]), std::abs(img->dim[2]), std::abs(img->dim[3]));
    copyParams.kind = hipMemcpyHostToDevice;
    // First timepoint
    copyParams.srcPtr = make_hipPitchedPtr(array1,
                                            copyParams.extent.width * sizeof(DataType),
                                            copyParams.extent.width,
                                            copyParams.extent.height);
    copyParams.dstArray = array1Cuda;
    NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    // Second timepoint
    copyParams.srcPtr = make_hipPitchedPtr(array2,
                                            copyParams.extent.width * sizeof(DataType),
                                            copyParams.extent.width,
                                            copyParams.extent.height);
    copyParams.dstArray = array2Cuda;
    NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
}
/* *************************************************************** */
template <class DataType>
void TransferNiftiToDevice(hipArray *array1Cuda, hipArray *array2Cuda, const nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if (img->datatype != NIFTI_TYPE_FLOAT32 || img->dim[5] < 2 || img->dim[4] > 1)
            NR_FATAL_ERROR("The specified image is not a single precision deformation field image");
        const float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        unique_ptr<float4[]> array1(new float4[voxelNumber]());
        unique_ptr<float4[]> array2(new float4[voxelNumber]());
        for (size_t i = 0; i < voxelNumber; i++)
            array1[i].x = *niftiImgValues++;
        for (size_t i = 0; i < voxelNumber; i++)
            array2[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].y = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].z = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].z = *niftiImgValues++;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].w = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].w = *niftiImgValues++;
        }

        hipMemcpy3DParms copyParams{};
        copyParams.extent = make_hipExtent(std::abs(img->dim[1]), std::abs(img->dim[2]), std::abs(img->dim[3]));
        copyParams.kind = hipMemcpyHostToDevice;
        // First timepoint
        copyParams.srcPtr = make_hipPitchedPtr(array1.get(),
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = array1Cuda;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
        // Second timepoint
        copyParams.srcPtr = make_hipPitchedPtr(array2.get(),
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = array2Cuda;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            TransferNiftiToDevice<DataType, float>(array1Cuda, array2Cuda, img);
            break;
        default:
            NR_FATAL_ERROR("The image data type is not supported");
        }
    }
}
template void TransferNiftiToDevice<float>(hipArray*, hipArray*, const nifti_image*);
template void TransferNiftiToDevice<double>(hipArray*, hipArray*, const nifti_image*);
template void TransferNiftiToDevice<float4>(hipArray*, hipArray*, const nifti_image*); // for deformation field
/* *************************************************************** */
template <class DataType, class NiftiType>
void TransferNiftiToDevice(DataType *arrayCuda, const nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType))
        NR_FATAL_ERROR("The host and device arrays are of different types");
    NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, img->data, img->nvox * sizeof(NiftiType), hipMemcpyHostToDevice));
}
/* *************************************************************** */
template <class DataType>
void TransferNiftiToDevice(DataType *arrayCuda, const nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if (img->datatype != NIFTI_TYPE_FLOAT32 || img->dim[5] < 2 || img->dim[4] > 1)
            NR_FATAL_ERROR("The specified image is not a single precision deformation field image");
        const float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        unique_ptr<float4[]> array(new float4[voxelNumber]());
        for (size_t i = 0; i < voxelNumber; i++)
            array[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].z = *niftiImgValues++;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].w = *niftiImgValues++;
        }
        NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, array.get(), voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            TransferNiftiToDevice<DataType, float>(arrayCuda, img);
            break;
        default:
            NR_FATAL_ERROR("The image data type is not supported");
        }
    }
}
template void TransferNiftiToDevice<int>(int*, const nifti_image*);
template void TransferNiftiToDevice<float>(float*, const nifti_image*);
template void TransferNiftiToDevice<double>(double*, const nifti_image*);
template void TransferNiftiToDevice<float4>(float4*, const nifti_image*);
/* *************************************************************** */
template <class DataType, class NiftiType>
void TransferNiftiToDevice(DataType *array1Cuda, DataType *array2Cuda, const nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType))
        NR_FATAL_ERROR("The host and device arrays are of different types");
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
    const size_t memSize = voxelNumber * sizeof(DataType);
    const NiftiType *array1 = static_cast<NiftiType*>(img->data);
    const NiftiType *array2 = &array1[voxelNumber];
    NR_CUDA_SAFE_CALL(hipMemcpy(array1Cuda, array1, memSize, hipMemcpyHostToDevice));
    NR_CUDA_SAFE_CALL(hipMemcpy(array2Cuda, array2, memSize, hipMemcpyHostToDevice));
}
/* *************************************************************** */
template <class DataType>
void TransferNiftiToDevice(DataType *array1Cuda, DataType *array2Cuda, const nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if (img->datatype != NIFTI_TYPE_FLOAT32 || img->dim[5] < 2 || img->dim[4] > 1)
            NR_FATAL_ERROR("The specified image is not a single precision deformation field image");
        const float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        unique_ptr<float4[]> array1(new float4[voxelNumber]());
        unique_ptr<float4[]> array2(new float4[voxelNumber]());
        for (size_t i = 0; i < voxelNumber; i++)
            array1[i].x = *niftiImgValues++;
        for (size_t i = 0; i < voxelNumber; i++)
            array2[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].y = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].z = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].z = *niftiImgValues++;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].w = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].w = *niftiImgValues++;
        }
        NR_CUDA_SAFE_CALL(hipMemcpy(array1Cuda, array1.get(), voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
        NR_CUDA_SAFE_CALL(hipMemcpy(array2Cuda, array2.get(), voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            TransferNiftiToDevice<DataType, float>(array1Cuda, array2Cuda, img);
            break;
        default:
            NR_FATAL_ERROR("The image data type is not supported");
        }
    }
}
template void TransferNiftiToDevice<float>(float*, float*, const nifti_image*);
template void TransferNiftiToDevice<double>(double*, double*, const nifti_image*);
template void TransferNiftiToDevice<float4>(float4*, float4*, const nifti_image*); // for deformation field
/* *************************************************************** */
template <class DataType>
void TransferNiftiToDevice(DataType *arrayCuda, const DataType *img, const size_t& nvox) {
    NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, img, nvox * sizeof(DataType), hipMemcpyHostToDevice));
}
template void TransferNiftiToDevice<int>(int*, const int*, const size_t&);
template void TransferNiftiToDevice<float>(float*, const float*, const size_t&);
template void TransferNiftiToDevice<double>(double*, const double*, const size_t&);
/* *************************************************************** */
void TransferFromDeviceToNifti(nifti_image *img, const hipArray *arrayCuda) {
    if (img->datatype != NIFTI_TYPE_FLOAT32)
        NR_FATAL_ERROR("The image data type is not supported");
    hipMemcpy3DParms copyParams{};
    copyParams.extent = make_hipExtent(std::abs(img->dim[1]), std::abs(img->dim[2]), std::abs(img->dim[3]));
    copyParams.srcArray = const_cast<hipArray*>(arrayCuda);
    copyParams.dstPtr = make_hipPitchedPtr(img->data,
                                            copyParams.extent.width * sizeof(float),
                                            copyParams.extent.width,
                                            copyParams.extent.height);
    copyParams.kind = hipMemcpyDeviceToHost;
    NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
}
/* *************************************************************** */
template <class DataType, class NiftiType>
void TransferFromDeviceToNifti(nifti_image *img, const DataType *arrayCuda) {
    if (sizeof(DataType) != sizeof(NiftiType))
        NR_FATAL_ERROR("The host and device arrays are of different types");
    NR_CUDA_SAFE_CALL(hipMemcpy(img->data, arrayCuda, img->nvox * sizeof(DataType), hipMemcpyDeviceToHost));
}
/* *************************************************************** */
template <class DataType>
void TransferFromDeviceToNifti(nifti_image *img, const DataType *arrayCuda) {
    if (sizeof(DataType) == sizeof(float4)) {
        // A nifti 5D volume is expected
        if (img->dim[0] < 5 || img->dim[4]>1 || img->dim[5] < 2 || img->datatype != NIFTI_TYPE_FLOAT32)
            NR_FATAL_ERROR("The nifti image is not a 5D volume");
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        thrust::device_ptr<const float4> arrayCudaPtr(reinterpret_cast<const float4*>(arrayCuda));
        const thrust::host_vector<float4> array(arrayCudaPtr, arrayCudaPtr + voxelNumber);
        float *niftiImgValues = static_cast<float*>(img->data);
        for (size_t i = 0; i < voxelNumber; i++)
            *niftiImgValues++ = array[i].x;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array[i].y;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array[i].z;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array[i].w;
        }
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            TransferFromDeviceToNifti<DataType, float>(img, arrayCuda);
            break;
        default:
            NR_FATAL_ERROR("The image data type is not supported");
        }
    }
}
template void TransferFromDeviceToNifti<float>(nifti_image*, const float*);
template void TransferFromDeviceToNifti<double>(nifti_image*, const double*);
template void TransferFromDeviceToNifti<float4>(nifti_image*, const float4*); // for deformation field
/* *************************************************************** */
template <class DataType, class NiftiType>
void TransferFromDeviceToNifti(nifti_image *img, const DataType *array1Cuda, const DataType *array2Cuda) {
    if (sizeof(DataType) != sizeof(NiftiType))
        NR_FATAL_ERROR("The host and device arrays are of different types");
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
    NiftiType *array1 = static_cast<NiftiType*>(img->data);
    NiftiType *array2 = &array1[voxelNumber];
    NR_CUDA_SAFE_CALL(hipMemcpy(array1, array1Cuda, voxelNumber * sizeof(DataType), hipMemcpyDeviceToHost));
    NR_CUDA_SAFE_CALL(hipMemcpy(array2, array2Cuda, voxelNumber * sizeof(DataType), hipMemcpyDeviceToHost));
}
/* *************************************************************** */
template <class DataType>
void TransferFromDeviceToNifti(nifti_image *img, const DataType *array1Cuda, const DataType *array2Cuda) {
    if (sizeof(DataType) == sizeof(float4)) {
        // A nifti 5D volume is expected
        if (img->dim[0] < 5 || img->dim[4]>1 || img->dim[5] < 2 || img->datatype != NIFTI_TYPE_FLOAT32)
            NR_FATAL_ERROR("The nifti image is not a 5D volume");
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        thrust::device_ptr<const float4> array1CudaPtr(reinterpret_cast<const float4*>(array1Cuda));
        thrust::device_ptr<const float4> array2CudaPtr(reinterpret_cast<const float4*>(array2Cuda));
        const thrust::host_vector<float4> array1(array1CudaPtr, array1CudaPtr + voxelNumber);
        const thrust::host_vector<float4> array2(array2CudaPtr, array2CudaPtr + voxelNumber);
        float *niftiImgValues = static_cast<float*>(img->data);
        for (size_t i = 0; i < voxelNumber; i++)
            *niftiImgValues++ = array1[i].x;
        for (size_t i = 0; i < voxelNumber; i++)
            *niftiImgValues++ = array2[i].x;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array1[i].y;
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array2[i].y;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array1[i].z;
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array2[i].z;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array1[i].w;
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array2[i].w;
        }
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            TransferFromDeviceToNifti<DataType, float>(img, array1Cuda, array2Cuda);
            break;
        default:
            NR_FATAL_ERROR("The image data type is not supported");
        }
    }
}
template void TransferFromDeviceToNifti<float>(nifti_image*, const float*, const float*);
template void TransferFromDeviceToNifti<double>(nifti_image*, const double*, const double*);
template void TransferFromDeviceToNifti<float4>(nifti_image*, const float4*, const float4*); // for deformation field
/* *************************************************************** */
template <class DataType>
void TransferFromDeviceToHost(DataType *array, const DataType *arrayCuda, const size_t& nElements) {
    NR_CUDA_SAFE_CALL(hipMemcpy(array, arrayCuda, nElements * sizeof(DataType), hipMemcpyDeviceToHost));
}
template void TransferFromDeviceToHost<float>(float*, const float*, const size_t&);
template void TransferFromDeviceToHost<double>(double*, const double*, const size_t&);
/* *************************************************************** */
template <class DataType>
void TransferFromHostToDevice(DataType *arrayCuda, const DataType *array, const size_t& nElements) {
    NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, array, nElements * sizeof(DataType), hipMemcpyHostToDevice));
}
template void TransferFromHostToDevice<int>(int*, const int*, const size_t&);
template void TransferFromHostToDevice<float>(float*, const float*, const size_t&);
template void TransferFromHostToDevice<double>(double*, const double*, const size_t&);
/* *************************************************************** */
void Free(hipArray *arrayCuda) {
    if (arrayCuda != nullptr)
        NR_CUDA_SAFE_CALL(hipFreeArray(arrayCuda));
}
/* *************************************************************** */
template <class DataType>
void Free(DataType *arrayCuda) {
    if (arrayCuda != nullptr)
        NR_CUDA_SAFE_CALL(hipFree(arrayCuda));
}
template void Free<int>(int*);
template void Free<float>(float*);
template void Free<double>(double*);
template void Free<float4>(float4*);
/* *************************************************************** */
void DestroyTextureObject(hipTextureObject_t *texObj) {
    NR_CUDA_SAFE_CALL(hipDestroyTextureObject(*texObj));
    delete texObj;
}
/* *************************************************************** */
UniqueTextureObjectPtr CreateTextureObject(const void *devPtr,
                                           const hipResourceType& resType,
                                           const size_t& size,
                                           const hipChannelFormatKind& channelFormat,
                                           const unsigned& channelCount,
                                           const hipTextureFilterMode& filterMode,
                                           const bool& normalizedCoordinates) {
    // Specify texture
    hipResourceDesc resDesc{};
    resDesc.resType = resType;
    switch (resType) {
    case hipResourceTypeLinear:
        resDesc.res.linear.devPtr = const_cast<void*>(devPtr);
        resDesc.res.linear.desc.f = channelFormat;
        resDesc.res.linear.desc.x = 32;
        if (channelCount > 1)
            resDesc.res.linear.desc.y = 32;
        if (channelCount > 2)
            resDesc.res.linear.desc.z = 32;
        if (channelCount > 3)
            resDesc.res.linear.desc.w = 32;
        resDesc.res.linear.sizeInBytes = size;
        break;
    case hipResourceTypeArray:
        resDesc.res.array.array = static_cast<hipArray*>(const_cast<void*>(devPtr));
        break;
    default:
        NR_FATAL_ERROR("Unsupported resource type");
    }

    // Specify texture object parameters
    hipTextureDesc texDesc{};
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.addressMode[2] = hipAddressModeWrap;
    texDesc.filterMode = filterMode;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = normalizedCoordinates;

    // Create texture object
    UniqueTextureObjectPtr texObj(new hipTextureObject_t(), DestroyTextureObject);
    NR_CUDA_SAFE_CALL(hipCreateTextureObject(texObj.get(), &resDesc, &texDesc, nullptr));

    return texObj;
}
/* *************************************************************** */
} // namespace NiftyReg::Cuda
/* *************************************************************** */
