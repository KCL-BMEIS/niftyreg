#include "hip/hip_runtime.h"
#include "_reg_optimiser_gpu.h"
#include "_reg_optimiser_kernels.cu"
#include "_reg_common_cuda_kernels.cu"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/inner_product.h>

/* *************************************************************** */
reg_optimiser_gpu::reg_optimiser_gpu(): reg_optimiser<float>::reg_optimiser() {
    this->currentDofCuda = nullptr;
    this->currentDofBwCuda = nullptr;
    this->bestDofCuda = nullptr;
    this->bestDofBwCuda = nullptr;
    this->gradientCuda = nullptr;
    this->gradientBwCuda = nullptr;
#ifndef NDEBUG
    reg_print_msg_debug("reg_optimiser_gpu::reg_optimiser_gpu() called\n");
#endif
}
/* *************************************************************** */
reg_optimiser_gpu::~reg_optimiser_gpu() {
    if (this->bestDofCuda) {
        cudaCommon_free(this->bestDofCuda);
        this->bestDofCuda = nullptr;
    }
    if (this->bestDofBwCuda) {
        cudaCommon_free(this->bestDofBwCuda);
        this->bestDofBwCuda = nullptr;
    }
#ifndef NDEBUG
    reg_print_msg_debug("reg_optimiser_gpu::~reg_optimiser_gpu() called\n");
#endif
}
/* *************************************************************** */
void reg_optimiser_gpu::Initialise(size_t nvox,
                                   int ndim,
                                   bool optX,
                                   bool optY,
                                   bool optZ,
                                   size_t maxIt,
                                   size_t startIt,
                                   InterfaceOptimiser *intOpt,
                                   float *cppData,
                                   float *gradData,
                                   size_t nvoxBw,
                                   float *cppDataBw,
                                   float *gradDataBw) {
    this->dofNumber = nvox;
    this->ndim = ndim;
    this->optimiseX = optX;
    this->optimiseY = optY;
    this->optimiseZ = optZ;
    this->maxIterationNumber = maxIt;
    this->currentIterationNumber = startIt;
    this->currentDofCuda = reinterpret_cast<float4*>(cppData);
    this->gradientCuda = reinterpret_cast<float4*>(gradData);

    cudaCommon_free(this->bestDofCuda);
    if (cudaCommon_allocateArrayToDevice(&this->bestDofCuda, (int)this->GetVoxNumber())) {
        reg_print_fct_error("reg_optimiser_gpu::Initialise()");
        reg_print_msg_error("Error when allocating the best control point array on the GPU");
        reg_exit();
    }

    this->isSymmetric = nvoxBw > 0 && cppDataBw && gradDataBw;
    if (this->isSymmetric) {
        this->dofNumberBw = nvoxBw;
        this->currentDofBwCuda = reinterpret_cast<float4*>(cppDataBw);
        this->gradientBwCuda = reinterpret_cast<float4*>(gradDataBw);
        cudaCommon_free(this->bestDofBwCuda);
        if (cudaCommon_allocateArrayToDevice(&this->bestDofBwCuda, (int)this->GetVoxNumberBw())) {
            reg_print_fct_error("reg_optimiser_gpu::Initialise()");
            reg_print_msg_error("Error when allocating the best control point backwards array on the GPU");
            reg_exit();
        }
    }

    this->StoreCurrentDof();

    this->intOpt = intOpt;
    this->bestObjFunctionValue = this->currentObjFunctionValue = this->intOpt->GetObjectiveFunctionValue();

#ifndef NDEBUG
    reg_print_msg_debug("reg_optimiser_gpu::Initialise() called");
#endif
}
/* *************************************************************** */
void reg_optimiser_gpu::RestoreBestDof() {
    // Restore forward transformation
    NR_CUDA_SAFE_CALL(hipMemcpy(this->currentDofCuda, this->bestDofCuda, this->GetVoxNumber() * sizeof(float4), hipMemcpyDeviceToDevice));
    // Restore backward transformation if required
    if (this->isSymmetric)
        NR_CUDA_SAFE_CALL(hipMemcpy(this->currentDofBwCuda, this->bestDofBwCuda, this->GetVoxNumberBw() * sizeof(float4), hipMemcpyDeviceToDevice));
}
/* *************************************************************** */
void reg_optimiser_gpu::StoreCurrentDof() {
    // Store forward transformation
    NR_CUDA_SAFE_CALL(hipMemcpy(this->bestDofCuda, this->currentDofCuda, this->GetVoxNumber() * sizeof(float4), hipMemcpyDeviceToDevice));
    // Store backward transformation if required
    if (this->isSymmetric)
        NR_CUDA_SAFE_CALL(hipMemcpy(this->bestDofBwCuda, this->currentDofBwCuda, this->GetVoxNumberBw() * sizeof(float4), hipMemcpyDeviceToDevice));
}
/* *************************************************************** */
void reg_optimiser_gpu::Perturbation(float length) {
    // TODO: Implement reg_optimiser_gpu::Perturbation()
}
/* *************************************************************** */
reg_conjugateGradient_gpu::reg_conjugateGradient_gpu(): reg_optimiser_gpu::reg_optimiser_gpu() {
    this->array1 = nullptr;
    this->array1Bw = nullptr;
    this->array2 = nullptr;
    this->array2Bw = nullptr;
#ifndef NDEBUG
    reg_print_msg_debug("reg_conjugateGradient_gpu::reg_conjugateGradient_gpu() called");
#endif
}
/* *************************************************************** */
reg_conjugateGradient_gpu::~reg_conjugateGradient_gpu() {
    if (this->array1) {
        cudaCommon_free(this->array1);
        this->array1 = nullptr;
    }
    if (this->array1Bw) {
        cudaCommon_free(this->array1Bw);
        this->array1Bw = nullptr;
    }
    if (this->array2) {
        cudaCommon_free(this->array2);
        this->array2 = nullptr;
    }
    if (this->array2Bw) {
        cudaCommon_free(this->array2Bw);
        this->array2Bw = nullptr;
    }
#ifndef NDEBUG
    reg_print_msg_debug("reg_conjugateGradient_gpu::~reg_conjugateGradient_gpu() called");
#endif
}
/* *************************************************************** */
void reg_conjugateGradient_gpu::Initialise(size_t nvox,
                                           int ndim,
                                           bool optX,
                                           bool optY,
                                           bool optZ,
                                           size_t maxIt,
                                           size_t startIt,
                                           InterfaceOptimiser *intOpt,
                                           float *cppData,
                                           float *gradData,
                                           size_t nvoxBw,
                                           float *cppDataBw,
                                           float *gradDataBw) {
    reg_optimiser_gpu::Initialise(nvox, ndim, optX, optY, optZ, maxIt, startIt, intOpt, cppData, gradData, nvoxBw, cppDataBw, gradDataBw);
    this->firstCall = true;
    cudaCommon_free(this->array1); cudaCommon_free(this->array2);
    if (cudaCommon_allocateArrayToDevice<float4>(&this->array1, (int)this->GetVoxNumber()) ||
        cudaCommon_allocateArrayToDevice<float4>(&this->array2, (int)this->GetVoxNumber())) {
        reg_print_fct_error("reg_conjugateGradient_gpu::Initialise()");
        reg_print_msg_error("Error when allocating the conjugate gradient array on the GPU");
        reg_exit();
    }
    if (this->isSymmetric) {
        cudaCommon_free(this->array1Bw); cudaCommon_free(this->array2Bw);
        if (cudaCommon_allocateArrayToDevice<float4>(&this->array1Bw, (int)this->GetVoxNumberBw()) ||
            cudaCommon_allocateArrayToDevice<float4>(&this->array2Bw, (int)this->GetVoxNumberBw())) {
            reg_print_fct_error("reg_conjugateGradient_gpu::Initialise()");
            reg_print_msg_error("Error when allocating the conjugate gradient array backwards on the GPU");
            reg_exit();
        }
    }
#ifndef NDEBUG
    reg_print_msg_debug("reg_conjugateGradient_gpu::Initialise() called");
#endif
}
/* *************************************************************** */
void reg_conjugateGradient_gpu::UpdateGradientValues() {
    if (this->firstCall) {
        reg_initialiseConjugateGradient_gpu(this->gradientCuda, this->array1, this->array2, this->GetVoxNumber());
        if (this->isSymmetric)
            reg_initialiseConjugateGradient_gpu(this->gradientBwCuda, this->array1Bw, this->array2Bw, this->GetVoxNumberBw());
        this->firstCall = false;
    } else {
        reg_getConjugateGradient_gpu(this->gradientCuda, this->array1, this->array2, this->GetVoxNumber(),
                                     this->isSymmetric, this->gradientBwCuda, this->array1Bw, this->array2Bw, this->GetVoxNumberBw());
    }
}
/* *************************************************************** */
void reg_conjugateGradient_gpu::Optimise(float maxLength,
                                         float smallLength,
                                         float& startLength) {
    this->UpdateGradientValues();
    reg_optimiser::Optimise(maxLength, smallLength, startLength);
}
/* *************************************************************** */
void reg_conjugateGradient_gpu::Perturbation(float length) {
    reg_optimiser_gpu::Perturbation(length);
    this->firstCall = true;
}
/* *************************************************************** */
void reg_initialiseConjugateGradient_gpu(float4 *gradientImageCuda,
                                         float4 *conjugateGCuda,
                                         float4 *conjugateHCuda,
                                         const size_t& nVoxels) {
    auto gradientImageTexture = cudaCommon_createTextureObject(gradientImageCuda, hipResourceTypeLinear,
                                                               nVoxels * sizeof(float4), hipChannelFormatKindFloat, 4);

    const unsigned blocks = NiftyReg::CudaContext::GetBlockSize()->reg_initialiseConjugateGradient;
    const unsigned grids = (unsigned)reg_ceil(sqrtf((float)nVoxels / (float)blocks));
    const dim3 gridDims(grids, grids, 1);
    const dim3 blockDims(blocks, 1, 1);

    reg_initialiseConjugateGradient_kernel<<<gridDims, blockDims>>>(conjugateGCuda, *gradientImageTexture, (unsigned)nVoxels);
    NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    NR_CUDA_SAFE_CALL(hipMemcpy(conjugateHCuda, conjugateGCuda, nVoxels * sizeof(float4), hipMemcpyDeviceToDevice));
}
/* *************************************************************** */
struct Float2Sum {
    __host__ __device__ double2 operator()(const float2& a, const float2& b) const {
        return make_double2((double)a.x + (double)b.x, (double)a.y + (double)b.y);
    }
};
/* *************************************************************** */
void reg_getConjugateGradient_gpu(float4 *gradientImageCuda,
                                  float4 *conjugateGCuda,
                                  float4 *conjugateHCuda,
                                  const size_t& nVoxels,
                                  const bool& isSymmetric,
                                  float4 *gradientImageBwCuda,
                                  float4 *conjugateGBwCuda,
                                  float4 *conjugateHBwCuda,
                                  const size_t& nVoxelsBw) {
    auto gradientImageTexture = cudaCommon_createTextureObject(gradientImageCuda, hipResourceTypeLinear,
                                                               nVoxels * sizeof(float4), hipChannelFormatKindFloat, 4);
    auto conjugateGTexture = cudaCommon_createTextureObject(conjugateGCuda, hipResourceTypeLinear,
                                                            nVoxels * sizeof(float4), hipChannelFormatKindFloat, 4);
    auto conjugateHTexture = cudaCommon_createTextureObject(conjugateHCuda, hipResourceTypeLinear,
                                                            nVoxels * sizeof(float4), hipChannelFormatKindFloat, 4);
    UniqueTextureObjectPtr gradientImageBwTexture(nullptr, nullptr), conjugateGBwTexture(nullptr, nullptr), conjugateHBwTexture(nullptr, nullptr);
    if (isSymmetric) {
        gradientImageBwTexture = std::move(cudaCommon_createTextureObject(gradientImageBwCuda, hipResourceTypeLinear,
                                                                          nVoxelsBw * sizeof(float4), hipChannelFormatKindFloat, 4));
        conjugateGBwTexture = std::move(cudaCommon_createTextureObject(conjugateGBwCuda, hipResourceTypeLinear,
                                                                       nVoxelsBw * sizeof(float4), hipChannelFormatKindFloat, 4));
        conjugateHBwTexture = std::move(cudaCommon_createTextureObject(conjugateHBwCuda, hipResourceTypeLinear,
                                                                       nVoxelsBw * sizeof(float4), hipChannelFormatKindFloat, 4));
    }

    // gam = sum((grad+g)*grad)/sum(HxG);
    unsigned blocks = NiftyReg::CudaContext::GetBlockSize()->reg_getConjugateGradient1;
    unsigned grids = (unsigned)reg_ceil(sqrtf((float)nVoxels / (float)blocks));
    dim3 blockDims(blocks, 1, 1);
    dim3 gridDims(grids, grids, 1);

    thrust::device_vector<float2> sumsCuda(nVoxels + nVoxels % 2);  // Make it even for thrust::inner_product
    reg_getConjugateGradient1_kernel<<<gridDims, blockDims>>>(sumsCuda.data().get(), *gradientImageTexture,
                                                              *conjugateGTexture, *conjugateHTexture, (unsigned)nVoxels);
    NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    const size_t sumsSizeHalf = sumsCuda.size() / 2;
    const double2 gg = thrust::inner_product(sumsCuda.begin(), sumsCuda.begin() + sumsSizeHalf, sumsCuda.begin() + sumsSizeHalf,
                                             make_double2(0, 0), thrust::plus<double2>(), Float2Sum());
    float gam = static_cast<float>(gg.x / gg.y);
    if (isSymmetric) {
        grids = (unsigned)reg_ceil(sqrtf((float)nVoxelsBw / (float)blocks));
        gridDims = dim3(blocks, 1, 1);
        blockDims = dim3(grids, grids, 1);
        thrust::device_vector<float2> sumsBwCuda(nVoxelsBw + nVoxelsBw % 2);  // Make it even for thrust::inner_product
        reg_getConjugateGradient1_kernel<<<gridDims, blockDims>>>(sumsBwCuda.data().get(), *gradientImageBwTexture,
                                                                  *conjugateGBwTexture, *conjugateHBwTexture, (unsigned)nVoxelsBw);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
        const size_t sumsBwSizeHalf = sumsBwCuda.size() / 2;
        const double2 ggBw = thrust::inner_product(sumsBwCuda.begin(), sumsBwCuda.begin() + sumsBwSizeHalf, sumsBwCuda.begin() + sumsBwSizeHalf,
                                                   make_double2(0, 0), thrust::plus<double2>(), Float2Sum());
        gam = static_cast<float>((gg.x + ggBw.x) / (gg.y + ggBw.y));
    }

    blocks = (unsigned)NiftyReg::CudaContext::GetBlockSize()->reg_getConjugateGradient2;
    grids = (unsigned)reg_ceil(sqrtf((float)nVoxels / (float)blocks));
    gridDims = dim3(blocks, 1, 1);
    blockDims = dim3(grids, grids, 1);
    reg_getConjugateGradient2_kernel<<<blockDims, gridDims>>>(gradientImageCuda, conjugateGCuda, conjugateHCuda, (unsigned)nVoxels, gam);
    NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    if (isSymmetric) {
        grids = (unsigned)reg_ceil(sqrtf((float)nVoxelsBw / (float)blocks));
        gridDims = dim3(blocks, 1, 1);
        blockDims = dim3(grids, grids, 1);
        reg_getConjugateGradient2_kernel<<<blockDims, gridDims>>>(gradientImageBwCuda, conjugateGBwCuda, conjugateHBwCuda, (unsigned)nVoxelsBw, gam);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }
}
/* *************************************************************** */
void reg_updateControlPointPosition_gpu(const size_t& nVoxels,
                                        float4 *controlPointImageCuda,
                                        const float4 *bestControlPointCuda,
                                        const float4 *gradientImageCuda,
                                        const float& scale,
                                        const bool& optimiseX,
                                        const bool& optimiseY,
                                        const bool& optimiseZ) {
    auto bestControlPointTexture = cudaCommon_createTextureObject(bestControlPointCuda, hipResourceTypeLinear,
                                                                  nVoxels * sizeof(float4), hipChannelFormatKindFloat, 4);
    auto gradientImageTexture = cudaCommon_createTextureObject(gradientImageCuda, hipResourceTypeLinear,
                                                               nVoxels * sizeof(float4), hipChannelFormatKindFloat, 4);

    const unsigned blocks = (unsigned)NiftyReg::CudaContext::GetBlockSize()->reg_updateControlPointPosition;
    const unsigned grids = (unsigned)reg_ceil(sqrtf((float)nVoxels / (float)blocks));
    const dim3 blockDims(blocks, 1, 1);
    const dim3 gridDims(grids, grids, 1);
    reg_updateControlPointPosition_kernel<<<gridDims, blockDims>>>(controlPointImageCuda, *bestControlPointTexture, *gradientImageTexture, (unsigned)nVoxels, scale, optimiseX, optimiseY, optimiseZ);
    NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
}
/* *************************************************************** */
