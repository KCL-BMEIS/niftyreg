#include "hip/hip_runtime.h"
/*
 *  _reg_spline_gpu.cu
 *
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#include "_reg_localTransformation_gpu.h"
#include "_reg_localTransformation_kernels.cu"

/* *************************************************************** */
void reg_spline_getDeformationField_gpu(const nifti_image *controlPointImage,
                                        const nifti_image *referenceImage,
                                        const float4 *controlPointImageCuda,
                                        float4 *deformationFieldCuda,
                                        const int *maskCuda,
                                        const size_t& activeVoxelNumber,
                                        const bool& bspline) {
    const size_t controlPointNumber = NiftiImage::calcVoxelNumber(controlPointImage, 3);
    const int3 referenceImageDim = make_int3(referenceImage->nx, referenceImage->ny, referenceImage->nz);
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
    const float3 controlPointVoxelSpacing = make_float3(controlPointImage->dx / referenceImage->dx,
                                                        controlPointImage->dy / referenceImage->dy,
                                                        controlPointImage->dz / referenceImage->dz);

    auto controlPointTexture = Cuda::CreateTextureObject(controlPointImageCuda, hipResourceTypeLinear,
                                                         controlPointNumber * sizeof(float4), hipChannelFormatKindFloat, 4);
    auto maskTexture = Cuda::CreateTextureObject(maskCuda, hipResourceTypeLinear,
                                                 activeVoxelNumber * sizeof(int), hipChannelFormatKindSigned, 1);

    if (referenceImage->nz > 1) {
        const unsigned blocks = CudaContext::GetBlockSize()->reg_spline_getDeformationField3D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)activeVoxelNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        // 8 floats of shared memory are allocated per thread
        reg_spline_getDeformationField3D<<<gridDims, blockDims, blocks * 8 * sizeof(float)>>>(deformationFieldCuda,
                                                                                              *controlPointTexture,
                                                                                              *maskTexture,
                                                                                              referenceImageDim,
                                                                                              controlPointImageDim,
                                                                                              controlPointVoxelSpacing,
                                                                                              (unsigned)activeVoxelNumber,
                                                                                              bspline);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    } else {
        const unsigned blocks = CudaContext::GetBlockSize()->reg_spline_getDeformationField2D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)activeVoxelNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        // 4 floats of shared memory are allocated per thread
        reg_spline_getDeformationField2D<<<gridDims, blockDims, blocks * 4 * sizeof(float)>>>(deformationFieldCuda,
                                                                                              *controlPointTexture,
                                                                                              *maskTexture,
                                                                                              referenceImageDim,
                                                                                              controlPointImageDim,
                                                                                              controlPointVoxelSpacing,
                                                                                              (unsigned)activeVoxelNumber,
                                                                                              bspline);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }
}
/* *************************************************************** */
float reg_spline_approxBendingEnergy_gpu(const nifti_image *controlPointImage, const float4 *controlPointImageCuda) {
    auto blockSize = CudaContext::GetBlockSize();
    const size_t controlPointNumber = NiftiImage::calcVoxelNumber(controlPointImage, 3);
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
    const size_t controlPointGridSize = controlPointNumber * sizeof(float4);
    auto controlPointTexture = Cuda::CreateTextureObject(controlPointImageCuda, hipResourceTypeLinear,
                                                         controlPointGridSize, hipChannelFormatKindFloat, 4);

    // First compute all the second derivatives
    float4 *secondDerivativeValuesCuda;
    size_t secondDerivativeValuesSize;
    if (controlPointImage->nz > 1) {
        secondDerivativeValuesSize = 6 * controlPointGridSize;
        NR_CUDA_SAFE_CALL(hipMalloc(&secondDerivativeValuesCuda, secondDerivativeValuesSize));
        const unsigned blocks = blockSize->reg_spline_getApproxSecondDerivatives3D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_getApproxSecondDerivatives3D<<<gridDims, blockDims>>>(secondDerivativeValuesCuda, *controlPointTexture,
                                                                         controlPointImageDim, (unsigned)controlPointNumber);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    } else {
        secondDerivativeValuesSize = 3 * controlPointGridSize;
        NR_CUDA_SAFE_CALL(hipMalloc(&secondDerivativeValuesCuda, secondDerivativeValuesSize));
        const unsigned blocks = blockSize->reg_spline_getApproxSecondDerivatives2D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_getApproxSecondDerivatives2D<<<gridDims, blockDims>>>(secondDerivativeValuesCuda, *controlPointTexture,
                                                                         controlPointImageDim, (unsigned)controlPointNumber);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }

    // Compute the bending energy from the second derivatives
    float *penaltyTermCuda;
    NR_CUDA_SAFE_CALL(hipMalloc(&penaltyTermCuda, controlPointNumber * sizeof(float)));
    auto secondDerivativesTexture = Cuda::CreateTextureObject(secondDerivativeValuesCuda, hipResourceTypeLinear,
                                                              secondDerivativeValuesSize, hipChannelFormatKindFloat, 4);
    if (controlPointImage->nz > 1) {
        const unsigned blocks = blockSize->reg_spline_getApproxBendingEnergy3D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_getApproxBendingEnergy3D_kernel<<<gridDims, blockDims>>>(penaltyTermCuda, *secondDerivativesTexture,
                                                                            (unsigned)controlPointNumber);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    } else {
        const unsigned blocks = blockSize->reg_spline_getApproxBendingEnergy2D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_getApproxBendingEnergy2D_kernel<<<gridDims, blockDims>>>(penaltyTermCuda, *secondDerivativesTexture,
                                                                            (unsigned)controlPointNumber);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }
    NR_CUDA_SAFE_CALL(hipFree(secondDerivativeValuesCuda));

    // Compute the mean bending energy value
    double penaltyValue = reg_sumReduction_gpu(penaltyTermCuda, controlPointNumber);
    NR_CUDA_SAFE_CALL(hipFree(penaltyTermCuda));

    return (float)(penaltyValue / (double)controlPointImage->nvox);
}
/* *************************************************************** */
void reg_spline_approxBendingEnergyGradient_gpu(const nifti_image *controlPointImage,
                                                const float4 *controlPointImageCuda,
                                                float4 *transGradientCuda,
                                                float bendingEnergyWeight) {
    auto blockSize = CudaContext::GetBlockSize();
    const size_t controlPointNumber = NiftiImage::calcVoxelNumber(controlPointImage, 3);
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
    const size_t controlPointGridSize = controlPointNumber * sizeof(float4);
    auto controlPointTexture = Cuda::CreateTextureObject(controlPointImageCuda, hipResourceTypeLinear,
                                                         controlPointGridSize, hipChannelFormatKindFloat, 4);

    // First compute all the second derivatives
    float4 *secondDerivativeValuesCuda;
    size_t secondDerivativeValuesSize;
    if (controlPointImage->nz > 1) {
        secondDerivativeValuesSize = 6 * controlPointGridSize * sizeof(float4);
        NR_CUDA_SAFE_CALL(hipMalloc(&secondDerivativeValuesCuda, secondDerivativeValuesSize));
        const unsigned blocks = blockSize->reg_spline_getApproxSecondDerivatives3D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_getApproxSecondDerivatives3D<<<gridDims, blockDims>>>(secondDerivativeValuesCuda, *controlPointTexture,
                                                                         controlPointImageDim, (unsigned)controlPointNumber);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    } else {
        secondDerivativeValuesSize = 3 * controlPointGridSize * sizeof(float4);
        NR_CUDA_SAFE_CALL(hipMalloc(&secondDerivativeValuesCuda, secondDerivativeValuesSize));
        const unsigned blocks = blockSize->reg_spline_getApproxSecondDerivatives2D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_getApproxSecondDerivatives2D<<<gridDims, blockDims>>>(secondDerivativeValuesCuda, *controlPointTexture,
                                                                         controlPointImageDim, (unsigned)controlPointNumber);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }

    // Compute the gradient
    bendingEnergyWeight *= 1.f / (float)controlPointNumber;
    auto secondDerivativesTexture = Cuda::CreateTextureObject(secondDerivativeValuesCuda, hipResourceTypeLinear,
                                                              secondDerivativeValuesSize, hipChannelFormatKindFloat, 4);
    if (controlPointImage->nz > 1) {
        const unsigned blocks = blockSize->reg_spline_getApproxBendingEnergyGradient3D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_getApproxBendingEnergyGradient3D_kernel<<<gridDims, blockDims>>>(transGradientCuda, *secondDerivativesTexture,
                                                                                    controlPointImageDim, (unsigned)controlPointNumber,
                                                                                    bendingEnergyWeight);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    } else {
        const unsigned blocks = blockSize->reg_spline_getApproxBendingEnergyGradient2D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_getApproxBendingEnergyGradient2D_kernel<<<gridDims, blockDims>>>(transGradientCuda, *secondDerivativesTexture,
                                                                                    controlPointImageDim, (unsigned)controlPointNumber,
                                                                                    bendingEnergyWeight);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }
    NR_CUDA_SAFE_CALL(hipFree(secondDerivativeValuesCuda));
}
/* *************************************************************** */
void reg_spline_ComputeApproxJacobianValues(const nifti_image *controlPointImage,
                                            const float4 *controlPointImageCuda,
                                            float *jacobianMatricesCuda,
                                            float *jacobianDetCuda) {
    auto blockSize = CudaContext::GetBlockSize();
    const size_t controlPointNumber = NiftiImage::calcVoxelNumber(controlPointImage, 3);
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
    auto controlPointTexture = Cuda::CreateTextureObject(controlPointImageCuda, hipResourceTypeLinear,
                                                         controlPointNumber * sizeof(float4), hipChannelFormatKindFloat, 4);

    // Need to reorient the Jacobian matrix using the header information - real to voxel conversion
    const mat33 reorientation = reg_mat44_to_mat33(controlPointImage->sform_code > 0 ? &controlPointImage->sto_xyz : &controlPointImage->qto_xyz);

    // The Jacobian matrix is computed for every control point
    if (controlPointImage->nz > 1) {
        const unsigned blocks = blockSize->reg_spline_getApproxJacobianValues3D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_getApproxJacobianValues3D_kernel<<<gridDims, blockDims>>>(jacobianMatricesCuda, jacobianDetCuda, *controlPointTexture,
                                                                             controlPointImageDim, (unsigned)controlPointNumber, reorientation);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    } else {
        const unsigned blocks = blockSize->reg_spline_getApproxJacobianValues2D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_getApproxJacobianValues2D_kernel<<<gridDims, blockDims>>>(jacobianMatricesCuda, jacobianDetCuda, *controlPointTexture,
                                                                             controlPointImageDim, (unsigned)controlPointNumber, reorientation);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }
}
/* *************************************************************** */
void reg_spline_ComputeJacobianValues(const nifti_image *controlPointImage,
                                      const nifti_image *referenceImage,
                                      const float4 *controlPointImageCuda,
                                      float *jacobianMatricesCuda,
                                      float *jacobianDetCuda) {
    auto blockSize = CudaContext::GetBlockSize();
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(referenceImage, 3);
    const size_t controlPointNumber = NiftiImage::calcVoxelNumber(controlPointImage, 3);
    const int3 referenceImageDim = make_int3(referenceImage->nx, referenceImage->ny, referenceImage->nz);
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
    const float3 controlPointSpacing = make_float3(controlPointImage->dx, controlPointImage->dy, controlPointImage->dz);
    auto controlPointTexture = Cuda::CreateTextureObject(controlPointImageCuda, hipResourceTypeLinear,
                                                         controlPointNumber * sizeof(float4), hipChannelFormatKindFloat, 4);

    // Need to reorient the Jacobian matrix using the header information - real to voxel conversion
    const mat33 reorientation = reg_mat44_to_mat33(controlPointImage->sform_code > 0 ? &controlPointImage->sto_xyz : &controlPointImage->qto_xyz);

    // The Jacobian matrix is computed for every voxel
    if (controlPointImage->nz > 1) {
        const unsigned blocks = blockSize->reg_spline_getJacobianValues3D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)voxelNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        // 8 floats of shared memory are allocated per thread
        const unsigned sharedMemSize = blocks * 8 * sizeof(float);
        reg_spline_getJacobianValues3D_kernel<<<gridDims, blockDims, sharedMemSize>>>(jacobianMatricesCuda, jacobianDetCuda, *controlPointTexture,
                                                                                      controlPointImageDim, controlPointSpacing, referenceImageDim,
                                                                                      (unsigned)voxelNumber, reorientation);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    } else {
        const unsigned blocks = blockSize->reg_spline_getJacobianValues2D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)voxelNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_getJacobianValues2D_kernel<<<gridDims, blockDims>>>(jacobianMatricesCuda, jacobianDetCuda, *controlPointTexture,
                                                                       controlPointImageDim, controlPointSpacing, referenceImageDim,
                                                                       (unsigned)voxelNumber, reorientation);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }
}
/* *************************************************************** */
double reg_spline_getJacobianPenaltyTerm_gpu(const nifti_image *referenceImage,
                                             const nifti_image *controlPointImage,
                                             const float4 *controlPointImageCuda,
                                             const bool& approx) {
    // The Jacobian matrices and determinants are computed
    float *jacobianMatricesCuda, *jacobianDetCuda;
    size_t jacNumber; double jacSum;
    if (approx) {
        jacNumber = NiftiImage::calcVoxelNumber(controlPointImage, 3);
        jacSum = (controlPointImage->nx - 2) * (controlPointImage->ny - 2);
        if (controlPointImage->nz > 1)
            jacSum *= controlPointImage->nz - 2;
        // Allocate 3x3 matrices for 3D, and 2x2 matrices for 2D
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianMatricesCuda, (controlPointImage->nz > 1 ? 9 : 4) * jacNumber * sizeof(float)));
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianDetCuda, jacNumber * sizeof(float)));
        reg_spline_ComputeApproxJacobianValues(controlPointImage, controlPointImageCuda, jacobianMatricesCuda, jacobianDetCuda);
    } else {
        jacNumber = NiftiImage::calcVoxelNumber(referenceImage, 3);
        jacSum = static_cast<double>(jacNumber);
        // Allocate 3x3 matrices for 3D, and 2x2 matrices for 2D
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianMatricesCuda, (controlPointImage->nz > 1 ? 9 : 4) * jacNumber * sizeof(float)));
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianDetCuda, jacNumber * sizeof(float)));
        reg_spline_ComputeJacobianValues(controlPointImage, referenceImage, controlPointImageCuda, jacobianMatricesCuda, jacobianDetCuda);
    }
    NR_CUDA_SAFE_CALL(hipFree(jacobianMatricesCuda));

    // The Jacobian determinant are squared and logged (might not be english but will do)
    const unsigned blocks = CudaContext::GetBlockSize()->reg_spline_logSquaredValues;
    const unsigned grids = (unsigned)ceilf(sqrtf((float)jacNumber / (float)blocks));
    const dim3 gridDims(grids, grids, 1);
    const dim3 blockDims(blocks, 1, 1);
    reg_spline_logSquaredValues_kernel<<<gridDims, blockDims>>>(jacobianDetCuda, (unsigned)jacNumber);
    NR_CUDA_CHECK_KERNEL(gridDims, blockDims);

    // Perform the reduction
    const double penaltyTermValue = reg_sumReduction_gpu(jacobianDetCuda, jacNumber);
    NR_CUDA_SAFE_CALL(hipFree(jacobianDetCuda));
    return penaltyTermValue / jacSum;
}
/* *************************************************************** */
void reg_spline_getJacobianPenaltyTermGradient_gpu(const nifti_image *referenceImage,
                                                   const nifti_image *controlPointImage,
                                                   const float4 *controlPointImageCuda,
                                                   float4 *transGradientCuda,
                                                   const float& jacobianWeight,
                                                   const bool& approx) {
    auto blockSize = CudaContext::GetBlockSize();

    // The Jacobian matrices and determinants are computed
    float *jacobianMatricesCuda, *jacobianDetCuda;
    size_t jacNumber;
    if (approx) {
        jacNumber = NiftiImage::calcVoxelNumber(controlPointImage, 3);
        // Allocate 3x3 matrices for 3D, and 2x2 matrices for 2D
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianMatricesCuda, (controlPointImage->nz > 1 ? 9 : 4) * jacNumber * sizeof(float)));
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianDetCuda, jacNumber * sizeof(float)));
        reg_spline_ComputeApproxJacobianValues(controlPointImage, controlPointImageCuda, jacobianMatricesCuda, jacobianDetCuda);
    } else {
        jacNumber = NiftiImage::calcVoxelNumber(referenceImage, 3);
        // Allocate 3x3 matrices for 3D, and 2x2 matrices for 2D
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianMatricesCuda, (controlPointImage->nz > 1 ? 9 : 4) * jacNumber * sizeof(float)));
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianDetCuda, jacNumber * sizeof(float)));
        reg_spline_ComputeJacobianValues(controlPointImage, referenceImage, controlPointImageCuda, jacobianMatricesCuda, jacobianDetCuda);
    }

    // Need to disorient the Jacobian matrix using the header information - voxel to real conversion
    const mat33 reorientation = reg_mat44_to_mat33(controlPointImage->sform_code > 0 ? &controlPointImage->sto_ijk : &controlPointImage->qto_ijk);

    const size_t controlPointNumber = NiftiImage::calcVoxelNumber(controlPointImage, 3);
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
    const float3 controlPointSpacing = make_float3(controlPointImage->dx, controlPointImage->dy, controlPointImage->dz);
    const float3 weight = make_float3(referenceImage->dx * jacobianWeight / ((float)jacNumber * controlPointImage->dx),
                                      referenceImage->dy * jacobianWeight / ((float)jacNumber * controlPointImage->dy),
                                      referenceImage->dz * jacobianWeight / ((float)jacNumber * controlPointImage->dz));
    auto jacobianDeterminantTexture = Cuda::CreateTextureObject(jacobianDetCuda, hipResourceTypeLinear, jacNumber * sizeof(float),
                                                                hipChannelFormatKindFloat, 1);
    auto jacobianMatricesTexture = Cuda::CreateTextureObject(jacobianMatricesCuda, hipResourceTypeLinear,
                                                             (controlPointImage->nz > 1 ? 9 : 4) * jacNumber * sizeof(float),
                                                             hipChannelFormatKindFloat, 1);
    if (approx) {
        if (controlPointImage->nz > 1) {
            const unsigned blocks = blockSize->reg_spline_computeApproxJacGradient3D;
            const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
            const dim3 gridDims(grids, grids, 1);
            const dim3 blockDims(blocks, 1, 1);
            reg_spline_computeApproxJacGradient3D_kernel<<<gridDims, blockDims>>>(transGradientCuda, *jacobianDeterminantTexture,
                                                                                  *jacobianMatricesTexture, controlPointImageDim,
                                                                                  (unsigned)controlPointNumber, reorientation, weight);
            NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
        } else {
            const unsigned blocks = blockSize->reg_spline_computeApproxJacGradient2D;
            const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
            const dim3 gridDims(grids, grids, 1);
            const dim3 blockDims(blocks, 1, 1);
            reg_spline_computeApproxJacGradient2D_kernel<<<gridDims, blockDims>>>(transGradientCuda, *jacobianDeterminantTexture,
                                                                                  *jacobianMatricesTexture, controlPointImageDim,
                                                                                  (unsigned)controlPointNumber, reorientation, weight);
            NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
        }
    } else {
        const int3 referenceImageDim = make_int3(referenceImage->nx, referenceImage->ny, referenceImage->nz);
        const float3 controlPointVoxelSpacing = make_float3(controlPointImage->dx / referenceImage->dx,
                                                            controlPointImage->dy / referenceImage->dy,
                                                            controlPointImage->dz / referenceImage->dz);
        if (controlPointImage->nz > 1) {
            const unsigned blocks = blockSize->reg_spline_computeJacGradient3D;
            const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
            const dim3 gridDims(grids, grids, 1);
            const dim3 blockDims(blocks, 1, 1);
            reg_spline_computeJacGradient3D_kernel<<<gridDims, blockDims>>>(transGradientCuda, *jacobianDeterminantTexture,
                                                                            *jacobianMatricesTexture, controlPointImageDim,
                                                                            controlPointVoxelSpacing, (unsigned)controlPointNumber,
                                                                            referenceImageDim, reorientation, weight);
            NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
        } else {
            const unsigned blocks = blockSize->reg_spline_computeJacGradient2D;
            const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
            const dim3 gridDims(grids, grids, 1);
            const dim3 blockDims(blocks, 1, 1);
            reg_spline_computeJacGradient2D_kernel<<<gridDims, blockDims>>>(transGradientCuda, *jacobianDeterminantTexture,
                                                                            *jacobianMatricesTexture, controlPointImageDim,
                                                                            controlPointVoxelSpacing, (unsigned)controlPointNumber,
                                                                            referenceImageDim, reorientation, weight);
            NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
        }
    }
    NR_CUDA_SAFE_CALL(hipFree(jacobianDetCuda));
    NR_CUDA_SAFE_CALL(hipFree(jacobianMatricesCuda));
}
/* *************************************************************** */
double reg_spline_correctFolding_gpu(const nifti_image *referenceImage,
                                     const nifti_image *controlPointImage,
                                     float4 *controlPointImageCuda,
                                     const bool& approx) {
    auto blockSize = CudaContext::GetBlockSize();

    // The Jacobian matrices and determinants are computed
    float *jacobianMatricesCuda, *jacobianDetCuda;
    size_t jacobianDetSize, jacobianMatricesSize;
    size_t jacNumber; double jacSum;
    if (approx) {
        jacNumber = NiftiImage::calcVoxelNumber(controlPointImage, 3);
        jacSum = (controlPointImage->nx - 2) * (controlPointImage->ny - 2) * (controlPointImage->nz - 2);
        jacobianDetSize = jacNumber * sizeof(float);
        jacobianMatricesSize = 9 * jacobianDetSize;
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianMatricesCuda, jacobianMatricesSize));
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianDetCuda, jacobianDetSize));
        reg_spline_ComputeApproxJacobianValues(controlPointImage, controlPointImageCuda, jacobianMatricesCuda, jacobianDetCuda);
    } else {
        jacNumber = NiftiImage::calcVoxelNumber(referenceImage, 3);
        jacSum = static_cast<double>(jacNumber);
        jacobianDetSize = jacNumber * sizeof(float);
        jacobianMatricesSize = 9 * jacobianDetSize;
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianMatricesCuda, jacobianMatricesSize));
        NR_CUDA_SAFE_CALL(hipMalloc(&jacobianDetCuda, jacobianDetSize));
        reg_spline_ComputeJacobianValues(controlPointImage, referenceImage, controlPointImageCuda, jacobianMatricesCuda, jacobianDetCuda);
    }

    // Check if the Jacobian determinant average
    float *jacobianDet2Cuda;
    NR_CUDA_SAFE_CALL(hipMalloc(&jacobianDet2Cuda, jacobianDetSize));
    NR_CUDA_SAFE_CALL(hipMemcpy(jacobianDet2Cuda, jacobianDetCuda, jacobianDetSize, hipMemcpyDeviceToDevice));
    const unsigned blocks = blockSize->reg_spline_logSquaredValues;
    const unsigned grids = (unsigned)ceilf(sqrtf((float)jacNumber / (float)blocks));
    const dim3 gridDims(grids, grids, 1);
    const dim3 blockDims(blocks, 1, 1);
    reg_spline_logSquaredValues_kernel<<<gridDims, blockDims>>>(jacobianDet2Cuda, (unsigned)jacNumber);
    NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    float *jacobianDet;
    NR_CUDA_SAFE_CALL(hipHostMalloc(&jacobianDet, jacobianDetSize));
    NR_CUDA_SAFE_CALL(hipMemcpy(jacobianDet, jacobianDet2Cuda, jacobianDetSize, hipMemcpyDeviceToHost));
    NR_CUDA_SAFE_CALL(hipFree(jacobianDet2Cuda));
    double penaltyTermValue = 0;
    for (int i = 0; i < jacNumber; ++i) penaltyTermValue += jacobianDet[i];
    NR_CUDA_SAFE_CALL(hipHostFree(jacobianDet));
    penaltyTermValue /= jacSum;
    if (penaltyTermValue == penaltyTermValue) {
        NR_CUDA_SAFE_CALL(hipFree(jacobianDetCuda));
        NR_CUDA_SAFE_CALL(hipFree(jacobianMatricesCuda));
        return penaltyTermValue;
    }

    // Need to disorient the Jacobian matrix using the header information - voxel to real conversion
    const mat33 reorientation = reg_mat44_to_mat33(controlPointImage->sform_code > 0 ? &controlPointImage->sto_ijk : &controlPointImage->qto_ijk);

    const size_t controlPointNumber = NiftiImage::calcVoxelNumber(controlPointImage, 3);
    const int3 controlPointImageDim = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
    const float3 controlPointSpacing = make_float3(controlPointImage->dx, controlPointImage->dy, controlPointImage->dz);
    auto jacobianDeterminantTexture = Cuda::CreateTextureObject(jacobianDetCuda, hipResourceTypeLinear, jacobianDetSize,
                                                                hipChannelFormatKindFloat, 1);
    auto jacobianMatricesTexture = Cuda::CreateTextureObject(jacobianMatricesCuda, hipResourceTypeLinear, jacobianMatricesSize,
                                                             hipChannelFormatKindFloat, 1);
    if (approx) {
        const unsigned blocks = blockSize->reg_spline_approxCorrectFolding3D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_approxCorrectFolding3D_kernel<<<gridDims, blockDims>>>(controlPointImageCuda, *jacobianDeterminantTexture,
                                                                          *jacobianMatricesTexture, controlPointImageDim,
                                                                          controlPointSpacing, (unsigned)controlPointNumber, reorientation);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    } else {
        const int3 referenceImageDim = make_int3(referenceImage->nx, referenceImage->ny, referenceImage->nz);
        const float3 controlPointVoxelSpacing = make_float3(controlPointImage->dx / referenceImage->dx,
                                                            controlPointImage->dy / referenceImage->dy,
                                                            controlPointImage->dz / referenceImage->dz);
        const unsigned blocks = blockSize->reg_spline_correctFolding3D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)controlPointNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_spline_correctFolding3D_kernel<<<gridDims, blockDims>>>(controlPointImageCuda, *jacobianDeterminantTexture,
                                                                    *jacobianMatricesTexture, controlPointImageDim, controlPointSpacing,
                                                                    controlPointVoxelSpacing, (unsigned)controlPointNumber,
                                                                    referenceImageDim, reorientation);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }
    NR_CUDA_SAFE_CALL(hipFree(jacobianDetCuda));
    NR_CUDA_SAFE_CALL(hipFree(jacobianMatricesCuda));
    return std::numeric_limits<double>::quiet_NaN();
}
/* *************************************************************** */
void reg_getDeformationFromDisplacement_gpu(const nifti_image *image, float4 *imageCuda, const bool& reverse = false) {
    // Bind the qform or sform
    const mat44 affineMatrix = image->sform_code > 0 ? image->sto_xyz : image->qto_xyz;
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(image, 3);
    const int3 imageDim = make_int3(image->nx, image->ny, image->nz);

    const unsigned blocks = CudaContext::GetBlockSize()->reg_getDeformationFromDisplacement;
    const unsigned grids = (unsigned)ceilf(sqrtf((float)voxelNumber / (float)blocks));
    const dim3 gridDims(grids, grids, 1);
    const dim3 blockDims(blocks, 1, 1);
    reg_getDeformationFromDisplacement3D_kernel<<<gridDims, blockDims>>>(imageCuda, imageDim, (unsigned)voxelNumber, affineMatrix, reverse);
    NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
}
/* *************************************************************** */
void reg_getDisplacementFromDeformation_gpu(const nifti_image *image, float4 *imageCuda) {
    reg_getDeformationFromDisplacement_gpu(image, imageCuda, true);
}
/* *************************************************************** */
void reg_getDeformationFieldFromVelocityGrid_gpu(const nifti_image *controlPointImage,
                                                 const nifti_image *deformationField,
                                                 const float4 *controlPointImageCuda,
                                                 float4 *deformationFieldCuda) {
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(deformationField, 3);

    // Create a mask array where no voxel are excluded
    int *maskCuda = nullptr;
    NR_CUDA_SAFE_CALL(hipMalloc(&maskCuda, voxelNumber * sizeof(int)));
    reg_fillMaskArray_gpu(maskCuda, voxelNumber);

    // Define some variables for the deformation fields
    float4 *tempDefCuda = nullptr;
    NR_CUDA_SAFE_CALL(hipMalloc(&tempDefCuda, voxelNumber * sizeof(float4)));

    // The deformation field is computed
    reg_spline_getDeformationField_gpu(controlPointImage, deformationField, controlPointImageCuda,
                                       deformationFieldCuda, maskCuda, voxelNumber, true); // non-interpolant spline is used

    // The deformation field is converted into a displacement field
    reg_getDisplacementFromDeformation_gpu(deformationField, deformationFieldCuda);

    // Scaling of the deformation field
    const unsigned squaringNumber = (unsigned)fabs(controlPointImage->intent_p1);
    const float scalingValue = pow(2.f, (float)squaringNumber);
    // Backward/forward deformation field is scaled down
    reg_multiplyValue_gpu((int)voxelNumber, deformationFieldCuda, (controlPointImage->intent_p1 < 0  ? -1.f : 1.f) / scalingValue);

    // The displacement field is converted back into a deformation field
    reg_getDeformationFromDisplacement_gpu(deformationField, deformationFieldCuda);

    // The deformation field is squared
    for (unsigned i = 0; i < squaringNumber; ++i) {
        // The deformation field arrays are updated
        NR_CUDA_SAFE_CALL(hipMemcpy(tempDefCuda, deformationFieldCuda, voxelNumber * sizeof(float4), hipMemcpyDeviceToDevice));

        // The deformation fields are composed
        reg_defField_compose_gpu(deformationField, tempDefCuda, deformationFieldCuda, voxelNumber);
    }

    NR_CUDA_SAFE_CALL(hipFree(tempDefCuda));
    NR_CUDA_SAFE_CALL(hipFree(maskCuda));
}
/* *************************************************************** */
void reg_defField_compose_gpu(const nifti_image *deformationField,
                              const float4 *deformationFieldCuda,
                              float4 *deformationFieldCudaOut,
                              const size_t& activeVoxelNumber) {
    auto blockSize = CudaContext::GetBlockSize();
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(deformationField, 3);
    const int3 referenceImageDim = make_int3(deformationField->nx, deformationField->ny, deformationField->nz);
    const mat44 affineMatrixB = deformationField->sform_code > 0 ? deformationField->sto_ijk : deformationField->qto_ijk;
    const mat44 affineMatrixC = deformationField->sform_code > 0 ? deformationField->sto_xyz : deformationField->qto_xyz;
    auto deformationFieldTexture = Cuda::CreateTextureObject(deformationFieldCuda, hipResourceTypeLinear,
                                                             activeVoxelNumber * sizeof(float4), hipChannelFormatKindFloat, 4);

    if (deformationField->nz > 1) {
        const unsigned blocks = blockSize->reg_defField_compose3D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)voxelNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_defField_compose3D_kernel<<<gridDims, blockDims>>>(deformationFieldCudaOut, *deformationFieldTexture, referenceImageDim,
                                                               (unsigned)voxelNumber, affineMatrixB, affineMatrixC);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    } else {
        const unsigned blocks = blockSize->reg_defField_compose2D;
        const unsigned grids = (unsigned)ceilf(sqrtf((float)voxelNumber / (float)blocks));
        const dim3 gridDims(grids, grids, 1);
        const dim3 blockDims(blocks, 1, 1);
        reg_defField_compose2D_kernel<<<gridDims, blockDims>>>(deformationFieldCudaOut, *deformationFieldTexture, referenceImageDim,
                                                               (unsigned)voxelNumber, affineMatrixB, affineMatrixC);
        NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
    }
}
/* *************************************************************** */
void reg_defField_getJacobianMatrix_gpu(const nifti_image *deformationField,
                                        const float4 *deformationFieldCuda,
                                        float *jacobianMatricesCuda) {
    const int3 referenceImageDim = make_int3(deformationField->nx, deformationField->ny, deformationField->nz);
    const size_t voxelNumber = NiftiImage::calcVoxelNumber(deformationField, 3);
    const mat33 reorientation = reg_mat44_to_mat33(deformationField->sform_code > 0 ? &deformationField->sto_xyz : &deformationField->qto_xyz);
    auto deformationFieldTexture = Cuda::CreateTextureObject(deformationFieldCuda, hipResourceTypeLinear,
                                                             voxelNumber * sizeof(float4), hipChannelFormatKindFloat, 4);

    const unsigned blocks = CudaContext::GetBlockSize()->reg_defField_getJacobianMatrix;
    const unsigned grids = (unsigned)ceilf(sqrtf((float)voxelNumber / (float)blocks));
    const dim3 gridDims(grids, grids, 1);
    const dim3 blockDims(blocks, 1, 1);
    reg_defField_getJacobianMatrix3D_kernel<<<gridDims, blockDims>>>(jacobianMatricesCuda, *deformationFieldTexture, referenceImageDim,
                                                                     (unsigned)voxelNumber, reorientation);
    NR_CUDA_CHECK_KERNEL(gridDims, blockDims);
}
/* *************************************************************** */
