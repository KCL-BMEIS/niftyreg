#include "hip/hip_runtime.h"
/*
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 *  See the LICENSE.txt file in the nifty_reg root folder
 */

#pragma once

/* *************************************************************** */
__device__ __inline__ float2 operator*(float a, float2 b) {
    return { a * b.x, a * b.y };
}
__device__ __inline__ float3 operator*(float a, float3 b) {
    return { a * b.x, a * b.y, a * b.z };
}
__device__ __inline__ float3 operator*(float3 a, float3 b) {
    return { a.x * b.x, a.y * b.y, a.z * b.z };
}
__device__ __inline__ float4 operator*(float4 a, float4 b) {
    return { a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w };
}
__device__ __inline__ float4 operator*(float a, float4 b) {
    return { a * b.x, a * b.y, a * b.z, 0.0f };
}
/* *************************************************************** */
__device__ __inline__ float2 operator/(float2 a, float2 b) {
    return { a.x / b.x, a.y / b.y };
}
__device__ __inline__ float3 operator/(float3 a, float b) {
    return { a.x / b, a.y / b, a.z / b };
}
__device__ __inline__ float3 operator/(float3 a, float3 b) {
    return { a.x / b.x, a.y / b.y, a.z / b.z };
}
/* *************************************************************** */
__device__ __inline__ float2 operator+(float2 a, float2 b) {
    return { a.x + b.x, a.y + b.y };
}
__device__ __inline__ float4 operator+(float4 a, float4 b) {
    return { a.x + b.x, a.y + b.y, a.z + b.z, 0.0f };
}
__device__ __inline__ float3 operator+(float3 a, float3 b) {
    return { a.x + b.x, a.y + b.y, a.z + b.z };
}
/* *************************************************************** */
__device__ __inline__ float3 operator-(float3 a, float3 b) {
    return { a.x - b.x, a.y - b.y, a.z - b.z };
}
__device__ __inline__ float4 operator-(float4 a, float4 b) {
    return { a.x - b.x, a.y - b.y, a.z - b.z, 0.f };
}
/* *************************************************************** */
__device__ __inline__ void reg_mat33_mul_cuda(const mat33& mat, const float (&in)[3], const float& weight, float (&out)[3], const bool& is3d) {
    out[0] = weight * (mat.m[0][0] * in[0] + mat.m[0][1] * in[1] + mat.m[0][2] * in[2]);
    out[1] = weight * (mat.m[1][0] * in[0] + mat.m[1][1] * in[1] + mat.m[1][2] * in[2]);
    out[2] = is3d ? weight * (mat.m[2][0] * in[0] + mat.m[2][1] * in[1] + mat.m[2][2] * in[2]) : 0;
}
/* *************************************************************** */
__device__ __inline__ void reg_mat44_mul_cuda(const mat44& mat, const float (&in)[3], float (&out)[3], const bool& is3d) {
    out[0] = mat.m[0][0] * in[0] + mat.m[0][1] * in[1] + mat.m[0][2] * in[2] + mat.m[0][3];
    out[1] = mat.m[1][0] * in[0] + mat.m[1][1] * in[1] + mat.m[1][2] * in[2] + mat.m[1][3];
    out[2] = is3d ? mat.m[2][0] * in[0] + mat.m[2][1] * in[1] + mat.m[2][2] * in[2] + mat.m[2][3] : 0;
}
/* *************************************************************** */
__device__ __inline__ void reg_div_cuda(const int num, const int denom, int& quot, int& rem) {
    // This will be optimised by the compiler into a single div instruction
    quot = num / denom;
    rem = num % denom;
}
/* *************************************************************** */
