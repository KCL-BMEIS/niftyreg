#include "hip/hip_runtime.h"
#include "CudaKernelConvolution.hpp"

/* *************************************************************** */
void NiftyReg::Cuda::KernelConvolution(const nifti_image *image,
                                       float4 *imageCuda,
                                       const float *sigma,
                                       const int kernelType,
                                       const bool *timePoints,
                                       const bool *axis) {
    if (image->nx > 2048 || image->ny > 2048 || image->nz > 2048)
        NR_FATAL_ERROR("This function does not support images with dimensions larger than 2048");

    bool axisToSmooth[3];
    if (axis == nullptr) {
        // All axis are smoothed by default
        axisToSmooth[0] = axisToSmooth[1] = axisToSmooth[2] = true;
    } else for (int i = 0; i < 3; i++) axisToSmooth[i] = axis[i];

    const auto activeTimePointCount = std::min(image->nt * image->nu, 4);
    bool activeTimePoints[4]{}; // 4 is the maximum number of time points
    if (timePoints == nullptr) {
        // All time points are considered as active
        for (auto i = 0; i < activeTimePointCount; i++) activeTimePoints[i] = true;
    } else for (auto i = 0; i < activeTimePointCount; i++) activeTimePoints[i] = timePoints[i];

    const size_t voxelNumber = NiftiImage::calcVoxelNumber(image, 3);
    const int3 imageDims = make_int3(image->nx, image->ny, image->nz);

    thrust::device_vector<float> densityCuda(voxelNumber);
    thrust::device_vector<bool> nanImageCuda(voxelNumber);
    thrust::device_vector<float> bufferIntensityCuda(voxelNumber);
    thrust::device_vector<float> bufferDensityCuda(voxelNumber);
    float *densityCudaPtr = densityCuda.data().get();
    bool *nanImageCudaPtr = nanImageCuda.data().get();
    float *bufferIntensityCudaPtr = bufferIntensityCuda.data().get();
    float *bufferDensityCudaPtr = bufferDensityCuda.data().get();

    for (int t = 0; t < activeTimePointCount; t++) {
        if (!activeTimePoints[t]) continue;

        thrust::for_each_n(thrust::device, thrust::make_counting_iterator<size_t>(0), voxelNumber, [=]__device__(const size_t index) {
            float& intensityVal = reinterpret_cast<float*>(&imageCuda[index])[t];
            float& densityVal = densityCudaPtr[index];
            bool& nanImageVal = nanImageCudaPtr[index];
            densityVal = intensityVal == intensityVal ? 1.f : 0;
            nanImageVal = !static_cast<bool>(densityVal);
            if (nanImageVal) intensityVal = 0;
        });

        // Loop over the x, y and z dimensions
        for (int n = 0; n < 3; n++) {
            if (!axisToSmooth[n] || image->dim[n] <= 1) continue;

            double temp;
            if (sigma[t] > 0) temp = sigma[t] / image->pixdim[n + 1]; // mm to voxel
            else temp = fabs(sigma[t]); // voxel-based if negative value
            int radius = 0;
            // Define the kernel size
            if (kernelType == MEAN_KERNEL || kernelType == LINEAR_KERNEL) {
                // Mean or linear filtering
                radius = static_cast<int>(temp);
            } else if (kernelType == GAUSSIAN_KERNEL) {
                // Gaussian kernel
                radius = static_cast<int>(temp * 3.0);
            } else if (kernelType == CUBIC_SPLINE_KERNEL) {
                // Spline kernel
                radius = static_cast<int>(temp * 2.0);
            } else {
                NR_FATAL_ERROR("Unknown kernel type");
            }
            if (radius <= 0) continue;

            // Allocate the kernel
            vector<float> kernel(2 * radius + 1);
            double kernelSum = 0;
            // Fill the kernel
            if (kernelType == CUBIC_SPLINE_KERNEL) {
                // Compute the Cubic Spline kernel
                for (int i = -radius; i <= radius; i++) {
                    // temp contains the kernel node spacing
                    double relative = fabs(i / temp);
                    if (relative < 1.0)
                        kernel[i + radius] = static_cast<float>(2.0 / 3.0 - Square(relative) + 0.5 * Cube(relative));
                    else if (relative < 2.0)
                        kernel[i + radius] = static_cast<float>(-Cube(relative - 2.0) / 6.0);
                    else kernel[i + radius] = 0;
                    kernelSum += kernel[i + radius];
                }
            } else if (kernelType == GAUSSIAN_KERNEL) {
                // Compute the Gaussian kernel
                for (int i = -radius; i <= radius; i++) {
                    // 2.506... = sqrt(2*pi)
                    // temp contains the sigma in voxel
                    kernel[i + radius] = static_cast<float>(exp(-Square(i) / (2.0 * Square(temp))) / (temp * 2.506628274631));
                    kernelSum += kernel[i + radius];
                }
            } else if (kernelType == LINEAR_KERNEL) {
                // Compute the linear kernel
                for (int i = -radius; i <= radius; i++) {
                    kernel[i + radius] = 1.f - fabs(i / static_cast<float>(radius));
                    kernelSum += kernel[i + radius];
                }
            } else if (kernelType == MEAN_KERNEL && imageDims.z == 1) {
                // Compute the mean kernel
                for (int i = -radius; i <= radius; i++) {
                    kernel[i + radius] = 1.f;
                    kernelSum += kernel[i + radius];
                }
            }
            // No kernel is required for the mean filtering
            // No need for kernel normalisation as this is handled by the density function
            NR_DEBUG("Convolution type[" << kernelType << "] dim[" << n << "] tp[" << t << "] radius[" << radius << "] kernelSum[" << kernelSum << "]");

            int planeCount, lineOffset;
            switch (n) {
            case 0:
                planeCount = imageDims.y * imageDims.z;
                lineOffset = 1;
                break;
            case 1:
                planeCount = imageDims.x * imageDims.z;
                lineOffset = imageDims.x;
                break;
            case 2:
                planeCount = imageDims.x * imageDims.y;
                lineOffset = planeCount;
                break;
            }

            thrust::device_vector<float> kernelCuda(kernel.begin(), kernel.end());
            float *kernelCudaPtr = kernelCuda.data().get();
            const int imageDim = reinterpret_cast<const int*>(&imageDims)[n];

            // Loop over the different voxel
            thrust::for_each_n(thrust::device, thrust::make_counting_iterator(0), planeCount, [=]__device__(const int planeIndex) {
                int realIndex = 0;
                switch (n) {
                case 0:
                    realIndex = planeIndex * imageDims.x;
                    break;
                case 1:
                    realIndex = (planeIndex / imageDims.x) * imageDims.x * imageDims.y + planeIndex % imageDims.x;
                    break;
                case 2:
                    realIndex = planeIndex;
                    break;
                }
                // Fetch the current line into a stack buffer
                float *bufferIntensityPtr = &bufferIntensityCudaPtr[planeIndex * imageDim];
                float *bufferDensityPtr = &bufferDensityCudaPtr[planeIndex * imageDim];
                float4 *currentIntensityPtr = &imageCuda[realIndex];
                float *currentDensityPtr = &densityCudaPtr[realIndex];
                for (int lineIndex = 0; lineIndex < imageDim; ++lineIndex) {
                    bufferIntensityPtr[lineIndex] = reinterpret_cast<float*>(currentIntensityPtr)[t];
                    bufferDensityPtr[lineIndex] = *currentDensityPtr;
                    currentIntensityPtr += lineOffset;
                    currentDensityPtr += lineOffset;
                }
                if (kernelSum > 0) {
                    // Perform the kernel convolution along 1 line
                    for (int lineIndex = 0; lineIndex < imageDim; ++lineIndex) {
                        // Define the kernel boundaries
                        int shiftPre = lineIndex - radius;
                        int shiftPst = lineIndex + radius + 1;
                        float *kernelPtr;
                        if (shiftPre < 0) {
                            kernelPtr = &kernelCudaPtr[-shiftPre];
                            shiftPre = 0;
                        } else kernelPtr = kernelCudaPtr;
                        if (shiftPst > imageDim) shiftPst = imageDim;
                        // Set the current values to zero
                        // Increment the current value by performing the weighted sum
                        double intensitySum = 0, densitySum = 0;
                        for (int k = shiftPre; k < shiftPst; ++k) {
                            float& kernelValue = *kernelPtr++;
                            intensitySum += kernelValue * bufferIntensityPtr[k];
                            densitySum += kernelValue * bufferDensityPtr[k];
                        }
                        // Store the computed value in place
                        reinterpret_cast<float*>(&imageCuda[realIndex])[t] = static_cast<float>(intensitySum);
                        densityCudaPtr[realIndex] = static_cast<float>(densitySum);
                        realIndex += lineOffset;
                    } // line convolution
                } else { // kernelSum <= 0
                    for (int lineIndex = 1; lineIndex < imageDim; ++lineIndex) {
                        bufferIntensityPtr[lineIndex] += bufferIntensityPtr[lineIndex - 1];
                        bufferDensityPtr[lineIndex] += bufferDensityPtr[lineIndex - 1];
                    }
                    int shiftPre = -radius - 1;
                    int shiftPst = radius;
                    for (int lineIndex = 0; lineIndex < imageDim; ++lineIndex, ++shiftPre, ++shiftPst) {
                        float bufferIntensityCur, bufferDensityCur;
                        if (shiftPre > -1) {
                            if (shiftPst < imageDim) {
                                bufferIntensityCur = bufferIntensityPtr[shiftPre] - bufferIntensityPtr[shiftPst];
                                bufferDensityCur = bufferDensityPtr[shiftPre] - bufferDensityPtr[shiftPst];
                            } else {
                                bufferIntensityCur = bufferIntensityPtr[shiftPre] - bufferIntensityPtr[imageDim - 1];
                                bufferDensityCur = bufferDensityPtr[shiftPre] - bufferDensityPtr[imageDim - 1];
                            }
                        } else {
                            if (shiftPst < imageDim) {
                                bufferIntensityCur = -bufferIntensityPtr[shiftPst];
                                bufferDensityCur = -bufferDensityPtr[shiftPst];
                            } else {
                                bufferIntensityCur = 0;
                                bufferDensityCur = 0;
                            }
                        }
                        reinterpret_cast<float*>(&imageCuda[realIndex])[t] = bufferIntensityCur;
                        densityCudaPtr[realIndex] = bufferDensityCur;
                        realIndex += lineOffset;
                    } // line convolution of mean filter
                } // No kernel computation
            }); // pixel in starting plane
        } // axes

        // Normalise per time point
        thrust::for_each_n(thrust::device, thrust::make_counting_iterator<size_t>(0), voxelNumber, [=]__device__(const size_t index) {
            float& intensityVal = reinterpret_cast<float*>(&imageCuda[index])[t];
            const float& densityVal = densityCudaPtr[index];
            const bool& nanImageVal = nanImageCudaPtr[index];
            intensityVal = nanImageVal ? std::numeric_limits<float>::quiet_NaN() : intensityVal / densityVal;
        });
    } // check if the time point is active
}
/* *************************************************************** */
