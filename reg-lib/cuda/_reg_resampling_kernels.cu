#include "hip/hip_runtime.h"
/*
 *  _reg_resampling_kernels.cu
 *
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

/* *************************************************************** */
__inline__ __device__ void InterpLinearKernel(float relative, float (&basis)[2]) {
    if (relative < 0)
        relative = 0;  // reg_rounding error
    basis[1] = relative;
    basis[0] = 1.f - relative;
}
/* *************************************************************** */
__global__ void reg_resampleImage2D_kernel(float *resultArray,
                                           hipTextureObject_t floatingTexture,
                                           hipTextureObject_t deformationFieldTexture,
                                           hipTextureObject_t maskTexture,
                                           const mat44 floatingMatrix,
                                           const int3 floatingDim,
                                           const unsigned activeVoxelNumber,
                                           const float paddingValue) {
    const unsigned tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (tid < activeVoxelNumber) {
        // Get the real world deformation in the floating space
        const int tid2 = tex1Dfetch<int>(maskTexture, tid);
        float4 realDeformation = tex1Dfetch<float4>(deformationFieldTexture, tid);

        // Get the voxel-based deformation in the floating space
        float2 voxelDeformation;
        voxelDeformation.x = (floatingMatrix.m[0][0] * realDeformation.x +
                              floatingMatrix.m[0][1] * realDeformation.y +
                              floatingMatrix.m[0][3]);
        voxelDeformation.y = (floatingMatrix.m[1][0] * realDeformation.x +
                              floatingMatrix.m[1][1] * realDeformation.y +
                              floatingMatrix.m[1][3]);

        // Compute the linear interpolation
        const int2 previous = { Floor(voxelDeformation.x), Floor(voxelDeformation.y) };
        const float2 relative = { voxelDeformation.x - previous.x, voxelDeformation.y - previous.y };
        float xBasis[2], yBasis[2];
        InterpLinearKernel(relative.x, xBasis);
        InterpLinearKernel(relative.y, yBasis);

        float intensity = 0;
        for (short b = 0; b < 2; b++) {
            const int y = previous.y + b;
            float xTempNewValue = 0;
            for (short a = 0; a < 2; a++) {
                const int x = previous.x + a;
                if (-1 < x && x < floatingDim.x && -1 < y && y < floatingDim.y) {
                    xTempNewValue += tex3D<float>(floatingTexture, x, y, 0) * xBasis[a];
                } else {
                    // Padding value
                    xTempNewValue += paddingValue * xBasis[a];
                }
            }
            intensity += xTempNewValue * yBasis[b];
        }

        resultArray[tid2] = intensity;
    }
}
/* *************************************************************** */
__global__ void reg_resampleImage3D_kernel(float *resultArray,
                                           hipTextureObject_t floatingTexture,
                                           hipTextureObject_t deformationFieldTexture,
                                           hipTextureObject_t maskTexture,
                                           const mat44 floatingMatrix,
                                           const int3 floatingDim,
                                           const unsigned activeVoxelNumber,
                                           const float paddingValue) {
    const unsigned tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (tid < activeVoxelNumber) {
        const int tid2 = tex1Dfetch<int>(maskTexture, tid);

        // Get the real world deformation in the floating space
        float4 realDeformation = tex1Dfetch<float4>(deformationFieldTexture, tid);

        // Get the voxel-based deformation in the floating space
        float3 voxelDeformation;
        voxelDeformation.x = (floatingMatrix.m[0][0] * realDeformation.x +
                              floatingMatrix.m[0][1] * realDeformation.y +
                              floatingMatrix.m[0][2] * realDeformation.z +
                              floatingMatrix.m[0][3]);
        voxelDeformation.y = (floatingMatrix.m[1][0] * realDeformation.x +
                              floatingMatrix.m[1][1] * realDeformation.y +
                              floatingMatrix.m[1][2] * realDeformation.z +
                              floatingMatrix.m[1][3]);
        voxelDeformation.z = (floatingMatrix.m[2][0] * realDeformation.x +
                              floatingMatrix.m[2][1] * realDeformation.y +
                              floatingMatrix.m[2][2] * realDeformation.z +
                              floatingMatrix.m[2][3]);

        // Compute the linear interpolation
        const int3 previous = { Floor(voxelDeformation.x), Floor(voxelDeformation.y), Floor(voxelDeformation.z) };
        const float3 relative = { voxelDeformation.x - previous.x, voxelDeformation.y - previous.y, voxelDeformation.z - previous.z };
        float xBasis[2], yBasis[2], zBasis[2];
        InterpLinearKernel(relative.x, xBasis);
        InterpLinearKernel(relative.y, yBasis);
        InterpLinearKernel(relative.z, zBasis);

        float intensity = 0;
        for (short c = 0; c < 2; c++) {
            const int z = previous.z + c;
            float yTempNewValue = 0;
            for (short b = 0; b < 2; b++) {
                const int y = previous.y + b;
                float xTempNewValue = 0;
                for (short a = 0; a < 2; a++) {
                    const int x = previous.x + a;
                    if (-1 < x && x < floatingDim.x && -1 < y && y < floatingDim.y && -1 < z && z < floatingDim.z) {
                        xTempNewValue += tex3D<float>(floatingTexture, x, y, z) * xBasis[a];
                    } else {
                        // Padding value
                        xTempNewValue += paddingValue * xBasis[a];
                    }
                }
                yTempNewValue += xTempNewValue * yBasis[b];
            }
            intensity += yTempNewValue * zBasis[c];
        }

        resultArray[tid2] = intensity;
    }
}
/* *************************************************************** */
__global__ void reg_getImageGradient2D_kernel(float4 *gradientArray,
                                              hipTextureObject_t floatingTexture,
                                              hipTextureObject_t deformationFieldTexture,
                                              const mat44 floatingMatrix,
                                              const int3 floatingDim,
                                              const unsigned activeVoxelNumber,
                                              const float paddingValue) {
    const unsigned tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (tid < activeVoxelNumber) {
        // Get the real world deformation in the floating space
        float4 realDeformation = tex1Dfetch<float4>(deformationFieldTexture, tid);

        // Get the voxel-based deformation in the floating space
        float2 voxelDeformation;
        voxelDeformation.x = (floatingMatrix.m[0][0] * realDeformation.x +
                              floatingMatrix.m[0][1] * realDeformation.y +
                              floatingMatrix.m[0][3]);
        voxelDeformation.y = (floatingMatrix.m[1][0] * realDeformation.x +
                              floatingMatrix.m[1][1] * realDeformation.y +
                              floatingMatrix.m[1][3]);

        // Compute the gradient
        const int2 previous = { Floor(voxelDeformation.x), Floor(voxelDeformation.y) };
        float xBasis[2], yBasis[2];
        const float2 relative = { voxelDeformation.x - previous.x, voxelDeformation.y - previous.y };
        InterpLinearKernel(relative.x, xBasis);
        InterpLinearKernel(relative.y, yBasis);
        constexpr float deriv[] = { -1.0f, 1.0f };

        float4 gradientValue{};
        for (short b = 0; b < 2; b++) {
            float2 tempValueX{};
            const int y = previous.y + b;
            for (short a = 0; a < 2; a++) {
                const int x = previous.x + a;
                float intensity = paddingValue;

                if (-1 < x && x < floatingDim.x && -1 < y && y < floatingDim.y)
                    intensity = tex3D<float>(floatingTexture, x, y, 0);

                tempValueX.x += intensity * deriv[a];
                tempValueX.y += intensity * xBasis[a];
            }
            gradientValue.x += tempValueX.x * yBasis[b];
            gradientValue.y += tempValueX.y * deriv[b];
        }

        if (gradientValue.x != gradientValue.x)
            gradientValue.x = 0;
        if (gradientValue.y != gradientValue.y)
            gradientValue.y = 0;

        gradientArray[tid] = gradientValue;
    }
}
/* *************************************************************** */
__global__ void reg_getImageGradient3D_kernel(float4 *gradientArray,
                                              hipTextureObject_t floatingTexture,
                                              hipTextureObject_t deformationFieldTexture,
                                              const mat44 floatingMatrix,
                                              const int3 floatingDim,
                                              const unsigned activeVoxelNumber,
                                              const float paddingValue) {
    const unsigned tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if (tid < activeVoxelNumber) {
        // Get the real world deformation in the floating space
        float4 realDeformation = tex1Dfetch<float4>(deformationFieldTexture, tid);

        // Get the voxel-based deformation in the floating space
        float3 voxelDeformation;
        voxelDeformation.x = (floatingMatrix.m[0][0] * realDeformation.x +
                              floatingMatrix.m[0][1] * realDeformation.y +
                              floatingMatrix.m[0][2] * realDeformation.z +
                              floatingMatrix.m[0][3]);
        voxelDeformation.y = (floatingMatrix.m[1][0] * realDeformation.x +
                              floatingMatrix.m[1][1] * realDeformation.y +
                              floatingMatrix.m[1][2] * realDeformation.z +
                              floatingMatrix.m[1][3]);
        voxelDeformation.z = (floatingMatrix.m[2][0] * realDeformation.x +
                              floatingMatrix.m[2][1] * realDeformation.y +
                              floatingMatrix.m[2][2] * realDeformation.z +
                              floatingMatrix.m[2][3]);

        // Compute the gradient
        const int3 previous = { Floor(voxelDeformation.x), Floor(voxelDeformation.y), Floor(voxelDeformation.z) };
        float xBasis[2], yBasis[2], zBasis[2];
        const float3 relative = { voxelDeformation.x - previous.x, voxelDeformation.y - previous.y, voxelDeformation.z - previous.z };
        InterpLinearKernel(relative.x, xBasis);
        InterpLinearKernel(relative.y, yBasis);
        InterpLinearKernel(relative.z, zBasis);
        constexpr float deriv[] = { -1.0f, 1.0f };

        float4 gradientValue{};
        for (short c = 0; c < 2; c++) {
            const int z = previous.z + c;
            float3 tempValueY{};
            for (short b = 0; b < 2; b++) {
                float2 tempValueX{};
                const int y = previous.y + b;
                for (short a = 0; a < 2; a++) {
                    const int x = previous.x + a;
                    float intensity = paddingValue;

                    if (-1 < x && x < floatingDim.x && -1 < y && y < floatingDim.y && -1 < z && z < floatingDim.z)
                        intensity = tex3D<float>(floatingTexture, x, y, z);

                    tempValueX.x += intensity * deriv[a];
                    tempValueX.y += intensity * xBasis[a];
                }
                tempValueY.x += tempValueX.x * yBasis[b];
                tempValueY.y += tempValueX.y * deriv[b];
                tempValueY.z += tempValueX.y * yBasis[b];
            }
            gradientValue.x += tempValueY.x * zBasis[c];
            gradientValue.y += tempValueY.y * zBasis[c];
            gradientValue.z += tempValueY.z * deriv[c];
        }

        if (gradientValue.x != gradientValue.x)
            gradientValue.x = 0;
        if (gradientValue.y != gradientValue.y)
            gradientValue.y = 0;
        if (gradientValue.z != gradientValue.z)
            gradientValue.z = 0;

        gradientArray[tid] = gradientValue;
    }
}
/* *************************************************************** */
