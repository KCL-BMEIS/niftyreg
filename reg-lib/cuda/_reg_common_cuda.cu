#include "hip/hip_runtime.h"
/**
 * @file _reg_common_cuda.cu
 * @author Marc Modat
 * @date 25/03/2009
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 * See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#include "_reg_common_cuda.h"
#include "_reg_tools.h"
#include "_reg_blocksize_gpu.h"

 /* ******************************** */
template <class NIFTI_TYPE>
int cudaCommon_transferNiftiToNiftiOnDevice1(nifti_image *image_d, nifti_image *img) {
    const unsigned int memSize = img->dim[1] * img->dim[2] * img->dim[3] * sizeof(NIFTI_TYPE);

    int *g_dim;
    float* g_pixdim;
    NIFTI_TYPE* g_data;

    NR_CUDA_SAFE_CALL(hipMalloc((void**)&g_dim, 8 * sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMalloc((void**)&g_pixdim, 8 * sizeof(float)));
    NR_CUDA_SAFE_CALL(hipMalloc((void**)&g_data, memSize));

    NIFTI_TYPE *array_h = static_cast<NIFTI_TYPE*>(img->data);
    NR_CUDA_SAFE_CALL(hipMemcpy(image_d, img, sizeof(nifti_image), hipMemcpyHostToDevice));

    NR_CUDA_SAFE_CALL(hipMemcpy(image_d->data, array_h, memSize, hipMemcpyHostToDevice));
    NR_CUDA_SAFE_CALL(hipMemcpy(image_d->dim, img->dim, 8 * sizeof(int), hipMemcpyHostToDevice));
    NR_CUDA_SAFE_CALL(hipMemcpy(image_d->pixdim, img->pixdim, 8 * sizeof(float), hipMemcpyHostToDevice));

    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToNiftiOnDevice1<float>(nifti_image*, nifti_image*);
template int cudaCommon_transferNiftiToNiftiOnDevice1<double>(nifti_image*, nifti_image*);
/* ******************************** */
template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferNiftiToArrayOnDevice1(DTYPE *array_d, nifti_image *img) {
    if (sizeof(DTYPE) != sizeof(NIFTI_TYPE)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        const unsigned int memSize = img->dim[1] * img->dim[2] * img->dim[3] * sizeof(DTYPE);
        NIFTI_TYPE *array_h = static_cast<NIFTI_TYPE*>(img->data);
        NR_CUDA_SAFE_CALL(hipMemcpy(array_d, array_h, memSize, hipMemcpyHostToDevice));
    }
    return EXIT_SUCCESS;
}
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferNiftiToArrayOnDevice(DTYPE *array_d, nifti_image *img) {
    if (sizeof(DTYPE) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = CalcVoxelNumber(*img);
        float4 *array_h = (float4*)calloc(voxelNumber, sizeof(float4));
        for (size_t i = 0; i < voxelNumber; i++)
            array_h[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].z = *niftiImgValues++;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].w = *niftiImgValues++;
        }
        NR_CUDA_SAFE_CALL(hipMemcpy(array_d, array_h, voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
        free(array_h);
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DTYPE, float>(array_d, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<double>(double*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float>(float*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<int>(int*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(float4*, nifti_image*);
/* ******************************** */
template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferNiftiToArrayOnDevice1(DTYPE *array_d, DTYPE *array2_d, nifti_image *img) {
    if (sizeof(DTYPE) != sizeof(NIFTI_TYPE)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        const unsigned int memSize = img->dim[1] * img->dim[2] * img->dim[3] * sizeof(DTYPE);
        NIFTI_TYPE *array_h = static_cast<NIFTI_TYPE*>(img->data);
        NIFTI_TYPE *array2_h = &array_h[img->dim[1] * img->dim[2] * img->dim[3]];
        NR_CUDA_SAFE_CALL(hipMemcpy(array_d, array_h, memSize, hipMemcpyHostToDevice));
        NR_CUDA_SAFE_CALL(hipMemcpy(array2_d, array2_h, memSize, hipMemcpyHostToDevice));
    }
    return EXIT_SUCCESS;
}
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferNiftiToArrayOnDevice(DTYPE *array_d, DTYPE *array2_d, nifti_image *img) {
    if (sizeof(DTYPE) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        float *niftiImgValues = static_cast<float *>(img->data);
        const size_t voxelNumber = CalcVoxelNumber(*img);
        float4 *array_h = (float4*)calloc(voxelNumber, sizeof(float4));
        float4 *array2_h = (float4*)calloc(voxelNumber, sizeof(float4));
        for (size_t i = 0; i < voxelNumber; i++)
            array_h[i].x = *niftiImgValues++;
        for (size_t i = 0; i < voxelNumber; i++)
            array2_h[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].y = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].z = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].z = *niftiImgValues++;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].w = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].w = *niftiImgValues++;
        }
        NR_CUDA_SAFE_CALL(hipMemcpy(array_d, array_h, voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
        NR_CUDA_SAFE_CALL(hipMemcpy(array2_d, array2_h, voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
        free(array_h);
        free(array2_h);
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DTYPE, float>(array_d, array2_d, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<float>(float*, float*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(double*, double*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(float4*, float4*, nifti_image*); // for deformation field
/* ******************************** */
template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferNiftiToArrayOnDevice1(hipArray *cuArray_d, nifti_image *img) {
    if (sizeof(DTYPE) != sizeof(NIFTI_TYPE)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        NIFTI_TYPE *array_h = static_cast<NIFTI_TYPE*>(img->data);

        hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
        copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
        copyParams.srcPtr = make_hipPitchedPtr((void*)array_h,
                                                copyParams.extent.width * sizeof(DTYPE),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray_d;
        copyParams.kind = hipMemcpyHostToDevice;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    }
    return EXIT_SUCCESS;
}
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferNiftiToArrayOnDevice(hipArray *cuArray_d, nifti_image *img) {
    if (sizeof(DTYPE) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        float *niftiImgValues = static_cast<float *>(img->data);
        const size_t voxelNumber = CalcVoxelNumber(*img);
        float4 *array_h = (float4*)calloc(voxelNumber, sizeof(float4));

        for (size_t i = 0; i < voxelNumber; i++)
            array_h[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].z = *niftiImgValues++;
        }
        if (img->dim[5] == 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].w = *niftiImgValues++;
        }
        hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
        copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
        copyParams.srcPtr = make_hipPitchedPtr((void*)array_h,
                                                copyParams.extent.width * sizeof(DTYPE),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray_d;
        copyParams.kind = hipMemcpyHostToDevice;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
        free(array_h);
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DTYPE, float>(cuArray_d, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<int>(hipArray*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float>(hipArray*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(hipArray*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(hipArray*, nifti_image*); // for deformation field
/* ******************************** */
template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferNiftiToArrayOnDevice1(hipArray *cuArray_d, hipArray *cuArray2_d, nifti_image *img) {
    if (sizeof(DTYPE) != sizeof(NIFTI_TYPE)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        NIFTI_TYPE *array_h = static_cast<NIFTI_TYPE*>(img->data);
        NIFTI_TYPE *array2_h = &array_h[img->dim[1] * img->dim[2] * img->dim[3]];

        hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
        copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
        copyParams.kind = hipMemcpyHostToDevice;
        // First timepoint
        copyParams.srcPtr = make_hipPitchedPtr((void*)array_h,
                                                copyParams.extent.width * sizeof(DTYPE),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray_d;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
        // Second timepoint
        copyParams.srcPtr = make_hipPitchedPtr((void*)array2_h,
                                                copyParams.extent.width * sizeof(DTYPE),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray2_d;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    }
    return EXIT_SUCCESS;
}
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferNiftiToArrayOnDevice(hipArray *cuArray_d, hipArray *cuArray2_d, nifti_image *img) {
    if (sizeof(DTYPE) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = CalcVoxelNumber(*img);
        float4 *array_h = (float4*)calloc(voxelNumber, sizeof(float4));
        float4 *array2_h = (float4*)calloc(voxelNumber, sizeof(float4));

        for (size_t i = 0; i < voxelNumber; i++)
            array_h[i].x = *niftiImgValues++;
        for (size_t i = 0; i < voxelNumber; i++)
            array2_h[i].x = *niftiImgValues++;

        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].y = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].y = *niftiImgValues++;
        }

        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].z = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].z = *niftiImgValues++;
        }

        if (img->dim[5] == 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].w = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].w = *niftiImgValues++;
        }

        hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
        copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
        copyParams.kind = hipMemcpyHostToDevice;
        // First timepoint
        copyParams.srcPtr = make_hipPitchedPtr((void*)array_h,
                                                copyParams.extent.width * sizeof(DTYPE),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray_d;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
        free(array_h);
        // Second timepoint
        copyParams.srcPtr = make_hipPitchedPtr((void*)array2_h,
                                                copyParams.extent.width * sizeof(DTYPE),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray2_d;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
        free(array2_h);
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DTYPE, float>(cuArray_d, cuArray2_d, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<float>(hipArray*, hipArray*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(hipArray*, hipArray*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(hipArray*, hipArray*, nifti_image*); // for deformation field
/* ******************************** */
template <class DTYPE>
int cudaCommon_allocateArrayToDevice(hipArray **cuArray_d, int *dim) {
    const hipExtent volumeSize = make_hipExtent(dim[1], dim[2], dim[3]);
    hipChannelFormatDesc texDesc = hipCreateChannelDesc<DTYPE>();
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(cuArray_d, &texDesc, volumeSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(hipArray**, int*);
template int cudaCommon_allocateArrayToDevice<double>(hipArray**, int*);
template int cudaCommon_allocateArrayToDevice<float4>(hipArray**, int*); // for deformation field
/* ******************************** */
template <class DTYPE>
int cudaCommon_allocateArrayToDevice(hipArray **cuArray_d, hipArray **cuArray2_d, int *dim) {
    const hipExtent volumeSize = make_hipExtent(dim[1], dim[2], dim[3]);
    hipChannelFormatDesc texDesc = hipCreateChannelDesc<DTYPE>();
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(cuArray_d, &texDesc, volumeSize));
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(cuArray2_d, &texDesc, volumeSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(hipArray**, hipArray**, int*);
template int cudaCommon_allocateArrayToDevice<double>(hipArray**, hipArray**, int*);
template int cudaCommon_allocateArrayToDevice<float4>(hipArray**, hipArray**, int*); // for deformation field
/* ******************************** */
template <class DTYPE>
int cudaCommon_allocateArrayToDevice(DTYPE **array_d, int *dim) {
    const unsigned int memSize = dim[1] * dim[2] * dim[3] * sizeof(DTYPE);
    NR_CUDA_SAFE_CALL(hipMalloc(array_d, memSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float**, int*);
template int cudaCommon_allocateArrayToDevice<double>(double**, int*);
template int cudaCommon_allocateArrayToDevice<int>(int**, int*);
template int cudaCommon_allocateArrayToDevice<float4>(float4**, int*); // for deformation field
/* ******************************** */
template <class DTYPE>
int cudaCommon_allocateArrayToDevice(DTYPE **array_d, int vox) {
    const unsigned int memSize = vox * sizeof(DTYPE);
    NR_CUDA_SAFE_CALL(hipMalloc(array_d, memSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float**, int);
template int cudaCommon_allocateArrayToDevice<double>(double**, int);
template int cudaCommon_allocateArrayToDevice<int>(int**, int);
template int cudaCommon_allocateArrayToDevice<float4>(float4**, int); // for deformation field
/* ******************************** */
template <class DTYPE>
int cudaCommon_allocateArrayToDevice(DTYPE **array_d, DTYPE **array2_d, int *dim) {
    const unsigned int memSize = dim[1] * dim[2] * dim[3] * sizeof(DTYPE);
    NR_CUDA_SAFE_CALL(hipMalloc(array_d, memSize));
    NR_CUDA_SAFE_CALL(hipMalloc(array2_d, memSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float**, float**, int*);
template int cudaCommon_allocateArrayToDevice<double>(double**, double**, int*);
template int  cudaCommon_allocateArrayToDevice<float4>(float4**, float4**, int*); // for deformation field
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferFromDeviceToCpu(DTYPE *cpuPtr, DTYPE *cuPtr, const unsigned int nElements) {
    NR_CUDA_SAFE_CALL(hipMemcpy((void*)cpuPtr, (void*)cuPtr, nElements * sizeof(DTYPE), hipMemcpyDeviceToHost));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToCpu<float>(float *cpuPtr, float *cuPtr, const unsigned int nElements);
template int cudaCommon_transferFromDeviceToCpu<double>(double *cpuPtr, double *cuPtr, const unsigned int nElements);
/* ******************************** */
template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferFromDeviceToNifti1(nifti_image *img, DTYPE *array_d) {
    if (sizeof(DTYPE) != sizeof(NIFTI_TYPE)) {
        reg_print_fct_error("cudaCommon_transferFromDeviceToNifti1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        NIFTI_TYPE *array_h = static_cast<NIFTI_TYPE*>(img->data);
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array_h, (void*)array_d, img->nvox * sizeof(DTYPE), hipMemcpyDeviceToHost));
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToNifti1<float, float>(nifti_image *img, float *array_d);
template int cudaCommon_transferFromDeviceToNifti1<double, double>(nifti_image *img, double *array_d);
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferFromDeviceToNifti(nifti_image *img, DTYPE *array_d) {
    if (sizeof(DTYPE) == sizeof(float4)) {
        // A nifti 5D volume is expected
        if (img->dim[0] < 5 || img->dim[4]>1 || img->dim[5] < 2 || img->datatype != NIFTI_TYPE_FLOAT32) {
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The nifti image is not a 5D volume");
            return EXIT_FAILURE;
        }

        float4 *array_h;
        const size_t voxelNumber = CalcVoxelNumber(*img);
        NR_CUDA_SAFE_CALL(hipHostMalloc(&array_h, voxelNumber * sizeof(float4)));
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array_h, (const void*)array_d, voxelNumber * sizeof(float4), hipMemcpyDeviceToHost));
        float *niftiImgValues = static_cast<float*>(img->data);

        for (size_t i = 0; i < voxelNumber; i++)
            *niftiImgValues++ = array_h[i].x;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array_h[i].y;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array_h[i].z;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array_h[i].w;
        }
        NR_CUDA_SAFE_CALL(hipHostFree(array_h));

        return EXIT_SUCCESS;
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferFromDeviceToNifti1<DTYPE, float>(img, array_d);
        default:
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
}
template int cudaCommon_transferFromDeviceToNifti<float>(nifti_image*, float*);
template int cudaCommon_transferFromDeviceToNifti<double>(nifti_image*, double*);
template int cudaCommon_transferFromDeviceToNifti<float4>(nifti_image*, float4*); // for deformation field
/* ******************************** */
template<>
int cudaCommon_transferFromDeviceToNifti(nifti_image *img, hipArray *cuArray_d) {
    if (img->datatype != NIFTI_TYPE_FLOAT32) {
        reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
        reg_print_msg_error("The image data type is not supported");
        return EXIT_FAILURE;
    }

    hipMemcpy3DParms copyParams = {0};
    copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
    copyParams.srcArray = cuArray_d;
    copyParams.dstPtr = make_hipPitchedPtr((void*)(img->data), copyParams.extent.width * sizeof(float),
                                            copyParams.extent.width, copyParams.extent.height);
    copyParams.kind = hipMemcpyDeviceToHost;
    NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    return EXIT_SUCCESS;
}
/* ******************************** */
template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferFromDeviceToNifti1(nifti_image *img, DTYPE *array_d, DTYPE *array2_d) {
    if (sizeof(DTYPE) != sizeof(NIFTI_TYPE)) {
        reg_print_fct_error("cudaCommon_transferFromDeviceToNifti1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        const size_t voxelNumber = CalcVoxelNumber(*img);
        NIFTI_TYPE *array_h = static_cast<NIFTI_TYPE*>(img->data);
        NIFTI_TYPE *array2_h = &array_h[voxelNumber];
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array_h, (void*)array_d, voxelNumber * sizeof(DTYPE), hipMemcpyDeviceToHost));
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array2_h, (void*)array2_d, voxelNumber * sizeof(DTYPE), hipMemcpyDeviceToHost));
    }
    return EXIT_SUCCESS;
}
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferFromDeviceToNifti(nifti_image *img, DTYPE *array_d, DTYPE *array2_d) {
    if (sizeof(DTYPE) == sizeof(float4)) {
        // A nifti 5D volume is expected
        if (img->dim[0] < 5 || img->dim[4]>1 || img->dim[5] < 2 || img->datatype != NIFTI_TYPE_FLOAT32) {
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The nifti image is not a 5D volume");
            return EXIT_FAILURE;
        }
        const size_t voxelNumber = CalcVoxelNumber(*img);
        float4 *array_h = nullptr;
        float4 *array2_h = nullptr;
        NR_CUDA_SAFE_CALL(hipHostMalloc(&array_h, voxelNumber * sizeof(float4)));
        NR_CUDA_SAFE_CALL(hipHostMalloc(&array2_h, voxelNumber * sizeof(float4)));
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array_h, (const void*)array_d, voxelNumber * sizeof(float4), hipMemcpyDeviceToHost));
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array2_h, (const void*)array2_d, voxelNumber * sizeof(float4), hipMemcpyDeviceToHost));
        float *niftiImgValues = static_cast<float *>(img->data);
        for (size_t i = 0; i < voxelNumber; i++) {
            *niftiImgValues++ = array_h[i].x;
        }
        for (size_t i = 0; i < voxelNumber; i++) {
            *niftiImgValues++ = array2_h[i].x;
        }
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array_h[i].y;
            }
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array2_h[i].y;
            }
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array_h[i].z;
            }
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array2_h[i].z;
            }
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array_h[i].w;
            }
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array2_h[i].w;
            }
        }
        NR_CUDA_SAFE_CALL(hipHostFree(array_h));
        NR_CUDA_SAFE_CALL(hipHostFree(array2_h));

        return EXIT_SUCCESS;
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferFromDeviceToNifti1<DTYPE, float>(img, array_d, array2_d);
        default:
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
}
template int cudaCommon_transferFromDeviceToNifti<float>(nifti_image*, float*, float*);
template int cudaCommon_transferFromDeviceToNifti<double>(nifti_image*, double*, double*);
template int cudaCommon_transferFromDeviceToNifti<float4>(nifti_image*, float4*, float4*); // for deformation field
/* ******************************** */
void cudaCommon_free(hipArray *cuArray_d) {
    NR_CUDA_SAFE_CALL(hipFreeArray(cuArray_d));
}
/* ******************************** */
template <class DTYPE>
void cudaCommon_free(DTYPE *array_d) {
    NR_CUDA_SAFE_CALL(hipFree(array_d));
}
template void cudaCommon_free<int>(int*);
template void cudaCommon_free<float>(float*);
template void cudaCommon_free<double>(double*);
template void cudaCommon_free<float4>(float4*);
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferFromDeviceToNiftiSimple(DTYPE *array_d, nifti_image *img) {
    NR_CUDA_SAFE_CALL(hipMemcpy(array_d, img->data, img->nvox * sizeof(DTYPE), hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToNiftiSimple<int>(int*, nifti_image*);
template int cudaCommon_transferFromDeviceToNiftiSimple<float>(float*, nifti_image*);
template int cudaCommon_transferFromDeviceToNiftiSimple<double>(double*, nifti_image*);
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferFromDeviceToNiftiSimple1(DTYPE *array_d, DTYPE *img, const unsigned int nvox) {
    NR_CUDA_SAFE_CALL(hipMemcpy(array_d, img, nvox * sizeof(DTYPE), hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToNiftiSimple1<int>(int*, int*, const unsigned);
template int cudaCommon_transferFromDeviceToNiftiSimple1<float>(float*, float*, const unsigned);
template int cudaCommon_transferFromDeviceToNiftiSimple1<double>(double*, double*, const unsigned);
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferArrayFromCpuToDevice(DTYPE *array_d, DTYPE *array_cpu, const unsigned int nElements) {
    const unsigned int memSize = nElements * sizeof(DTYPE);
    NR_CUDA_SAFE_CALL(hipMemcpy(array_d, array_cpu, memSize, hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferArrayFromCpuToDevice<int>(int*, int*, const unsigned int);
template int cudaCommon_transferArrayFromCpuToDevice<float>(float*, float*, const unsigned int);
template int cudaCommon_transferArrayFromCpuToDevice<double>(double*, double*, const unsigned int);
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferArrayFromDeviceToCpu(DTYPE *array_cpu, DTYPE *array_d, const unsigned int nElements) {
    const unsigned int memSize = nElements * sizeof(DTYPE);
    NR_CUDA_SAFE_CALL(hipMemcpy(array_cpu, array_d, memSize, hipMemcpyDeviceToHost));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferArrayFromDeviceToCpu<int>(int*, int*, const unsigned int);
template int cudaCommon_transferArrayFromDeviceToCpu<float>(float*, float*, const unsigned int);
template int cudaCommon_transferArrayFromDeviceToCpu<double>(double*, double*, const unsigned int);
/* ******************************** */
