#include "hip/hip_runtime.h"
/**
 * @file _reg_common_cuda.cu
 * @author Marc Modat
 * @date 25/03/2009
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 * See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#include "_reg_common_cuda.h"
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>

/* *************************************************************** */
template <class NiftiType>
int cudaCommon_transferNiftiToNiftiOnDevice1(nifti_image *imageCuda, const nifti_image *img) {
    const size_t memSize = NiftiImage::calcVoxelNumber(img, 3) * sizeof(NiftiType);
    NR_CUDA_SAFE_CALL(hipMemcpy(imageCuda, img, sizeof(nifti_image), hipMemcpyHostToDevice));
    NR_CUDA_SAFE_CALL(hipMemcpy(imageCuda->data, img->data, memSize, hipMemcpyHostToDevice));
    NR_CUDA_SAFE_CALL(hipMemcpy(imageCuda->dim, img->dim, 8 * sizeof(int), hipMemcpyHostToDevice));
    NR_CUDA_SAFE_CALL(hipMemcpy(imageCuda->pixdim, img->pixdim, 8 * sizeof(float), hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferNiftiToArrayOnDevice1(DataType *arrayCuda, const nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        const size_t memSize = NiftiImage::calcVoxelNumber(img, 3) * sizeof(NiftiType);
        NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, img->data, memSize, hipMemcpyHostToDevice));
    }
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferNiftiToArrayOnDevice(DataType *arrayCuda, const nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        const float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        unique_ptr<float4[]> array(new float4[voxelNumber]());
        for (size_t i = 0; i < voxelNumber; i++)
            array[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].z = *niftiImgValues++;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].w = *niftiImgValues++;
        }
        NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, array.get(), voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DataType, float>(arrayCuda, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<double>(double*, const nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float>(float*, const nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<int>(int*, const nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(float4*, const nifti_image*);
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferNiftiToArrayOnDevice1(DataType *array1Cuda, DataType *array2Cuda, const nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        const size_t memSize = voxelNumber * sizeof(DataType);
        const NiftiType *array1 = static_cast<NiftiType*>(img->data);
        const NiftiType *array2 = &array1[voxelNumber];
        NR_CUDA_SAFE_CALL(hipMemcpy(array1Cuda, array1, memSize, hipMemcpyHostToDevice));
        NR_CUDA_SAFE_CALL(hipMemcpy(array2Cuda, array2, memSize, hipMemcpyHostToDevice));
    }
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferNiftiToArrayOnDevice(DataType *array1Cuda, DataType *array2Cuda, const nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        const float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        unique_ptr<float4[]> array1(new float4[voxelNumber]());
        unique_ptr<float4[]> array2(new float4[voxelNumber]());
        for (size_t i = 0; i < voxelNumber; i++)
            array1[i].x = *niftiImgValues++;
        for (size_t i = 0; i < voxelNumber; i++)
            array2[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].y = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].z = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].z = *niftiImgValues++;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].w = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].w = *niftiImgValues++;
        }
        NR_CUDA_SAFE_CALL(hipMemcpy(array1Cuda, array1.get(), voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
        NR_CUDA_SAFE_CALL(hipMemcpy(array2Cuda, array2.get(), voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DataType, float>(array1Cuda, array2Cuda, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<float>(float*, float*, const nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(double*, double*, const nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(float4*, float4*, const nifti_image*); // for deformation field
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferNiftiToArrayOnDevice1(hipArray *arrayCuda, const nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        hipMemcpy3DParms copyParams{};
        copyParams.extent = make_hipExtent(std::abs(img->dim[1]), std::abs(img->dim[2]), std::abs(img->dim[3]));
        copyParams.srcPtr = make_hipPitchedPtr(img->data,
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = arrayCuda;
        copyParams.kind = hipMemcpyHostToDevice;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    }
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferNiftiToArrayOnDevice(hipArray *arrayCuda, const nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        const float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        unique_ptr<float4[]> array(new float4[voxelNumber]());
        for (size_t i = 0; i < voxelNumber; i++)
            array[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].z = *niftiImgValues++;
        }
        if (img->dim[5] == 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array[i].w = *niftiImgValues++;
        }
        hipMemcpy3DParms copyParams{};
        copyParams.extent = make_hipExtent(std::abs(img->dim[1]), std::abs(img->dim[2]), std::abs(img->dim[3]));
        copyParams.srcPtr = make_hipPitchedPtr(array.get(),
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = arrayCuda;
        copyParams.kind = hipMemcpyHostToDevice;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DataType, float>(arrayCuda, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<int>(hipArray*, const nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float>(hipArray*, const nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(hipArray*, const nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(hipArray*, const nifti_image*); // for deformation field
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferNiftiToArrayOnDevice1(hipArray *array1Cuda, hipArray *array2Cuda, const nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        NiftiType *array1 = static_cast<NiftiType*>(img->data);
        NiftiType *array2 = &array1[NiftiImage::calcVoxelNumber(img, 3)];
        hipMemcpy3DParms copyParams{};
        copyParams.extent = make_hipExtent(std::abs(img->dim[1]), std::abs(img->dim[2]), std::abs(img->dim[3]));
        copyParams.kind = hipMemcpyHostToDevice;
        // First timepoint
        copyParams.srcPtr = make_hipPitchedPtr(array1,
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = array1Cuda;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
        // Second timepoint
        copyParams.srcPtr = make_hipPitchedPtr(array2,
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = array2Cuda;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    }
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferNiftiToArrayOnDevice(hipArray *array1Cuda, hipArray *array2Cuda, const nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        const float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        unique_ptr<float4[]> array1(new float4[voxelNumber]());
        unique_ptr<float4[]> array2(new float4[voxelNumber]());
        for (size_t i = 0; i < voxelNumber; i++)
            array1[i].x = *niftiImgValues++;
        for (size_t i = 0; i < voxelNumber; i++)
            array2[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].y = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].z = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].z = *niftiImgValues++;
        }
        if (img->dim[5] == 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array1[i].w = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2[i].w = *niftiImgValues++;
        }

        hipMemcpy3DParms copyParams{};
        copyParams.extent = make_hipExtent(std::abs(img->dim[1]), std::abs(img->dim[2]), std::abs(img->dim[3]));
        copyParams.kind = hipMemcpyHostToDevice;
        // First timepoint
        copyParams.srcPtr = make_hipPitchedPtr(array1.get(),
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = array1Cuda;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
        // Second timepoint
        copyParams.srcPtr = make_hipPitchedPtr(array2.get(),
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = array2Cuda;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DataType, float>(array1Cuda, array2Cuda, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<float>(hipArray*, hipArray*, const nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(hipArray*, hipArray*, const nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(hipArray*, hipArray*, const nifti_image*); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_allocateArrayToDevice(hipArray **arrayCuda, const int *dim) {
    const hipExtent volumeSize = make_hipExtent(std::abs(dim[1]), std::abs(dim[2]), std::abs(dim[3]));
    hipChannelFormatDesc texDesc = hipCreateChannelDesc<DataType>();
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(arrayCuda, &texDesc, volumeSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(hipArray**, const int*);
template int cudaCommon_allocateArrayToDevice<double>(hipArray**, const int*);
template int cudaCommon_allocateArrayToDevice<float4>(hipArray**, const int*); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_allocateArrayToDevice(hipArray **array1Cuda, hipArray **array2Cuda, const int *dim) {
    const hipExtent volumeSize = make_hipExtent(std::abs(dim[1]), std::abs(dim[2]), std::abs(dim[3]));
    hipChannelFormatDesc texDesc = hipCreateChannelDesc<DataType>();
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(array1Cuda, &texDesc, volumeSize));
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(array2Cuda, &texDesc, volumeSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(hipArray**, hipArray**, const int*);
template int cudaCommon_allocateArrayToDevice<double>(hipArray**, hipArray**, const int*);
template int cudaCommon_allocateArrayToDevice<float4>(hipArray**, hipArray**, const int*); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_allocateArrayToDevice(DataType **arrayCuda, const int *dim) {
    const size_t memSize = (size_t)std::abs(dim[1]) * (size_t)std::abs(dim[2]) * (size_t)std::abs(dim[3]) * sizeof(DataType);
    NR_CUDA_SAFE_CALL(hipMalloc(arrayCuda, memSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float**, const int*);
template int cudaCommon_allocateArrayToDevice<double>(double**, const int*);
template int cudaCommon_allocateArrayToDevice<int>(int**, const int*);
template int cudaCommon_allocateArrayToDevice<float4>(float4**, const int*); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_allocateArrayToDevice(DataType **arrayCuda, const size_t& nVoxels) {
    NR_CUDA_SAFE_CALL(hipMalloc(arrayCuda, nVoxels * sizeof(DataType)));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float**, const size_t&);
template int cudaCommon_allocateArrayToDevice<double>(double**, const size_t&);
template int cudaCommon_allocateArrayToDevice<int>(int**, const size_t&);
template int cudaCommon_allocateArrayToDevice<float4>(float4**, const size_t&); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_allocateArrayToDevice(DataType **array1Cuda, DataType **array2Cuda, const int *dim) {
    const size_t memSize = (size_t)std::abs(dim[1]) * (size_t)std::abs(dim[2]) * (size_t)std::abs(dim[3]) * sizeof(DataType);
    NR_CUDA_SAFE_CALL(hipMalloc(array1Cuda, memSize));
    NR_CUDA_SAFE_CALL(hipMalloc(array2Cuda, memSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float**, float**, const int*);
template int cudaCommon_allocateArrayToDevice<double>(double**, double**, const int*);
template int  cudaCommon_allocateArrayToDevice<float4>(float4**, float4**, const int*); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferFromDeviceToCpu(DataType *cpuPtr, const DataType *cuPtr, const size_t& nElements) {
    NR_CUDA_SAFE_CALL(hipMemcpy(cpuPtr, cuPtr, nElements * sizeof(DataType), hipMemcpyDeviceToHost));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToCpu<float>(float*, const float*, const size_t&);
template int cudaCommon_transferFromDeviceToCpu<double>(double*, const double*, const size_t&);
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferFromDeviceToNifti1(nifti_image *img, const DataType *arrayCuda) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferFromDeviceToNifti1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        NR_CUDA_SAFE_CALL(hipMemcpy(img->data, arrayCuda, img->nvox * sizeof(DataType), hipMemcpyDeviceToHost));
    }
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferFromDeviceToNifti(nifti_image *img, const DataType *arrayCuda) {
    if (sizeof(DataType) == sizeof(float4)) {
        // A nifti 5D volume is expected
        if (img->dim[0] < 5 || img->dim[4]>1 || img->dim[5] < 2 || img->datatype != NIFTI_TYPE_FLOAT32) {
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The nifti image is not a 5D volume");
            return EXIT_FAILURE;
        }
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        thrust::device_ptr<const float4> arrayCudaPtr(reinterpret_cast<const float4*>(arrayCuda));
        const thrust::host_vector<float4> array(arrayCudaPtr, arrayCudaPtr + voxelNumber);
        float *niftiImgValues = static_cast<float*>(img->data);
        for (size_t i = 0; i < voxelNumber; i++)
            *niftiImgValues++ = array[i].x;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array[i].y;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array[i].z;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array[i].w;
        }
        return EXIT_SUCCESS;
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferFromDeviceToNifti1<DataType, float>(img, arrayCuda);
        default:
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
}
template int cudaCommon_transferFromDeviceToNifti<float>(nifti_image*, const float*);
template int cudaCommon_transferFromDeviceToNifti<double>(nifti_image*, const double*);
template int cudaCommon_transferFromDeviceToNifti<float4>(nifti_image*, const float4*); // for deformation field
/* *************************************************************** */
template<>
int cudaCommon_transferFromDeviceToNifti(nifti_image *img, const hipArray *arrayCuda) {
    if (img->datatype != NIFTI_TYPE_FLOAT32) {
        reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
        reg_print_msg_error("The image data type is not supported");
        return EXIT_FAILURE;
    }
    hipMemcpy3DParms copyParams{};
    copyParams.extent = make_hipExtent(std::abs(img->dim[1]), std::abs(img->dim[2]), std::abs(img->dim[3]));
    copyParams.srcArray = const_cast<hipArray*>(arrayCuda);
    copyParams.dstPtr = make_hipPitchedPtr(img->data,
                                            copyParams.extent.width * sizeof(float),
                                            copyParams.extent.width,
                                            copyParams.extent.height);
    copyParams.kind = hipMemcpyDeviceToHost;
    NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferFromDeviceToNifti1(nifti_image *img, const DataType *array1Cuda, const DataType *array2Cuda) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferFromDeviceToNifti1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        NiftiType *array1 = static_cast<NiftiType*>(img->data);
        NiftiType *array2 = &array1[voxelNumber];
        NR_CUDA_SAFE_CALL(hipMemcpy(array1, array1Cuda, voxelNumber * sizeof(DataType), hipMemcpyDeviceToHost));
        NR_CUDA_SAFE_CALL(hipMemcpy(array2, array2Cuda, voxelNumber * sizeof(DataType), hipMemcpyDeviceToHost));
    }
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferFromDeviceToNifti(nifti_image *img, const DataType *array1Cuda, const DataType *array2Cuda) {
    if (sizeof(DataType) == sizeof(float4)) {
        // A nifti 5D volume is expected
        if (img->dim[0] < 5 || img->dim[4]>1 || img->dim[5] < 2 || img->datatype != NIFTI_TYPE_FLOAT32) {
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The nifti image is not a 5D volume");
            return EXIT_FAILURE;
        }
        const size_t voxelNumber = NiftiImage::calcVoxelNumber(img, 3);
        thrust::device_ptr<const float4> array1CudaPtr(reinterpret_cast<const float4*>(array1Cuda));
        thrust::device_ptr<const float4> array2CudaPtr(reinterpret_cast<const float4*>(array2Cuda));
        const thrust::host_vector<float4> array1(array1CudaPtr, array1CudaPtr + voxelNumber);
        const thrust::host_vector<float4> array2(array2CudaPtr, array2CudaPtr + voxelNumber);
        float *niftiImgValues = static_cast<float*>(img->data);
        for (size_t i = 0; i < voxelNumber; i++) {
            *niftiImgValues++ = array1[i].x;
        }
        for (size_t i = 0; i < voxelNumber; i++) {
            *niftiImgValues++ = array2[i].x;
        }
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array1[i].y;
            }
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array2[i].y;
            }
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array1[i].z;
            }
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array2[i].z;
            }
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array1[i].w;
            }
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array2[i].w;
            }
        }
        return EXIT_SUCCESS;
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferFromDeviceToNifti1<DataType, float>(img, array1Cuda, array2Cuda);
        default:
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
}
template int cudaCommon_transferFromDeviceToNifti<float>(nifti_image*, const float*, const float*);
template int cudaCommon_transferFromDeviceToNifti<double>(nifti_image*, const double*, const double*);
template int cudaCommon_transferFromDeviceToNifti<float4>(nifti_image*, const float4*, const float4*); // for deformation field
/* *************************************************************** */
void cudaCommon_free(hipArray *arrayCuda) {
    if (arrayCuda != nullptr)
        NR_CUDA_SAFE_CALL(hipFreeArray(arrayCuda));
}
/* *************************************************************** */
template <class DataType>
void cudaCommon_free(DataType *arrayCuda) {
    if (arrayCuda != nullptr)
        NR_CUDA_SAFE_CALL(hipFree(arrayCuda));
}
template void cudaCommon_free<int>(int*);
template void cudaCommon_free<float>(float*);
template void cudaCommon_free<double>(double*);
template void cudaCommon_free<float4>(float4*);
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferFromDeviceToNiftiSimple(DataType *arrayCuda, const nifti_image *img) {
    NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, img->data, img->nvox * sizeof(DataType), hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToNiftiSimple<int>(int*, const nifti_image*);
template int cudaCommon_transferFromDeviceToNiftiSimple<float>(float*, const nifti_image*);
template int cudaCommon_transferFromDeviceToNiftiSimple<double>(double*, const nifti_image*);
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferFromDeviceToNiftiSimple1(DataType *arrayCuda, const DataType *img, const size_t& nvox) {
    NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, img, nvox * sizeof(DataType), hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToNiftiSimple1<int>(int*, const int*, const size_t&);
template int cudaCommon_transferFromDeviceToNiftiSimple1<float>(float*, const float*, const size_t&);
template int cudaCommon_transferFromDeviceToNiftiSimple1<double>(double*, const double*, const size_t&);
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferArrayFromCpuToDevice(DataType *arrayCuda, const DataType *arrayCpu, const size_t& nElements) {
    NR_CUDA_SAFE_CALL(hipMemcpy(arrayCuda, arrayCpu, nElements * sizeof(DataType), hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferArrayFromCpuToDevice<int>(int*, const int*, const size_t&);
template int cudaCommon_transferArrayFromCpuToDevice<float>(float*, const float*, const size_t&);
template int cudaCommon_transferArrayFromCpuToDevice<double>(double*, const double*, const size_t&);
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferArrayFromDeviceToCpu(DataType *arrayCpu, const DataType *arrayCuda, const size_t& nElements) {
    NR_CUDA_SAFE_CALL(hipMemcpy(arrayCpu, arrayCuda, nElements * sizeof(DataType), hipMemcpyDeviceToHost));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferArrayFromDeviceToCpu<int>(int*, const int*, const size_t&);
template int cudaCommon_transferArrayFromDeviceToCpu<float>(float*, const float*, const size_t&);
template int cudaCommon_transferArrayFromDeviceToCpu<double>(double*, const double*, const size_t&);
/* *************************************************************** */
void cudaCommon_destroyTextureObject(hipTextureObject_t *texObj) {
    NR_CUDA_SAFE_CALL(hipDestroyTextureObject(*texObj));
    delete texObj;
}
/* *************************************************************** */
UniqueTextureObjectPtr cudaCommon_createTextureObject(const void *devPtr,
                                                      const hipResourceType& resType,
                                                      const size_t& size,
                                                      const hipChannelFormatKind& channelFormat,
                                                      const unsigned& channelCount,
                                                      const hipTextureFilterMode& filterMode,
                                                      const bool& normalizedCoordinates) {
    // Specify texture
    hipResourceDesc resDesc{};
    resDesc.resType = resType;
    switch (resType) {
    case hipResourceTypeLinear:
        resDesc.res.linear.devPtr = const_cast<void*>(devPtr);
        resDesc.res.linear.desc.f = channelFormat;
        resDesc.res.linear.desc.x = 32;
        if (channelCount > 1)
            resDesc.res.linear.desc.y = 32;
        if (channelCount > 2)
            resDesc.res.linear.desc.z = 32;
        if (channelCount > 3)
            resDesc.res.linear.desc.w = 32;
        resDesc.res.linear.sizeInBytes = size;
        break;
    case hipResourceTypeArray:
        resDesc.res.array.array = static_cast<hipArray*>(const_cast<void*>(devPtr));
        break;
    default:
        reg_print_fct_error("cudaCommon_createTextureObject");
        reg_print_msg_error("Unsupported resource type");
        reg_exit();
    }

    // Specify texture object parameters
    hipTextureDesc texDesc{};
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.addressMode[2] = hipAddressModeWrap;
    texDesc.filterMode = filterMode;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = normalizedCoordinates;

    // Create texture object
    UniqueTextureObjectPtr texObj(new hipTextureObject_t(), cudaCommon_destroyTextureObject);
    NR_CUDA_SAFE_CALL(hipCreateTextureObject(texObj.get(), &resDesc, &texDesc, nullptr));

    return texObj;
}
/* *************************************************************** */
