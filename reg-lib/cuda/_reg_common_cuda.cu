#include "hip/hip_runtime.h"
/**
 * @file _reg_comon_gpu.cu
 * @author Marc Modat
 * @date 25/03/2009
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 * See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _REG_COMMON_GPU_CU
#define _REG_COMMON_GPU_CU

#include "_reg_common_cuda.h"
#include "_reg_tools.h"
/* ******************************** */
/* ******************************** */
template <class NIFTI_TYPE>
int cudaCommon_transferNiftiToNiftiOnDevice1(nifti_image **image_d, nifti_image *img) {

	const unsigned int memSize = img->dim[1] * img->dim[2] * img->dim[3] * sizeof(NIFTI_TYPE);

	int *g_dim;
	float* g_pixdim;
	NIFTI_TYPE* g_data;

	NR_CUDA_SAFE_CALL(hipMalloc((void**)&g_dim, 8 * sizeof(int)));
	NR_CUDA_SAFE_CALL(hipMalloc((void**)&g_pixdim, 8 * sizeof(float)));
	NR_CUDA_SAFE_CALL(hipMalloc((void**)&g_data, memSize));

	NIFTI_TYPE *array_h = static_cast<NIFTI_TYPE *>( img->data );
	NR_CUDA_SAFE_CALL(hipMemcpy(( *image_d ), img, sizeof(nifti_image), hipMemcpyHostToDevice));

	NR_CUDA_SAFE_CALL(hipMemcpy((*image_d)->data, array_h, memSize, hipMemcpyHostToDevice));
	NR_CUDA_SAFE_CALL(hipMemcpy(( *image_d )->dim, img->dim, 8 * sizeof(int), hipMemcpyHostToDevice));
	NR_CUDA_SAFE_CALL(hipMemcpy(( *image_d )->pixdim, img->pixdim, 8 * sizeof(float), hipMemcpyHostToDevice));

	return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToNiftiOnDevice1<float>(nifti_image **image_d, nifti_image *img);
template int cudaCommon_transferNiftiToNiftiOnDevice1<double>(nifti_image **image_d, nifti_image *img);
/* ******************************** */
/* ******************************** */
template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferNiftiToArrayOnDevice1(DTYPE **array_d, nifti_image *img)
{
	if(sizeof(DTYPE)!=sizeof(NIFTI_TYPE)){
		reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
		reg_print_msg_error("The host and device arrays are of different types");
		return EXIT_FAILURE;
	}
	else{
		const unsigned int memSize = img->dim[1] * img->dim[2] * img->dim[3] * sizeof(DTYPE);
		NIFTI_TYPE *array_h=static_cast<NIFTI_TYPE *>(img->data);
		NR_CUDA_SAFE_CALL(hipMemcpy(*array_d, array_h, memSize, hipMemcpyHostToDevice));
	}
	return EXIT_SUCCESS;
}
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferNiftiToArrayOnDevice(DTYPE **array_d, nifti_image *img)
{
	if( sizeof(DTYPE)==sizeof(float4) ){
		if( (img->datatype!=NIFTI_TYPE_FLOAT32) || (img->dim[5]<2) || (img->dim[4]>1)){
			reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
			reg_print_msg_error("The specified image is not a single precision deformation field image");
			return EXIT_FAILURE;
		}
		float *niftiImgValues = static_cast<float *>(img->data);
		float4 *array_h=(float4 *)calloc(img->nx*img->ny*img->nz,sizeof(float4));
		const int voxelNumber = img->nx*img->ny*img->nz;
		for(int i=0; i<voxelNumber; i++)
			array_h[i].x= *niftiImgValues++;
		if(img->dim[5]>=2){
			for(int i=0; i<voxelNumber; i++)
				array_h[i].y= *niftiImgValues++;
		}
		if(img->dim[5]>=3){
			for(int i=0; i<voxelNumber; i++)
				array_h[i].z= *niftiImgValues++;
		}
		if(img->dim[5]>=4){
			for(int i=0; i<voxelNumber; i++)
				array_h[i].w= *niftiImgValues++;
		}
		NR_CUDA_SAFE_CALL(hipMemcpy(*array_d, array_h, img->nx*img->ny*img->nz*sizeof(float4), hipMemcpyHostToDevice));
		free(array_h);
	}
	else{ // All these else could be removed but the nvcc compiler would warn for unreachable statement
		switch(img->datatype){
		case NIFTI_TYPE_FLOAT32:
			return cudaCommon_transferNiftiToArrayOnDevice1<DTYPE,float>(array_d, img);
		default:
			reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
			reg_print_msg_error("The image data type is not supported");
			return EXIT_FAILURE;
		}
	}
	return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<double>(double **, nifti_image *);
template int cudaCommon_transferNiftiToArrayOnDevice<float>(float **, nifti_image *);
template int cudaCommon_transferNiftiToArrayOnDevice<int>(int **, nifti_image *);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(float4 **, nifti_image *);
/* ******************************** */

template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferNiftiToArrayOnDevice1(DTYPE **array_d, DTYPE **array2_d, nifti_image *img)
{
	if(sizeof(DTYPE)!=sizeof(NIFTI_TYPE)){
		reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
		reg_print_msg_error("The host and device arrays are of different types");
		return EXIT_FAILURE;
	}
	else{
		const unsigned int memSize = img->dim[1] * img->dim[2] * img->dim[3] * sizeof(DTYPE);
		NIFTI_TYPE *array_h=static_cast<NIFTI_TYPE *>(img->data);
		NIFTI_TYPE *array2_h=&array_h[img->dim[1] * img->dim[2] * img->dim[3]];
		NR_CUDA_SAFE_CALL(hipMemcpy(*array_d, array_h, memSize, hipMemcpyHostToDevice));
		NR_CUDA_SAFE_CALL(hipMemcpy(*array2_d, array2_h, memSize, hipMemcpyHostToDevice));
	}
	return EXIT_SUCCESS;
}
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferNiftiToArrayOnDevice(DTYPE **array_d, DTYPE **array2_d, nifti_image *img)
{
	if(sizeof(DTYPE)==sizeof(float4) ){
		if( (img->datatype!=NIFTI_TYPE_FLOAT32) || (img->dim[5]<2) || (img->dim[4]>1)){
			reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
			reg_print_msg_error("The specified image is not a single precision deformation field image");
			return EXIT_FAILURE;
		}
		float *niftiImgValues = static_cast<float *>(img->data);
		float4 *array_h=(float4 *)calloc(img->nx*img->ny*img->nz,sizeof(float4));
		float4 *array2_h=(float4 *)calloc(img->nx*img->ny*img->nz,sizeof(float4));
		const int voxelNumber = img->nx*img->ny*img->nz;
		for(int i=0; i<voxelNumber; i++)
			array_h[i].x= *niftiImgValues++;
		for(int i=0; i<voxelNumber; i++)
			array2_h[i].x= *niftiImgValues++;
		if(img->dim[5]>=2){
			for(int i=0; i<voxelNumber; i++)
				array_h[i].y= *niftiImgValues++;
			for(int i=0; i<voxelNumber; i++)
				array2_h[i].y= *niftiImgValues++;
		}
		if(img->dim[5]>=3){
			for(int i=0; i<voxelNumber; i++)
				array_h[i].z= *niftiImgValues++;
			for(int i=0; i<voxelNumber; i++)
				array2_h[i].z= *niftiImgValues++;
		}
		if(img->dim[5]>=4){
			for(int i=0; i<voxelNumber; i++)
				array_h[i].w= *niftiImgValues++;
			for(int i=0; i<voxelNumber; i++)
				array2_h[i].w= *niftiImgValues++;
		}
		NR_CUDA_SAFE_CALL(hipMemcpy(*array_d, array_h, img->nx*img->ny*img->nz*sizeof(float4), hipMemcpyHostToDevice));
		NR_CUDA_SAFE_CALL(hipMemcpy(*array2_d, array2_h, img->nx*img->ny*img->nz*sizeof(float4), hipMemcpyHostToDevice));
		free(array_h);
		free(array2_h);
	}
	else{ // All these else could be removed but the nvcc compiler would warn for unreachable statement
		switch(img->datatype){
		case NIFTI_TYPE_FLOAT32:
			return cudaCommon_transferNiftiToArrayOnDevice1<DTYPE,float>(array_d, array2_d, img);
		default:
			reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
			reg_print_msg_error("The image data type is not supported");
			return EXIT_FAILURE;
		}
	}
	return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<float>(float **,float **, nifti_image *);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(double **,double **, nifti_image *);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(float4 **,float4 **, nifti_image *); // for deformation field
/* ******************************** */
/* ******************************** */
template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferNiftiToArrayOnDevice1(hipArray **cuArray_d, nifti_image *img)
{
	if(sizeof(DTYPE)!=sizeof(NIFTI_TYPE)){
		reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
		reg_print_msg_error("The host and device arrays are of different types");
		return EXIT_FAILURE;
	}
	else{
		NIFTI_TYPE *array_h=static_cast<NIFTI_TYPE *>(img->data);

		hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
		copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
		copyParams.srcPtr = make_hipPitchedPtr((void *) array_h,
												copyParams.extent.width*sizeof(DTYPE),
												copyParams.extent.width,
												copyParams.extent.height);
		copyParams.dstArray = *cuArray_d;
		copyParams.kind = hipMemcpyHostToDevice;
		NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
	}
	return EXIT_SUCCESS;
}
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferNiftiToArrayOnDevice(hipArray **cuArray_d, nifti_image *img)
{
	if( sizeof(DTYPE)==sizeof(float4) ){
		if( (img->datatype!=NIFTI_TYPE_FLOAT32) || (img->dim[5]<2) || (img->dim[4]>1) ){
			reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
			reg_print_msg_error("The specified image is not a single precision deformation field image");
			return EXIT_FAILURE;
		}
		float *niftiImgValues = static_cast<float *>(img->data);
		float4 *array_h=(float4 *)calloc(img->nx*img->ny*img->nz,sizeof(float4));

		for(int i=0; i<img->nx*img->ny*img->nz; i++)
			array_h[i].x= *niftiImgValues++;
		if(img->dim[5]>=2)
		{
			for(int i=0; i<img->nx*img->ny*img->nz; i++)
				array_h[i].y= *niftiImgValues++;
		}
		if(img->dim[5]>=3)
		{
			for(int i=0; i<img->nx*img->ny*img->nz; i++)
				array_h[i].z= *niftiImgValues++;
		}
		if(img->dim[5]==3)
		{
			for(int i=0; i<img->nx*img->ny*img->nz; i++)
				array_h[i].w= *niftiImgValues++;
		}
		hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
		copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
		copyParams.srcPtr = make_hipPitchedPtr((void *) array_h,
												copyParams.extent.width*sizeof(DTYPE),
												copyParams.extent.width,
												copyParams.extent.height);
		copyParams.dstArray = *cuArray_d;
		copyParams.kind = hipMemcpyHostToDevice;
		NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams))
		free(array_h);
	}
	else{ // All these else could be removed but the nvcc compiler would warn for unreachable statement
		switch(img->datatype){
		case NIFTI_TYPE_FLOAT32:
			return cudaCommon_transferNiftiToArrayOnDevice1<DTYPE,float>(cuArray_d, img);
		default:
			reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
			reg_print_msg_error("The image data type is not supported");
			return EXIT_FAILURE;
		}
	}
	return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<float>(hipArray **, nifti_image *);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(hipArray **, nifti_image *);
template int cudaCommon_transferNiftiToArrayOnDevice<int>(hipArray **, nifti_image *);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(hipArray **, nifti_image *); // for deformation field
/* ******************************** */
/* ******************************** */
template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferNiftiToArrayOnDevice1(hipArray **cuArray_d, hipArray **cuArray2_d, nifti_image *img)
{
	if(sizeof(DTYPE)!=sizeof(NIFTI_TYPE)){
		reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
		reg_print_msg_error("The host and device arrays are of different types");
		return EXIT_FAILURE;
	}
	else{
		NIFTI_TYPE *array_h = static_cast<NIFTI_TYPE *>(img->data);
		NIFTI_TYPE *array2_h = &array_h[img->dim[1]*img->dim[2]*img->dim[3]];

		hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
		copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
		copyParams.kind = hipMemcpyHostToDevice;
		// First timepoint
		copyParams.srcPtr = make_hipPitchedPtr((void *) array_h,
												copyParams.extent.width*sizeof(DTYPE),
												copyParams.extent.width,
												copyParams.extent.height);
		copyParams.dstArray = *cuArray_d;
		NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
		// Second timepoint
		copyParams.srcPtr = make_hipPitchedPtr((void *) array2_h,
												copyParams.extent.width*sizeof(DTYPE),
												copyParams.extent.width,
												copyParams.extent.height);
		copyParams.dstArray = *cuArray2_d;
		NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
	}
	return EXIT_SUCCESS;
}
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferNiftiToArrayOnDevice(hipArray **cuArray_d, hipArray **cuArray2_d, nifti_image *img)
{
	if( sizeof(DTYPE)==sizeof(float4) ){
		if( (img->datatype!=NIFTI_TYPE_FLOAT32) || (img->dim[5]<2) || (img->dim[4]>1) )
		{
			reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
			reg_print_msg_error("The specified image is not a single precision deformation field image");
			return EXIT_FAILURE;
		}
		float *niftiImgValues = static_cast<float *>(img->data);
		float4 *array_h=(float4 *)calloc(img->nx*img->ny*img->nz,sizeof(float4));
		float4 *array2_h=(float4 *)calloc(img->nx*img->ny*img->nz,sizeof(float4));

		for(int i=0; i<img->nx*img->ny*img->nz; i++)
			array_h[i].x= *niftiImgValues++;
		for(int i=0; i<img->nx*img->ny*img->nz; i++)
			array2_h[i].x= *niftiImgValues++;

		if(img->dim[5]>=2){
			for(int i=0; i<img->nx*img->ny*img->nz; i++)
				array_h[i].y= *niftiImgValues++;
			for(int i=0; i<img->nx*img->ny*img->nz; i++)
				array2_h[i].y= *niftiImgValues++;
		}

		if(img->dim[5]>=3){
			for(int i=0; i<img->nx*img->ny*img->nz; i++)
				array_h[i].z= *niftiImgValues++;
			for(int i=0; i<img->nx*img->ny*img->nz; i++)
				array2_h[i].z= *niftiImgValues++;
		}

		if(img->dim[5]==3){
			for(int i=0; i<img->nx*img->ny*img->nz; i++)
				array_h[i].w= *niftiImgValues++;
			for(int i=0; i<img->nx*img->ny*img->nz; i++)
				array2_h[i].w= *niftiImgValues++;
		}

		hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
		copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
		copyParams.kind = hipMemcpyHostToDevice;
		// First timepoint
		copyParams.srcPtr = make_hipPitchedPtr((void *) array_h,
												copyParams.extent.width*sizeof(DTYPE),
												copyParams.extent.width,
												copyParams.extent.height);
		copyParams.dstArray = *cuArray_d;
		NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
		free(array_h);
		// Second timepoint
		copyParams.srcPtr = make_hipPitchedPtr((void *) array2_h,
												copyParams.extent.width*sizeof(DTYPE),
												copyParams.extent.width,
												copyParams.extent.height);
		copyParams.dstArray = *cuArray2_d;
		NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
		free(array2_h);
	}
	else{ // All these else could be removed but the nvcc compiler would warn for unreachable statement
		switch(img->datatype){
		case NIFTI_TYPE_FLOAT32:
			return cudaCommon_transferNiftiToArrayOnDevice1<DTYPE,float>(cuArray_d, cuArray2_d, img);
		default:
			reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
			reg_print_msg_error("The image data type is not supported");
			return EXIT_FAILURE;
		}
	}
	return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<float>(hipArray **, hipArray **, nifti_image *);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(hipArray **, hipArray **, nifti_image *);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(hipArray **, hipArray **, nifti_image *); // for deformation field
/* ******************************** */
/* ******************************** */
template <class DTYPE>
int cudaCommon_allocateArrayToDevice(hipArray **cuArray_d, int *dim)
{
	const hipExtent volumeSize = make_hipExtent(dim[1], dim[2], dim[3]);
	hipChannelFormatDesc texDesc = hipCreateChannelDesc<DTYPE>();
	NR_CUDA_SAFE_CALL(hipMalloc3DArray(cuArray_d, &texDesc, volumeSize));
	return EXIT_SUCCESS;
}template int cudaCommon_allocateArrayToDevice<float>(hipArray **, int *);
template int cudaCommon_allocateArrayToDevice<double>(hipArray **, int *);
template int cudaCommon_allocateArrayToDevice<float4>(hipArray **, int *); // for deformation field
/* ******************************** */
/* ******************************** */
template <class DTYPE>
int cudaCommon_allocateArrayToDevice(hipArray **cuArray_d, hipArray **cuArray2_d, int *dim)
{
	const hipExtent volumeSize = make_hipExtent(dim[1], dim[2], dim[3]);
	hipChannelFormatDesc texDesc = hipCreateChannelDesc<DTYPE>();
	NR_CUDA_SAFE_CALL(hipMalloc3DArray(cuArray_d, &texDesc, volumeSize));
	NR_CUDA_SAFE_CALL(hipMalloc3DArray(cuArray2_d, &texDesc, volumeSize));
	return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(hipArray **,hipArray **, int *);
template int cudaCommon_allocateArrayToDevice<double>(hipArray **,hipArray **, int *);
template int cudaCommon_allocateArrayToDevice<float4>(hipArray **,hipArray **, int *); // for deformation field
/* ******************************** */
/* ******************************** */
template <class DTYPE>
int cudaCommon_allocateArrayToDevice(DTYPE **array_d, int *dim)
{
	const unsigned int memSize = dim[1] * dim[2] * dim[3] * sizeof(DTYPE);
	NR_CUDA_SAFE_CALL(hipMalloc(array_d, memSize));
	return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float **, int *);
template int cudaCommon_allocateArrayToDevice<double>(double **, int *);
template int cudaCommon_allocateArrayToDevice<int>(int **, int *);
template int cudaCommon_allocateArrayToDevice<float4>(float4 **, int *); // for deformation field
/* ******************************** */
template <class DTYPE>
int cudaCommon_allocateArrayToDevice(DTYPE **array_d, int vox)
{
	const unsigned int memSize = vox * sizeof(DTYPE);
	NR_CUDA_SAFE_CALL(hipMalloc(array_d, memSize));
	return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float **, int);
template int cudaCommon_allocateArrayToDevice<double>(double **, int);
template int cudaCommon_allocateArrayToDevice<int>(int **, int);
template int cudaCommon_allocateArrayToDevice<float4>(float4 **, int); // for deformation field
/* ******************************** */
/* ******************************** */
template <class DTYPE>
int cudaCommon_allocateArrayToDevice(DTYPE **array_d, DTYPE **array2_d, int *dim)
{
	const unsigned int memSize = dim[1] * dim[2] * dim[3] * sizeof(DTYPE);
	NR_CUDA_SAFE_CALL(hipMalloc(array_d, memSize));
	NR_CUDA_SAFE_CALL(hipMalloc(array2_d, memSize));
	return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float **, float **, int *);
template int cudaCommon_allocateArrayToDevice<double>(double **, double **, int *);
template int  cudaCommon_allocateArrayToDevice<float4>(float4 **, float4 **, int *); // for deformation field
/* ******************************** */
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferFromDeviceToCpu(DTYPE *cpuPtr, DTYPE **cuPtr, const unsigned int nElements)
{

	NR_CUDA_SAFE_CALL(hipMemcpy((void *)cpuPtr, (void *)*cuPtr, nElements*sizeof(DTYPE), hipMemcpyDeviceToHost));
	//NR_CUDA_SAFE_CALL(hipDeviceSynchronize());
	return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToCpu<float>(float *cpuPtr, float **cuPtr, const unsigned int nElements);
template int cudaCommon_transferFromDeviceToCpu<double>(double *cpuPtr, double **cuPtr, const unsigned int nElements);

/* ******************************** */
/* ******************************** */
/* ******************************** */
template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferFromDeviceToNifti1(nifti_image *img, DTYPE **array_d)
{
	if(sizeof(DTYPE)!=sizeof(NIFTI_TYPE)){
		reg_print_fct_error("cudaCommon_transferFromDeviceToNifti1");
		reg_print_msg_error("The host and device arrays are of different types");
		return EXIT_FAILURE;
	}
	else
	{
		NIFTI_TYPE *array_h=static_cast<NIFTI_TYPE *>(img->data);
		NR_CUDA_SAFE_CALL(hipMemcpy((void *)array_h, (void *)*array_d, img->nvox*sizeof(DTYPE), hipMemcpyDeviceToHost));
	}
	return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToNifti1<float, float>(nifti_image *img, float **array_d);
template int cudaCommon_transferFromDeviceToNifti1<double, double>(nifti_image *img, double **array_d);
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferFromDeviceToNifti(nifti_image *img, DTYPE **array_d)
{
	if(sizeof(DTYPE)==sizeof(float4)){
		// A nifti 5D volume is expected
		if(img->dim[0]<5 || img->dim[4]>1 || img->dim[5]<2 || img->datatype!=NIFTI_TYPE_FLOAT32){
			reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
			reg_print_msg_error("The nifti image is not a 5D volume");
			return EXIT_FAILURE;
		}
		const int voxelNumber = img->nx*img->ny*img->nz;

		float4 *array_h;
		NR_CUDA_SAFE_CALL(hipHostMalloc(&array_h, voxelNumber*sizeof(float4)));
		NR_CUDA_SAFE_CALL(hipMemcpy((void *)array_h, (const void *)*array_d, voxelNumber*sizeof(float4), hipMemcpyDeviceToHost));
		float *niftiImgValues = static_cast<float *>(img->data);

		for(int i=0; i<voxelNumber; i++)
			*niftiImgValues++ = array_h[i].x;
		if(img->dim[5]>=2){
			for(int i=0; i<voxelNumber; i++)
				*niftiImgValues++ = array_h[i].y;
		}
		if(img->dim[5]>=3){
			for(int i=0; i<voxelNumber; i++)
				*niftiImgValues++ = array_h[i].z;
		}
		if(img->dim[5]>=4){
			for(int i=0; i<voxelNumber; i++)
				*niftiImgValues++ = array_h[i].w;
		}
		NR_CUDA_SAFE_CALL(hipHostFree(array_h));

		return EXIT_SUCCESS;
	}
	else{
		switch(img->datatype){
		case NIFTI_TYPE_FLOAT32:
			return cudaCommon_transferFromDeviceToNifti1<DTYPE,float>(img, array_d);
		default:
			reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
			reg_print_msg_error("The image data type is not supported");
			return EXIT_FAILURE;
		}
	}
}
template int cudaCommon_transferFromDeviceToNifti<float>(nifti_image *, float **);
template int cudaCommon_transferFromDeviceToNifti<double>(nifti_image *, double **);
template int cudaCommon_transferFromDeviceToNifti<float4>(nifti_image *, float4 **); // for deformation field
/* ******************************** */
/* ******************************** */
template <class DTYPE, class NIFTI_TYPE>
int cudaCommon_transferFromDeviceToNifti1(nifti_image *img, DTYPE **array_d, DTYPE **array2_d)
{
	if(sizeof(DTYPE)!=sizeof(NIFTI_TYPE)){
		reg_print_fct_error("cudaCommon_transferFromDeviceToNifti1");
		reg_print_msg_error("The host and device arrays are of different types");
		return EXIT_FAILURE;
	}
	else{
		unsigned int voxelNumber=img->nx*img->ny*img->nz;
		NIFTI_TYPE *array_h=static_cast<NIFTI_TYPE *>(img->data);
		NIFTI_TYPE *array2_h=&array_h[voxelNumber];
		NR_CUDA_SAFE_CALL(hipMemcpy((void *)array_h, (void *)*array_d, voxelNumber*sizeof(DTYPE), hipMemcpyDeviceToHost));
		NR_CUDA_SAFE_CALL(hipMemcpy((void *)array2_h, (void *)*array2_d, voxelNumber*sizeof(DTYPE), hipMemcpyDeviceToHost));
	}
	return EXIT_SUCCESS;
}
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferFromDeviceToNifti(nifti_image *img, DTYPE **array_d, DTYPE **array2_d)
{
	if(sizeof(DTYPE)==sizeof(float4)){
		// A nifti 5D volume is expected
		if(img->dim[0]<5 || img->dim[4]>1 || img->dim[5]<2 || img->datatype!=NIFTI_TYPE_FLOAT32){
			reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
			reg_print_msg_error("The nifti image is not a 5D volume");
			return EXIT_FAILURE;
		}
		const int voxelNumber = img->nx*img->ny*img->nz;
		float4 *array_h=NULL;
		float4 *array2_h=NULL;
		NR_CUDA_SAFE_CALL(hipHostMalloc(&array_h, voxelNumber*sizeof(float4)));
		NR_CUDA_SAFE_CALL(hipHostMalloc(&array2_h, voxelNumber*sizeof(float4)));
		NR_CUDA_SAFE_CALL(hipMemcpy((void *)array_h, (const void *)*array_d, voxelNumber*sizeof(float4), hipMemcpyDeviceToHost));
		NR_CUDA_SAFE_CALL(hipMemcpy((void *)array2_h, (const void *)*array2_d, voxelNumber*sizeof(float4), hipMemcpyDeviceToHost));
		float *niftiImgValues = static_cast<float *>(img->data);
		for(int i=0; i<voxelNumber; i++){
			*niftiImgValues++ = array_h[i].x;
		}
		for(int i=0; i<voxelNumber; i++){
			*niftiImgValues++ = array2_h[i].x;
		}
		if(img->dim[5]>=2){
			for(int i=0; i<voxelNumber; i++){
				*niftiImgValues++ = array_h[i].y;
			}
			for(int i=0; i<voxelNumber; i++){
				*niftiImgValues++ = array2_h[i].y;
			}
		}
		if(img->dim[5]>=3){
			for(int i=0; i<voxelNumber; i++){
				*niftiImgValues++ = array_h[i].z;
			}
			for(int i=0; i<voxelNumber; i++){
				*niftiImgValues++ = array2_h[i].z;
			}
		}
		if(img->dim[5]>=4){
			for(int i=0; i<voxelNumber; i++){
				*niftiImgValues++ = array_h[i].w;
			}
			for(int i=0; i<voxelNumber; i++){
				*niftiImgValues++ = array2_h[i].w;
			}
		}
		NR_CUDA_SAFE_CALL(hipHostFree(array_h));
		NR_CUDA_SAFE_CALL(hipHostFree(array2_h));

		return EXIT_SUCCESS;
	}
	else{
		switch(img->datatype){
		case NIFTI_TYPE_FLOAT32:
			return cudaCommon_transferFromDeviceToNifti1<DTYPE,float>(img, array_d, array2_d);
		default:
			reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
			reg_print_msg_error("The image data type is not supported");
			return EXIT_FAILURE;
		}
	}
}
template int cudaCommon_transferFromDeviceToNifti<float>(nifti_image *, float **, float **);
template int cudaCommon_transferFromDeviceToNifti<double>(nifti_image *, double **, double **);
template int cudaCommon_transferFromDeviceToNifti<float4>(nifti_image *, float4 **, float4 **); // for deformation field
/* ******************************** */
/* ******************************** */
void cudaCommon_free(hipArray **cuArray_d)
{
		NR_CUDA_SAFE_CALL(hipFreeArray(*cuArray_d));
	return;
}
/* ******************************** */
/* ******************************** */
template <class DTYPE>
void cudaCommon_free(DTYPE **array_d)
{
	NR_CUDA_SAFE_CALL(hipFree(*array_d));
	return;
}
template void cudaCommon_free<int>(int **);
template void cudaCommon_free<float>(float **);
template void cudaCommon_free<double>(double **);
template void cudaCommon_free<float4>(float4 **);
/* ******************************** */
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferFromDeviceToNiftiSimple(DTYPE **array_d, nifti_image *img)
{
	NR_CUDA_SAFE_CALL(hipMemcpy(*array_d, img->data, img->nvox * sizeof(DTYPE), hipMemcpyHostToDevice));

	return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToNiftiSimple<int>(int **array_d, nifti_image *img);
template int cudaCommon_transferFromDeviceToNiftiSimple<float>(float **array_d, nifti_image *img);
template int cudaCommon_transferFromDeviceToNiftiSimple<double>(double **array_d, nifti_image *img);
/* ******************************** */
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferFromDeviceToNiftiSimple1(DTYPE **array_d, DTYPE *img, const unsigned int nvox)
{
	NR_CUDA_SAFE_CALL(hipMemcpy(*array_d, img, nvox * sizeof(DTYPE), hipMemcpyHostToDevice));
	return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToNiftiSimple1<int>(int **array_d, int *img, const unsigned);
template int cudaCommon_transferFromDeviceToNiftiSimple1<float>(float **array_d, float *img, const unsigned);
template int cudaCommon_transferFromDeviceToNiftiSimple1<double>(double **array_d, double *img, const unsigned);
/* ******************************** */
/* ******************************** */
/* ******************************** */
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferArrayFromCpuToDevice(DTYPE **array_d, DTYPE *array_cpu, const unsigned int nElements) {

    const unsigned int memSize = nElements * sizeof(DTYPE);
    //copyData
    NR_CUDA_SAFE_CALL(hipMemcpy(*array_d, array_cpu, memSize, hipMemcpyHostToDevice));
    //
    return EXIT_SUCCESS;
}
template int cudaCommon_transferArrayFromCpuToDevice<int>(int **array_d, int *array_cpu, const unsigned int nElements);
template int cudaCommon_transferArrayFromCpuToDevice<float>(float **array_d, float *array_cpu, const unsigned int nElements);
template int cudaCommon_transferArrayFromCpuToDevice<double>(double **array_d, double *array_cpu, const unsigned int nElements);
/* ******************************** */
/* ******************************** */
/* ******************************** */
/* ******************************** */
template <class DTYPE>
int cudaCommon_transferArrayFromDeviceToCpu(DTYPE *array_cpu, DTYPE **array_d, const unsigned int nElements) {

    const unsigned int memSize = nElements * sizeof(DTYPE);
    //copyData
    NR_CUDA_SAFE_CALL(hipMemcpy(array_cpu, *array_d, memSize, hipMemcpyDeviceToHost));
    //
    return EXIT_SUCCESS;
}
template int cudaCommon_transferArrayFromDeviceToCpu<int>(int *array_cpu, int **array_d, const unsigned int nElements);
template int cudaCommon_transferArrayFromDeviceToCpu<float>(float *array_cpu, float **array_d, const unsigned int nElements);
template int cudaCommon_transferArrayFromDeviceToCpu<double>(double *array_cpu, double **array_d, const unsigned int nElements);
/* ******************************** */
void showCUDACardInfo(void)
{
   // The CUDA card is setup
   hipInit(0);

   int device_count=0;
   hipGetDeviceCount(&device_count);
   printf("-----------------------------------\n");
   printf("[NiftyReg CUDA] %i device(s) detected\n", device_count);
   printf("-----------------------------------\n");

   hipCtx_t cucontext;

   struct hipDeviceProp_t deviceProp;
   // following code is from cutGetMaxGflopsDeviceId()
   int current_device = 0;
   while(current_device<device_count){
       hipGetDeviceProperties(&deviceProp, current_device);
       if(deviceProp.major>0){

          NR_CUDA_SAFE_CALL(hipSetDevice(current_device));
          NR_CUDA_SAFE_CALL(hipCtxCreate(&cucontext, hipDeviceScheduleSpin, current_device));

          printf("[NiftyReg CUDA] Device id [%i]\n", current_device);
          printf("[NiftyReg CUDA] Device name: %s\n", deviceProp.name);
          size_t free=0;
          size_t total=0;
          hipMemGetInfo(&free, &total);
          printf("[NiftyReg CUDA] It has %lu Mb free out of %lu Mb\n",
                 (unsigned long int)(free/(1024*1024)),
                 (unsigned long int)(total/(1024*1024)));
          printf("[NiftyReg CUDA] Card compute capability: %i.%i\n",
                 deviceProp.major,
                 deviceProp.minor);
          printf("[NiftyReg CUDA] Shared memory size in bytes: %zu\n",
                 deviceProp.sharedMemPerBlock);
          printf("[NiftyReg CUDA] CUDA version %i\n",
                 CUDART_VERSION);
          printf("[NiftyReg CUDA] Card clock rate (Mhz): %i\n",
                 deviceProp.clockRate/1000);
          printf("[NiftyReg CUDA] Card has %i multiprocessor(s)\n",
                 deviceProp.multiProcessorCount);
       }
       hipCtxDestroy(cucontext);
       ++current_device;
       printf("-----------------------------------\n");
   }
}
#endif
/* ******************************** */
/* ******************************** */
