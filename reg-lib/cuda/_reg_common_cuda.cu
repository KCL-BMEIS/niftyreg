#include "hip/hip_runtime.h"
/**
 * @file _reg_common_cuda.cu
 * @author Marc Modat
 * @date 25/03/2009
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 * See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#include "_reg_common_cuda.h"

/* *************************************************************** */
template <class NiftiType>
int cudaCommon_transferNiftiToNiftiOnDevice1(nifti_image *image_d, nifti_image *img) {
    const unsigned memSize = img->dim[1] * img->dim[2] * img->dim[3] * sizeof(NiftiType);

    int *g_dim;
    float* g_pixdim;
    NiftiType* g_data;

    NR_CUDA_SAFE_CALL(hipMalloc((void**)&g_dim, 8 * sizeof(int)));
    NR_CUDA_SAFE_CALL(hipMalloc((void**)&g_pixdim, 8 * sizeof(float)));
    NR_CUDA_SAFE_CALL(hipMalloc((void**)&g_data, memSize));

    NiftiType *array_h = static_cast<NiftiType*>(img->data);
    NR_CUDA_SAFE_CALL(hipMemcpy(image_d, img, sizeof(nifti_image), hipMemcpyHostToDevice));

    NR_CUDA_SAFE_CALL(hipMemcpy(image_d->data, array_h, memSize, hipMemcpyHostToDevice));
    NR_CUDA_SAFE_CALL(hipMemcpy(image_d->dim, img->dim, 8 * sizeof(int), hipMemcpyHostToDevice));
    NR_CUDA_SAFE_CALL(hipMemcpy(image_d->pixdim, img->pixdim, 8 * sizeof(float), hipMemcpyHostToDevice));

    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToNiftiOnDevice1<float>(nifti_image*, nifti_image*);
template int cudaCommon_transferNiftiToNiftiOnDevice1<double>(nifti_image*, nifti_image*);
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferNiftiToArrayOnDevice1(DataType *array_d, nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        const unsigned memSize = img->dim[1] * img->dim[2] * img->dim[3] * sizeof(DataType);
        NiftiType *array_h = static_cast<NiftiType*>(img->data);
        NR_CUDA_SAFE_CALL(hipMemcpy(array_d, array_h, memSize, hipMemcpyHostToDevice));
    }
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferNiftiToArrayOnDevice(DataType *array_d, nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = CalcVoxelNumber(*img);
        float4 *array_h = (float4*)calloc(voxelNumber, sizeof(float4));
        for (size_t i = 0; i < voxelNumber; i++)
            array_h[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].z = *niftiImgValues++;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].w = *niftiImgValues++;
        }
        NR_CUDA_SAFE_CALL(hipMemcpy(array_d, array_h, voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
        free(array_h);
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DataType, float>(array_d, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<double>(double*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float>(float*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<int>(int*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(float4*, nifti_image*);
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferNiftiToArrayOnDevice1(DataType *array_d, DataType *array2_d, nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        const unsigned memSize = img->dim[1] * img->dim[2] * img->dim[3] * sizeof(DataType);
        NiftiType *array_h = static_cast<NiftiType*>(img->data);
        NiftiType *array2_h = &array_h[img->dim[1] * img->dim[2] * img->dim[3]];
        NR_CUDA_SAFE_CALL(hipMemcpy(array_d, array_h, memSize, hipMemcpyHostToDevice));
        NR_CUDA_SAFE_CALL(hipMemcpy(array2_d, array2_h, memSize, hipMemcpyHostToDevice));
    }
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferNiftiToArrayOnDevice(DataType *array_d, DataType *array2_d, nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        float *niftiImgValues = static_cast<float *>(img->data);
        const size_t voxelNumber = CalcVoxelNumber(*img);
        float4 *array_h = (float4*)calloc(voxelNumber, sizeof(float4));
        float4 *array2_h = (float4*)calloc(voxelNumber, sizeof(float4));
        for (size_t i = 0; i < voxelNumber; i++)
            array_h[i].x = *niftiImgValues++;
        for (size_t i = 0; i < voxelNumber; i++)
            array2_h[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].y = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].z = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].z = *niftiImgValues++;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].w = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].w = *niftiImgValues++;
        }
        NR_CUDA_SAFE_CALL(hipMemcpy(array_d, array_h, voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
        NR_CUDA_SAFE_CALL(hipMemcpy(array2_d, array2_h, voxelNumber * sizeof(float4), hipMemcpyHostToDevice));
        free(array_h);
        free(array2_h);
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DataType, float>(array_d, array2_d, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<float>(float*, float*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(double*, double*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(float4*, float4*, nifti_image*); // for deformation field
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferNiftiToArrayOnDevice1(hipArray *cuArray_d, nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        NiftiType *array_h = static_cast<NiftiType*>(img->data);

        hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
        copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
        copyParams.srcPtr = make_hipPitchedPtr((void*)array_h,
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray_d;
        copyParams.kind = hipMemcpyHostToDevice;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    }
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferNiftiToArrayOnDevice(hipArray *cuArray_d, nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        float *niftiImgValues = static_cast<float *>(img->data);
        const size_t voxelNumber = CalcVoxelNumber(*img);
        float4 *array_h = (float4*)calloc(voxelNumber, sizeof(float4));

        for (size_t i = 0; i < voxelNumber; i++)
            array_h[i].x = *niftiImgValues++;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].y = *niftiImgValues++;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].z = *niftiImgValues++;
        }
        if (img->dim[5] == 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].w = *niftiImgValues++;
        }
        hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
        copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
        copyParams.srcPtr = make_hipPitchedPtr((void*)array_h,
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray_d;
        copyParams.kind = hipMemcpyHostToDevice;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
        free(array_h);
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DataType, float>(cuArray_d, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<int>(hipArray*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float>(hipArray*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(hipArray*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(hipArray*, nifti_image*); // for deformation field
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferNiftiToArrayOnDevice1(hipArray *cuArray_d, hipArray *cuArray2_d, nifti_image *img) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        NiftiType *array_h = static_cast<NiftiType*>(img->data);
        NiftiType *array2_h = &array_h[img->dim[1] * img->dim[2] * img->dim[3]];

        hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
        copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
        copyParams.kind = hipMemcpyHostToDevice;
        // First timepoint
        copyParams.srcPtr = make_hipPitchedPtr((void*)array_h,
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray_d;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
        // Second timepoint
        copyParams.srcPtr = make_hipPitchedPtr((void*)array2_h,
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray2_d;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    }
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferNiftiToArrayOnDevice(hipArray *cuArray_d, hipArray *cuArray2_d, nifti_image *img) {
    if (sizeof(DataType) == sizeof(float4)) {
        if ((img->datatype != NIFTI_TYPE_FLOAT32) || (img->dim[5] < 2) || (img->dim[4] > 1)) {
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
            reg_print_msg_error("The specified image is not a single precision deformation field image");
            return EXIT_FAILURE;
        }
        float *niftiImgValues = static_cast<float*>(img->data);
        const size_t voxelNumber = CalcVoxelNumber(*img);
        float4 *array_h = (float4*)calloc(voxelNumber, sizeof(float4));
        float4 *array2_h = (float4*)calloc(voxelNumber, sizeof(float4));

        for (size_t i = 0; i < voxelNumber; i++)
            array_h[i].x = *niftiImgValues++;
        for (size_t i = 0; i < voxelNumber; i++)
            array2_h[i].x = *niftiImgValues++;

        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].y = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].y = *niftiImgValues++;
        }

        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].z = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].z = *niftiImgValues++;
        }

        if (img->dim[5] == 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                array_h[i].w = *niftiImgValues++;
            for (size_t i = 0; i < voxelNumber; i++)
                array2_h[i].w = *niftiImgValues++;
        }

        hipMemcpy3DParms copyParams; memset(&copyParams, 0, sizeof(copyParams));
        copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
        copyParams.kind = hipMemcpyHostToDevice;
        // First timepoint
        copyParams.srcPtr = make_hipPitchedPtr((void*)array_h,
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray_d;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
        free(array_h);
        // Second timepoint
        copyParams.srcPtr = make_hipPitchedPtr((void*)array2_h,
                                                copyParams.extent.width * sizeof(DataType),
                                                copyParams.extent.width,
                                                copyParams.extent.height);
        copyParams.dstArray = cuArray2_d;
        NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
        free(array2_h);
    } else { // All these else could be removed but the nvcc compiler would warn for unreachable statement
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferNiftiToArrayOnDevice1<DataType, float>(cuArray_d, cuArray2_d, img);
        default:
            reg_print_fct_error("cudaCommon_transferNiftiToArrayOnDevice1");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferNiftiToArrayOnDevice<float>(hipArray*, hipArray*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<double>(hipArray*, hipArray*, nifti_image*);
template int cudaCommon_transferNiftiToArrayOnDevice<float4>(hipArray*, hipArray*, nifti_image*); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_allocateArrayToDevice(hipArray **cuArray_d, int *dim) {
    const hipExtent volumeSize = make_hipExtent(dim[1], dim[2], dim[3]);
    hipChannelFormatDesc texDesc = hipCreateChannelDesc<DataType>();
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(cuArray_d, &texDesc, volumeSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(hipArray**, int*);
template int cudaCommon_allocateArrayToDevice<double>(hipArray**, int*);
template int cudaCommon_allocateArrayToDevice<float4>(hipArray**, int*); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_allocateArrayToDevice(hipArray **cuArray_d, hipArray **cuArray2_d, int *dim) {
    const hipExtent volumeSize = make_hipExtent(dim[1], dim[2], dim[3]);
    hipChannelFormatDesc texDesc = hipCreateChannelDesc<DataType>();
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(cuArray_d, &texDesc, volumeSize));
    NR_CUDA_SAFE_CALL(hipMalloc3DArray(cuArray2_d, &texDesc, volumeSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(hipArray**, hipArray**, int*);
template int cudaCommon_allocateArrayToDevice<double>(hipArray**, hipArray**, int*);
template int cudaCommon_allocateArrayToDevice<float4>(hipArray**, hipArray**, int*); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_allocateArrayToDevice(DataType **array_d, int *dim) {
    const unsigned memSize = dim[1] * dim[2] * dim[3] * sizeof(DataType);
    NR_CUDA_SAFE_CALL(hipMalloc(array_d, memSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float**, int*);
template int cudaCommon_allocateArrayToDevice<double>(double**, int*);
template int cudaCommon_allocateArrayToDevice<int>(int**, int*);
template int cudaCommon_allocateArrayToDevice<float4>(float4**, int*); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_allocateArrayToDevice(DataType **array_d, int vox) {
    const unsigned memSize = vox * sizeof(DataType);
    NR_CUDA_SAFE_CALL(hipMalloc(array_d, memSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float**, int);
template int cudaCommon_allocateArrayToDevice<double>(double**, int);
template int cudaCommon_allocateArrayToDevice<int>(int**, int);
template int cudaCommon_allocateArrayToDevice<float4>(float4**, int); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_allocateArrayToDevice(DataType **array_d, DataType **array2_d, int *dim) {
    const unsigned memSize = dim[1] * dim[2] * dim[3] * sizeof(DataType);
    NR_CUDA_SAFE_CALL(hipMalloc(array_d, memSize));
    NR_CUDA_SAFE_CALL(hipMalloc(array2_d, memSize));
    return EXIT_SUCCESS;
}
template int cudaCommon_allocateArrayToDevice<float>(float**, float**, int*);
template int cudaCommon_allocateArrayToDevice<double>(double**, double**, int*);
template int  cudaCommon_allocateArrayToDevice<float4>(float4**, float4**, int*); // for deformation field
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferFromDeviceToCpu(DataType *cpuPtr, DataType *cuPtr, const unsigned nElements) {
    NR_CUDA_SAFE_CALL(hipMemcpy((void*)cpuPtr, (void*)cuPtr, nElements * sizeof(DataType), hipMemcpyDeviceToHost));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToCpu<float>(float *cpuPtr, float *cuPtr, const unsigned nElements);
template int cudaCommon_transferFromDeviceToCpu<double>(double *cpuPtr, double *cuPtr, const unsigned nElements);
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferFromDeviceToNifti1(nifti_image *img, DataType *array_d) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferFromDeviceToNifti1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        NiftiType *array_h = static_cast<NiftiType*>(img->data);
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array_h, (void*)array_d, img->nvox * sizeof(DataType), hipMemcpyDeviceToHost));
    }
    return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToNifti1<float, float>(nifti_image *img, float *array_d);
template int cudaCommon_transferFromDeviceToNifti1<double, double>(nifti_image *img, double *array_d);
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferFromDeviceToNifti(nifti_image *img, DataType *array_d) {
    if (sizeof(DataType) == sizeof(float4)) {
        // A nifti 5D volume is expected
        if (img->dim[0] < 5 || img->dim[4]>1 || img->dim[5] < 2 || img->datatype != NIFTI_TYPE_FLOAT32) {
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The nifti image is not a 5D volume");
            return EXIT_FAILURE;
        }

        float4 *array_h;
        const size_t voxelNumber = CalcVoxelNumber(*img);
        NR_CUDA_SAFE_CALL(hipHostMalloc(&array_h, voxelNumber * sizeof(float4)));
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array_h, (const void*)array_d, voxelNumber * sizeof(float4), hipMemcpyDeviceToHost));
        float *niftiImgValues = static_cast<float*>(img->data);

        for (size_t i = 0; i < voxelNumber; i++)
            *niftiImgValues++ = array_h[i].x;
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array_h[i].y;
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array_h[i].z;
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++)
                *niftiImgValues++ = array_h[i].w;
        }
        NR_CUDA_SAFE_CALL(hipHostFree(array_h));

        return EXIT_SUCCESS;
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferFromDeviceToNifti1<DataType, float>(img, array_d);
        default:
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
}
template int cudaCommon_transferFromDeviceToNifti<float>(nifti_image*, float*);
template int cudaCommon_transferFromDeviceToNifti<double>(nifti_image*, double*);
template int cudaCommon_transferFromDeviceToNifti<float4>(nifti_image*, float4*); // for deformation field
/* *************************************************************** */
template<>
int cudaCommon_transferFromDeviceToNifti(nifti_image *img, hipArray *cuArray_d) {
    if (img->datatype != NIFTI_TYPE_FLOAT32) {
        reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
        reg_print_msg_error("The image data type is not supported");
        return EXIT_FAILURE;
    }

    hipMemcpy3DParms copyParams = {0};
    copyParams.extent = make_hipExtent(img->dim[1], img->dim[2], img->dim[3]);
    copyParams.srcArray = cuArray_d;
    copyParams.dstPtr = make_hipPitchedPtr((void*)(img->data), copyParams.extent.width * sizeof(float),
                                            copyParams.extent.width, copyParams.extent.height);
    copyParams.kind = hipMemcpyDeviceToHost;
    NR_CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType, class NiftiType>
int cudaCommon_transferFromDeviceToNifti1(nifti_image *img, DataType *array_d, DataType *array2_d) {
    if (sizeof(DataType) != sizeof(NiftiType)) {
        reg_print_fct_error("cudaCommon_transferFromDeviceToNifti1");
        reg_print_msg_error("The host and device arrays are of different types");
        return EXIT_FAILURE;
    } else {
        const size_t voxelNumber = CalcVoxelNumber(*img);
        NiftiType *array_h = static_cast<NiftiType*>(img->data);
        NiftiType *array2_h = &array_h[voxelNumber];
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array_h, (void*)array_d, voxelNumber * sizeof(DataType), hipMemcpyDeviceToHost));
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array2_h, (void*)array2_d, voxelNumber * sizeof(DataType), hipMemcpyDeviceToHost));
    }
    return EXIT_SUCCESS;
}
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferFromDeviceToNifti(nifti_image *img, DataType *array_d, DataType *array2_d) {
    if (sizeof(DataType) == sizeof(float4)) {
        // A nifti 5D volume is expected
        if (img->dim[0] < 5 || img->dim[4]>1 || img->dim[5] < 2 || img->datatype != NIFTI_TYPE_FLOAT32) {
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The nifti image is not a 5D volume");
            return EXIT_FAILURE;
        }
        const size_t voxelNumber = CalcVoxelNumber(*img);
        float4 *array_h = nullptr;
        float4 *array2_h = nullptr;
        NR_CUDA_SAFE_CALL(hipHostMalloc(&array_h, voxelNumber * sizeof(float4)));
        NR_CUDA_SAFE_CALL(hipHostMalloc(&array2_h, voxelNumber * sizeof(float4)));
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array_h, (const void*)array_d, voxelNumber * sizeof(float4), hipMemcpyDeviceToHost));
        NR_CUDA_SAFE_CALL(hipMemcpy((void*)array2_h, (const void*)array2_d, voxelNumber * sizeof(float4), hipMemcpyDeviceToHost));
        float *niftiImgValues = static_cast<float *>(img->data);
        for (size_t i = 0; i < voxelNumber; i++) {
            *niftiImgValues++ = array_h[i].x;
        }
        for (size_t i = 0; i < voxelNumber; i++) {
            *niftiImgValues++ = array2_h[i].x;
        }
        if (img->dim[5] >= 2) {
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array_h[i].y;
            }
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array2_h[i].y;
            }
        }
        if (img->dim[5] >= 3) {
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array_h[i].z;
            }
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array2_h[i].z;
            }
        }
        if (img->dim[5] >= 4) {
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array_h[i].w;
            }
            for (size_t i = 0; i < voxelNumber; i++) {
                *niftiImgValues++ = array2_h[i].w;
            }
        }
        NR_CUDA_SAFE_CALL(hipHostFree(array_h));
        NR_CUDA_SAFE_CALL(hipHostFree(array2_h));

        return EXIT_SUCCESS;
    } else {
        switch (img->datatype) {
        case NIFTI_TYPE_FLOAT32:
            return cudaCommon_transferFromDeviceToNifti1<DataType, float>(img, array_d, array2_d);
        default:
            reg_print_fct_error("cudaCommon_transferFromDeviceToNifti");
            reg_print_msg_error("The image data type is not supported");
            return EXIT_FAILURE;
        }
    }
}
template int cudaCommon_transferFromDeviceToNifti<float>(nifti_image*, float*, float*);
template int cudaCommon_transferFromDeviceToNifti<double>(nifti_image*, double*, double*);
template int cudaCommon_transferFromDeviceToNifti<float4>(nifti_image*, float4*, float4*); // for deformation field
/* *************************************************************** */
void cudaCommon_free(hipArray *cuArray_d) {
    NR_CUDA_SAFE_CALL(hipFreeArray(cuArray_d));
}
/* *************************************************************** */
template <class DataType>
void cudaCommon_free(DataType *array_d) {
    if (array_d != nullptr)
        NR_CUDA_SAFE_CALL(hipFree(array_d));
}
template void cudaCommon_free<int>(int*);
template void cudaCommon_free<float>(float*);
template void cudaCommon_free<double>(double*);
template void cudaCommon_free<float4>(float4*);
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferFromDeviceToNiftiSimple(DataType *array_d, nifti_image *img) {
    NR_CUDA_SAFE_CALL(hipMemcpy(array_d, img->data, img->nvox * sizeof(DataType), hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToNiftiSimple<int>(int*, nifti_image*);
template int cudaCommon_transferFromDeviceToNiftiSimple<float>(float*, nifti_image*);
template int cudaCommon_transferFromDeviceToNiftiSimple<double>(double*, nifti_image*);
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferFromDeviceToNiftiSimple1(DataType *array_d, DataType *img, const unsigned nvox) {
    NR_CUDA_SAFE_CALL(hipMemcpy(array_d, img, nvox * sizeof(DataType), hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferFromDeviceToNiftiSimple1<int>(int*, int*, const unsigned);
template int cudaCommon_transferFromDeviceToNiftiSimple1<float>(float*, float*, const unsigned);
template int cudaCommon_transferFromDeviceToNiftiSimple1<double>(double*, double*, const unsigned);
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferArrayFromCpuToDevice(DataType *array_d, DataType *array_cpu, const unsigned nElements) {
    const unsigned memSize = nElements * sizeof(DataType);
    NR_CUDA_SAFE_CALL(hipMemcpy(array_d, array_cpu, memSize, hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferArrayFromCpuToDevice<int>(int*, int*, const unsigned);
template int cudaCommon_transferArrayFromCpuToDevice<float>(float*, float*, const unsigned);
template int cudaCommon_transferArrayFromCpuToDevice<double>(double*, double*, const unsigned);
/* *************************************************************** */
template <class DataType>
int cudaCommon_transferArrayFromDeviceToCpu(DataType *array_cpu, DataType *array_d, const unsigned nElements) {
    const unsigned memSize = nElements * sizeof(DataType);
    NR_CUDA_SAFE_CALL(hipMemcpy(array_cpu, array_d, memSize, hipMemcpyDeviceToHost));
    return EXIT_SUCCESS;
}
template int cudaCommon_transferArrayFromDeviceToCpu<int>(int*, int*, const unsigned);
template int cudaCommon_transferArrayFromDeviceToCpu<float>(float*, float*, const unsigned);
template int cudaCommon_transferArrayFromDeviceToCpu<double>(double*, double*, const unsigned);
/* *************************************************************** */
void cudaCommon_destroyTextureObject(hipTextureObject_t *texObj) {
    NR_CUDA_SAFE_CALL(hipDestroyTextureObject(*texObj));
    delete texObj;
}
/* *************************************************************** */
UniqueTextureObjectPtr cudaCommon_createTextureObject(const void *devPtr,
                                                      const hipResourceType& resType,
                                                      const size_t& size,
                                                      const hipChannelFormatKind& channelFormat,
                                                      const unsigned& channelCount,
                                                      const hipTextureFilterMode& filterMode,
                                                      const bool& normalizedCoordinates) {
    // Specify texture
    hipResourceDesc resDesc{};
    resDesc.resType = resType;
    switch (resType) {
    case hipResourceTypeLinear:
        resDesc.res.linear.devPtr = const_cast<void*>(devPtr);
        resDesc.res.linear.desc.f = channelFormat;
        resDesc.res.linear.desc.x = 32;
        if (channelCount > 1)
            resDesc.res.linear.desc.y = 32;
        if (channelCount > 2)
            resDesc.res.linear.desc.z = 32;
        if (channelCount > 3)
            resDesc.res.linear.desc.w = 32;
        resDesc.res.linear.sizeInBytes = size;
        break;
    case hipResourceTypeArray:
        resDesc.res.array.array = static_cast<hipArray*>(const_cast<void*>(devPtr));
        break;
    default:
        reg_print_fct_error("cudaCommon_createTextureObject");
        reg_print_msg_error("Unsupported resource type");
        reg_exit();
    }

    // Specify texture object parameters
    hipTextureDesc texDesc{};
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.addressMode[2] = hipAddressModeWrap;
    texDesc.filterMode = filterMode;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = normalizedCoordinates;

    // Create texture object
    UniqueTextureObjectPtr texObj(new hipTextureObject_t(), cudaCommon_destroyTextureObject);
    NR_CUDA_SAFE_CALL(hipCreateTextureObject(texObj.get(), &resDesc, &texDesc, nullptr));

    return texObj;
}
/* *************************************************************** */
