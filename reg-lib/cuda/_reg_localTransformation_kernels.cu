#include "hip/hip_runtime.h"
/*
 *  _reg_spline_kernels.cu
 *
 *
 *  Created by Marc Modat on 24/03/2009.
 *  Copyright (c) 2009-2018, University College London
 *  Copyright (c) 2018, NiftyReg Developers.
 *  All rights reserved.
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _reg_spline_KERNELS_CU
#define _reg_spline_KERNELS_CU

#include "_reg_common_gpu.h"

__device__ __constant__ int c_UseBSpline;
__device__ __constant__ int c_VoxelNumber;
__device__ __constant__ int c_ControlPointNumber;
__device__ __constant__ int3 c_ReferenceImageDim;
__device__ __constant__ int3 c_ControlPointImageDim;
__device__ __constant__ float3 c_ControlPointVoxelSpacing;
__device__ __constant__ float3 c_ControlPointSpacing;
__device__ __constant__ float3 c_ReferenceSpacing;
__device__ __constant__ float c_Weight;
__device__ __constant__ float3 c_Weight3;
__device__ __constant__ int c_ActiveVoxelNumber;
__device__ __constant__ bool c_Type;
__device__ __constant__ float3 c_AffineMatrix0;
__device__ __constant__ float3 c_AffineMatrix1;
__device__ __constant__ float3 c_AffineMatrix2;
__device__ __constant__ float4 c_AffineMatrix0b;
__device__ __constant__ float4 c_AffineMatrix1b;
__device__ __constant__ float4 c_AffineMatrix2b;
__device__ __constant__ float4 c_AffineMatrix0c;
__device__ __constant__ float4 c_AffineMatrix1c;
__device__ __constant__ float4 c_AffineMatrix2c;
/* *************************************************************** */
/* *************************************************************** */
texture<float4, 1, hipReadModeElementType> controlPointTexture;
texture<float4, 1, hipReadModeElementType> secondDerivativesTexture;
texture<float4,1, hipReadModeElementType> voxelDeformationTexture;
texture<int, 1, hipReadModeElementType> maskTexture;
texture<float,1, hipReadModeElementType> jacobianDeterminantTexture;
texture<float,1, hipReadModeElementType> jacobianMatricesTexture;
/* *************************************************************** */
/* *************************************************************** */
__device__ float2 operator*(float a, float2 b){
	return make_float2(a*b.x, a*b.y);
}
__device__ float3 operator*(float a, float3 b){
	return make_float3(a*b.x, a*b.y, a*b.z);
}
__device__ float3 operator*(float3 a, float3 b){
	return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}
__device__ float4 operator*(float4 a, float4 b){
	return make_float4(a.x*b.x, a.y*b.y, a.z*b.z, a.w*b.w);
}
__device__ float4 operator*(float a, float4 b){
	return make_float4(a*b.x, a*b.y, a*b.z, 0.0f);
}
/* *************************************************************** */
__device__ float2 operator/(float2 a, float2 b){
	return make_float2(a.x/b.x, a.y/b.y);
}
__device__ float3 operator/(float3 a, float b){
	return make_float3(a.x/b, a.y/b, a.z/b);
}
__device__ float3 operator/(float3 a, float3 b){
	return make_float3(a.x/b.x, a.y/b.y, a.z/b.z);
}
/* *************************************************************** */
__device__ float2 operator+(float2 a, float2 b){
	return make_float2(a.x+b.x, a.y+b.y);
}
__device__ float4 operator+(float4 a, float4 b){
	return make_float4(a.x+b.x, a.y+b.y, a.z+b.z, 0.0f);
}
__device__ float3 operator+(float3 a, float3 b){
	return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}
/* *************************************************************** */
__device__ float3 operator-(float3 a, float3 b){
	return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}
__device__ float4 operator-(float4 a, float4 b){
	return make_float4(a.x-b.x, a.y-b.y, a.z-b.z, 0.f);
}
/* *************************************************************** */
/* *************************************************************** */
__device__ void GetBasisBSplineValues(float basis, float *values)
{
	float FF= basis*basis;
	float FFF= FF*basis;
	float MF=1.f-basis;
	values[0] = (MF)*(MF)*(MF)/(6.f);
	values[1] = (3.f*FFF - 6.f*FF + 4.f)/6.f;
	values[2] = (-3.f*FFF + 3.f*FF + 3.f*basis + 1.f)/6.f;
	values[3] = (FFF/6.f);
}
/* *************************************************************** */
__device__ void GetFirstBSplineValues(float basis, float *values, float *first)
{
	GetBasisBSplineValues(basis, values);
	first[3]= basis * basis / 2.f;
	first[0]= basis - 0.5f - first[3];
	first[2]= 1.f + first[0] - 2.f*first[3];
	first[1]= - first[0] - first[2] - first[3];
}
/* *************************************************************** */
/* *************************************************************** */
__device__ void GetBasisSplineValues(float basis, float *values)
{
	float FF= basis*basis;
	values[0] = (basis * ((2.f-basis)*basis - 1.f))/2.f;
	values[1] = (FF * (3.f*basis-5.f) + 2.f)/2.f;
	values[2] = (basis * ((4.f-3.f*basis)*basis + 1.f))/2.f;
	values[3] = (basis-1.f) * FF/2.f;
}
/* *************************************************************** */
__device__ void GetBasisSplineValuesX(float basis, float4 *values)
{
	float FF= basis*basis;
	values->x = (basis * ((2.f-basis)*basis - 1.f))/2.f;
	values->y = (FF * (3.f*basis-5.f) + 2.f)/2.f;
	values->z = (basis * ((4.f-3.f*basis)*basis + 1.f))/2.f;
	values->w = (basis-1.f) * FF/2.f;
}
/* *************************************************************** */
__device__ void getBSplineBasisValue(float basis, int index, float *value, float *first)
{
	switch(index){
		case 0:
			*value = (1.f-basis)*(1.f-basis)*(1.f-basis)/6.f;
			*first = (2.f*basis - basis*basis - 1.f)/2.f;
			break;
		case 1:
			*value = (3.f*basis*basis*basis - 6.f*basis*basis + 4.f)/6.f;
			*first = (3.f*basis*basis - 4.f*basis)/2.f;
			break;
		case 2:
			*value = (3.f*basis*basis - 3.f*basis*basis*basis + 3.f*basis + 1.f)/6.f;
			*first = (2.f*basis - 3.f*basis*basis + 1.f)/2.f;
			break;
		case 3:
			*value = basis*basis*basis/6.f;
			*first = basis*basis/2.f;
			break;
		 default:
			*value = 0.f;
			*first = 0.f;
			break;
	}
}
/* *************************************************************** */
__device__ void GetFirstDerivativeBasisValues2D(int index,
												float *xBasis,
												float *yBasis){
	switch(index){
	case 0: xBasis[0]=-0.0833333f;yBasis[0]=-0.0833333f;break;
	case 1: xBasis[1]=0.f;yBasis[1]=-0.333333f;break;
	case 2: xBasis[2]=0.0833333f;yBasis[2]=-0.0833333f;break;
	case 3: xBasis[3]=-0.333333f;yBasis[3]=0.f;break;
	case 4: xBasis[4]=0.f;yBasis[4]=0.f;break;
	case 5: xBasis[5]=0.333333f;yBasis[5]=0.f;break;
	case 6: xBasis[6]=-0.0833333f;yBasis[6]=0.0833333f;break;
	case 7: xBasis[7]=0.f;yBasis[7]=0.333333f;break;
	case 8: xBasis[8]=0.0833333f;yBasis[8]=0.0833333f;break;
	}
}
/* *************************************************************** */
__device__ void GetFirstDerivativeBasisValues3D(int index,
												float *xBasis,
												float *yBasis,
												float *zBasis){
	switch(index){
	case 0: xBasis[0]=-0.013889f;yBasis[0]=-0.013889f;zBasis[0]=-0.013889f;break;
	case 1: xBasis[1]=0.000000f;yBasis[1]=-0.055556f;zBasis[1]=-0.055556f;break;
	case 2: xBasis[2]=0.013889f;yBasis[2]=-0.013889f;zBasis[2]=-0.013889f;break;
	case 3: xBasis[3]=-0.055556f;yBasis[3]=0.000000f;zBasis[3]=-0.055556f;break;
	case 4: xBasis[4]=0.000000f;yBasis[4]=0.000000f;zBasis[4]=-0.222222f;break;
	case 5: xBasis[5]=0.055556f;yBasis[5]=0.000000f;zBasis[5]=-0.055556f;break;
	case 6: xBasis[6]=-0.013889f;yBasis[6]=0.013889f;zBasis[6]=-0.013889f;break;
	case 7: xBasis[7]=0.000000f;yBasis[7]=0.055556f;zBasis[7]=-0.055556f;break;
	case 8: xBasis[8]=0.013889f;yBasis[8]=0.013889f;zBasis[8]=-0.013889f;break;
	case 9: xBasis[9]=-0.055556f;yBasis[9]=-0.055556f;zBasis[9]=0.000000f;break;
	case 10: xBasis[10]=0.000000f;yBasis[10]=-0.222222f;zBasis[10]=0.000000f;break;
	case 11: xBasis[11]=0.055556f;yBasis[11]=-0.055556f;zBasis[11]=0.000000f;break;
	case 12: xBasis[12]=-0.222222f;yBasis[12]=0.000000f;zBasis[12]=0.000000f;break;
	case 13: xBasis[13]=0.000000f;yBasis[13]=0.000000f;zBasis[13]=0.000000f;break;
	case 14: xBasis[14]=0.222222f;yBasis[14]=0.000000f;zBasis[14]=0.000000f;break;
	case 15: xBasis[15]=-0.055556f;yBasis[15]=0.055556f;zBasis[15]=0.000000f;break;
	case 16: xBasis[16]=0.000000f;yBasis[16]=0.222222f;zBasis[16]=0.000000f;break;
	case 17: xBasis[17]=0.055556f;yBasis[17]=0.055556f;zBasis[17]=0.000000f;break;
	case 18: xBasis[18]=-0.013889f;yBasis[18]=-0.013889f;zBasis[18]=0.013889f;break;
	case 19: xBasis[19]=0.000000f;yBasis[19]=-0.055556f;zBasis[19]=0.055556f;break;
	case 20: xBasis[20]=0.013889f;yBasis[20]=-0.013889f;zBasis[20]=0.013889f;break;
	case 21: xBasis[21]=-0.055556f;yBasis[21]=0.000000f;zBasis[21]=0.055556f;break;
	case 22: xBasis[22]=0.000000f;yBasis[22]=0.000000f;zBasis[22]=0.222222f;break;
	case 23: xBasis[23]=0.055556f;yBasis[23]=0.000000f;zBasis[23]=0.055556f;break;
	case 24: xBasis[24]=-0.013889f;yBasis[24]=0.013889f;zBasis[24]=0.013889f;break;
	case 25: xBasis[25]=0.000000f;yBasis[25]=0.055556f;zBasis[25]=0.055556f;break;
	case 26: xBasis[26]=0.013889f;yBasis[26]=0.013889f;zBasis[26]=0.013889f;break;
	}
}
/* *************************************************************** */
__device__ void GetSecondDerivativeBasisValues2D(int index,
												 float *xxBasis,
												 float *yyBasis,
												 float *xyBasis){
	switch(index){
	case 0:
		xxBasis[0]=0.166667f;yyBasis[0]=0.166667f;xyBasis[0]=0.25f;
		break;
	case 1:
		xxBasis[1]=-0.333333f;yyBasis[1]=0.666667f;xyBasis[1]=-0.f;
		break;
	case 2:
		xxBasis[2]=0.166667f;yyBasis[2]=0.166667f;xyBasis[2]=-0.25f;
		break;
	case 3:
		xxBasis[3]=0.666667f;yyBasis[3]=-0.333333f;xyBasis[3]=-0.f;
		break;
	case 4:
		xxBasis[4]=-1.33333f;yyBasis[4]=-1.33333f;xyBasis[4]=0.f;
		break;
	case 5:
		xxBasis[5]=0.666667f;yyBasis[5]=-0.333333f;xyBasis[5]=0.f;
		break;
	case 6:
		xxBasis[6]=0.166667f;yyBasis[6]=0.166667f;xyBasis[6]=-0.25f;
		break;
	case 7:
		xxBasis[7]=-0.333333f;yyBasis[7]=0.666667f;xyBasis[7]=0.f;
		break;
	case 8:
		xxBasis[8]=0.166667f;yyBasis[8]=0.166667f;xyBasis[8]=0.25f;
		break;
	}
}
/* *************************************************************** */
__device__ void GetSecondDerivativeBasisValues3D(int index,
												 float *xxBasis,
												 float *yyBasis,
												 float *zzBasis,
												 float *xyBasis,
												 float *yzBasis,
												 float *xzBasis){
	switch(index){
		case 0:
			xxBasis[0]=0.027778f;yyBasis[0]=0.027778f;zzBasis[0]=0.027778f;
			xyBasis[0]=0.041667f;yzBasis[0]=0.041667f;xzBasis[0]=0.041667f;
			break;
		case 1:
			xxBasis[1]=-0.055556f;yyBasis[1]=0.111111f;zzBasis[1]=0.111111f;
			xyBasis[1]=-0.000000f;yzBasis[1]=0.166667f;xzBasis[1]=-0.000000f;
			break;
		case 2:
			xxBasis[2]=0.027778f;yyBasis[2]=0.027778f;zzBasis[2]=0.027778f;
			xyBasis[2]=-0.041667f;yzBasis[2]=0.041667f;xzBasis[2]=-0.041667f;
			break;
		case 3:
			xxBasis[3]=0.111111f;yyBasis[3]=-0.055556f;zzBasis[3]=0.111111f;
			xyBasis[3]=-0.000000f;yzBasis[3]=-0.000000f;xzBasis[3]=0.166667f;
			break;
		case 4:
			xxBasis[4]=-0.222222f;yyBasis[4]=-0.222222f;zzBasis[4]=0.444444f;
			xyBasis[4]=0.000000f;yzBasis[4]=-0.000000f;xzBasis[4]=-0.000000f;
			break;
		case 5:
			xxBasis[5]=0.111111f;yyBasis[5]=-0.055556f;zzBasis[5]=0.111111f;
			xyBasis[5]=0.000000f;yzBasis[5]=-0.000000f;xzBasis[5]=-0.166667f;
			break;
		case 6:
			xxBasis[6]=0.027778f;yyBasis[6]=0.027778f;zzBasis[6]=0.027778f;
			xyBasis[6]=-0.041667f;yzBasis[6]=-0.041667f;xzBasis[6]=0.041667f;
			break;
		case 7:
			xxBasis[7]=-0.055556f;yyBasis[7]=0.111111f;zzBasis[7]=0.111111f;
			xyBasis[7]=0.000000f;yzBasis[7]=-0.166667f;xzBasis[7]=-0.000000f;
			break;
		case 8:
			xxBasis[8]=0.027778f;yyBasis[8]=0.027778f;zzBasis[8]=0.027778f;
			xyBasis[8]=0.041667f;yzBasis[8]=-0.041667f;xzBasis[8]=-0.041667f;
			break;
		case 9:
			xxBasis[9]=0.111111f;yyBasis[9]=0.111111f;zzBasis[9]=-0.055556f;
			xyBasis[9]=0.166667f;yzBasis[9]=-0.000000f;xzBasis[9]=-0.000000f;
			break;
		case 10:
			xxBasis[10]=-0.222222f;yyBasis[10]=0.444444f;zzBasis[10]=-0.222222f;
			xyBasis[10]=-0.000000f;yzBasis[10]=-0.000000f;xzBasis[10]=0.000000f;
			break;
		case 11:
			xxBasis[11]=0.111111f;yyBasis[11]=0.111111f;zzBasis[11]=-0.055556f;
			xyBasis[11]=-0.166667f;yzBasis[11]=-0.000000f;xzBasis[11]=0.000000f;
			break;
		case 12:
			xxBasis[12]=0.444444f;yyBasis[12]=-0.222222f;zzBasis[12]=-0.222222f;
			xyBasis[12]=-0.000000f;yzBasis[12]=0.000000f;xzBasis[12]=-0.000000f;
			break;
		case 13:
			xxBasis[13]=-0.888889f;yyBasis[13]=-0.888889f;zzBasis[13]=-0.888889f;
			xyBasis[13]=0.000000f;yzBasis[13]=0.000000f;xzBasis[13]=0.000000f;
			break;
		case 14:
			xxBasis[14]=0.444444f;yyBasis[14]=-0.222222f;zzBasis[14]=-0.222222f;
			xyBasis[14]=0.000000f;yzBasis[14]=0.000000f;xzBasis[14]=0.000000f;
			break;
		case 15:
			xxBasis[15]=0.111111f;yyBasis[15]=0.111111f;zzBasis[15]=-0.055556f;
			xyBasis[15]=-0.166667f;yzBasis[15]=0.000000f;xzBasis[15]=-0.000000f;
			break;
		case 16:
			xxBasis[16]=-0.222222f;yyBasis[16]=0.444444f;zzBasis[16]=-0.222222f;
			xyBasis[16]=0.000000f;yzBasis[16]=0.000000f;xzBasis[16]=0.000000f;
			break;
		case 17:
			xxBasis[17]=0.111111f;yyBasis[17]=0.111111f;zzBasis[17]=-0.055556f;
			xyBasis[17]=0.166667f;yzBasis[17]=0.000000f;xzBasis[17]=0.000000f;
			break;
		case 18:
			xxBasis[18]=0.027778f;yyBasis[18]=0.027778f;zzBasis[18]=0.027778f;
			xyBasis[18]=0.041667f;yzBasis[18]=-0.041667f;xzBasis[18]=-0.041667f;
			break;
		case 19:
			xxBasis[19]=-0.055556f;yyBasis[19]=0.111111f;zzBasis[19]=0.111111f;
			xyBasis[19]=-0.000000f;yzBasis[19]=-0.166667f;xzBasis[19]=0.000000f;
			break;
		case 20:
			xxBasis[20]=0.027778f;yyBasis[20]=0.027778f;zzBasis[20]=0.027778f;
			xyBasis[20]=-0.041667f;yzBasis[20]=-0.041667f;xzBasis[20]=0.041667f;
			break;
		case 21:
			xxBasis[21]=0.111111f;yyBasis[21]=-0.055556f;zzBasis[21]=0.111111f;
			xyBasis[21]=-0.000000f;yzBasis[21]=0.000000f;xzBasis[21]=-0.166667f;
			break;
		case 22:
			xxBasis[22]=-0.222222f;yyBasis[22]=-0.222222f;zzBasis[22]=0.444444f;
			xyBasis[22]=0.000000f;yzBasis[22]=0.000000f;xzBasis[22]=0.000000f;
			break;
		case 23:
			xxBasis[23]=0.111111f;yyBasis[23]=-0.055556f;zzBasis[23]=0.111111f;
			xyBasis[23]=0.000000f;yzBasis[23]=0.000000f;xzBasis[23]=0.166667f;
			break;
		case 24:
			xxBasis[24]=0.027778f;yyBasis[24]=0.027778f;zzBasis[24]=0.027778f;
			xyBasis[24]=-0.041667f;yzBasis[24]=0.041667f;xzBasis[24]=-0.041667f;
			break;
		case 25:
			xxBasis[25]=-0.055556f;yyBasis[25]=0.111111f;zzBasis[25]=0.111111f;
			xyBasis[25]=0.000000f;yzBasis[25]=0.166667f;xzBasis[25]=0.000000f;
			break;
		case 26:
			xxBasis[26]=0.027778f;yyBasis[26]=0.027778f;zzBasis[26]=0.027778f;
			xyBasis[26]=0.041667f;yzBasis[26]=0.041667f;xzBasis[26]=0.041667f;
			break;
	}
}
/* *************************************************************** */
/* *************************************************************** */
__device__ float4 get_SlidedValues_gpu(int x, int y)
{
	int newX=x;
	int newY=y;
	if(x<0){
		newX=0;
	}
	else if(x>=c_ReferenceImageDim.x){
		newX=c_ReferenceImageDim.x-1;
	}
	if(y<0){
		newY=0;
	}
	else if(y>=c_ReferenceImageDim.y){
		newY=c_ReferenceImageDim.y-1;
	}

	x=x-newX;
	y=y-newY;
	float4 slidedValues = make_float4(
				x * c_AffineMatrix0c.x +
				y * c_AffineMatrix0c.y,
				x * c_AffineMatrix1c.x +
				y * c_AffineMatrix1c.y,
				0.f,
				0.f);
	slidedValues = slidedValues +
			tex1Dfetch(voxelDeformationTexture,
					   newY*c_ReferenceImageDim.x+newX);
	return slidedValues;
}
/* *************************************************************** */
/* *************************************************************** */
__device__ float4 get_SlidedValues_gpu(int x, int y, int z)
{
	int newX=x;
	int newY=y;
	int newZ=z;
	if(x<0){
		newX=0;
	}
	else if(x>=c_ReferenceImageDim.x){
		newX=c_ReferenceImageDim.x-1;
	}
	if(y<0){
		newY=0;
	}
	else if(y>=c_ReferenceImageDim.y){
		newY=c_ReferenceImageDim.y-1;
	}
	if(z<0){
		newZ=0;
	}
	else if(z>=c_ReferenceImageDim.z){
		newZ=c_ReferenceImageDim.z-1;
	}

	x=x-newX;
	y=y-newY;
	z=z-newZ;
	float4 slidedValues = make_float4(
				x * c_AffineMatrix0c.x +
				y * c_AffineMatrix0c.y +
				z * c_AffineMatrix0c.z,
				x * c_AffineMatrix1c.x +
				y * c_AffineMatrix1c.y +
				z * c_AffineMatrix1c.z,
				x * c_AffineMatrix2c.x +
				y * c_AffineMatrix2c.y +
				z * c_AffineMatrix2c.z,
				0.f);
	slidedValues = slidedValues +
			tex1Dfetch(voxelDeformationTexture,
					   (newZ*c_ReferenceImageDim.y+newY)*c_ReferenceImageDim.x+newX);
	return slidedValues;
}
/* *************************************************************** */
/* *************************************************************** */
__global__ void reg_spline_getDeformationField3D(float4 *positionField)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ActiveVoxelNumber){

		// Allocate the shared memory
		extern __shared__ float yBasis[];
		// Compute the shared memory offset which correspond to four times the number of thread per block
		float *zBasis=&yBasis[4*blockDim.x*blockDim.y*blockDim.z];

		int3 imageSize = c_ReferenceImageDim;

		unsigned int tempIndex=tex1Dfetch(maskTexture,tid);
		const int z = tempIndex/(imageSize.x*imageSize.y);
		tempIndex  -= z*imageSize.x*imageSize.y;
		const int y = tempIndex/imageSize.x;
		const int x = tempIndex - y*imageSize.x;

		// the "nearest previous" node is determined [0,0,0]
		int3 nodeAnte;
		float3 gridVoxelSpacing = c_ControlPointVoxelSpacing;
		nodeAnte.x = (int)floorf((float)x/gridVoxelSpacing.x);
		nodeAnte.y = (int)floorf((float)y/gridVoxelSpacing.y);
		nodeAnte.z = (int)floorf((float)z/gridVoxelSpacing.z);

		const int shareMemIndex = 4*threadIdx.x;

		// Z basis values
		float relative = fabsf((float)z/gridVoxelSpacing.z-(float)nodeAnte.z);
		relative=relative>0?relative:0.f;
		if(c_UseBSpline) GetBasisBSplineValues(relative, &zBasis[shareMemIndex]);
		else GetBasisSplineValues(relative, &zBasis[shareMemIndex]);
		// Y basis values
		relative = fabsf((float)y/gridVoxelSpacing.y-(float)nodeAnte.y);
		relative=relative>0?relative:0.f;
		if(c_UseBSpline) GetBasisBSplineValues(relative, &yBasis[shareMemIndex]);
		else GetBasisSplineValues(relative, &yBasis[shareMemIndex]);
		// X basis values
		float xBasis[4];
		relative = fabsf((float)x/gridVoxelSpacing.x-(float)nodeAnte.x);
		relative=relative>0?relative:0.f;
		if(c_UseBSpline) GetBasisBSplineValues(relative, xBasis);
		else GetBasisSplineValues(relative, xBasis);

		int3 controlPointImageDim = c_ControlPointImageDim;
		float4 displacement=make_float4(0.0f,0.0f,0.0f,0.0f);
		float basis;
		float3 tempDisplacement;

		for(int c=0; c<4; c++){
			tempDisplacement=make_float3(0.0f,0.0f,0.0f);
			int indexYZ= ( (nodeAnte.z + c) * controlPointImageDim.y + nodeAnte.y) * controlPointImageDim.x;
			for(int b=0; b<4; b++){

				int indexXYZ = indexYZ + nodeAnte.x;
				float4 nodeCoefficientA = tex1Dfetch(controlPointTexture,indexXYZ++);
				float4 nodeCoefficientB = tex1Dfetch(controlPointTexture,indexXYZ++);
				float4 nodeCoefficientC = tex1Dfetch(controlPointTexture,indexXYZ++);
				float4 nodeCoefficientD = tex1Dfetch(controlPointTexture,indexXYZ);

				basis=yBasis[shareMemIndex+b];
				tempDisplacement.x += (
							nodeCoefficientA.x * xBasis[0] +
							nodeCoefficientB.x * xBasis[1] +
							nodeCoefficientC.x * xBasis[2] +
							nodeCoefficientD.x * xBasis[3] ) * basis;

				tempDisplacement.y += (
							nodeCoefficientA.y * xBasis[0] +
							nodeCoefficientB.y * xBasis[1] +
							nodeCoefficientC.y * xBasis[2] +
							nodeCoefficientD.y * xBasis[3] ) * basis;

				tempDisplacement.z += (
							nodeCoefficientA.z * xBasis[0] +
							nodeCoefficientB.z * xBasis[1] +
							nodeCoefficientC.z * xBasis[2] +
							nodeCoefficientD.z * xBasis[3] ) * basis;

				indexYZ += controlPointImageDim.x;
			}

			basis = zBasis[shareMemIndex+c];
			displacement.x += tempDisplacement.x * basis;
			displacement.y += tempDisplacement.y * basis;
			displacement.z += tempDisplacement.z * basis;
		}
		positionField[tid] = displacement;
	}
	return;
}
/* *************************************************************** */
/* *************************************************************** */
__global__ void reg_spline_getDeformationField2D(float4 *positionField)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ActiveVoxelNumber){

		// Allocate the shared memory
		extern __shared__ float yBasis[];

		int3 imageSize = c_ReferenceImageDim;

		unsigned int tempIndex=tex1Dfetch(maskTexture,tid);
		const int y = tempIndex/imageSize.x;
		const int x = tempIndex - y*imageSize.x;

		// the "nearest previous" node is determined [0,0,0]
		int2 nodeAnte;
		float2 gridVoxelSpacing = make_float2(c_ControlPointVoxelSpacing.x,
											  c_ControlPointVoxelSpacing.y);
		nodeAnte.x = (int)floorf((float)x/gridVoxelSpacing.x);
		nodeAnte.y = (int)floorf((float)y/gridVoxelSpacing.y);

		const int shareMemIndex = 4*threadIdx.x;

		// Y basis values
		float relative = fabsf((float)y/gridVoxelSpacing.y-(float)nodeAnte.y);
		if(c_UseBSpline) GetBasisBSplineValues(relative, &yBasis[shareMemIndex]);
		else GetBasisSplineValues(relative, &yBasis[shareMemIndex]);
		// X basis values
		float xBasis[4];
		relative = fabsf((float)x/gridVoxelSpacing.x-(float)nodeAnte.x);
		if(c_UseBSpline) GetBasisBSplineValues(relative, xBasis);
		else GetBasisSplineValues(relative, xBasis);

		int2 controlPointImageDim = make_int2(c_ControlPointImageDim.x,
											  c_ControlPointImageDim.y);
		float4 displacement=make_float4(0.0f,0.0f,0.0f,0.0f);
		float basis;

		for(int b=0; b<4; b++){
			int index =  (nodeAnte.y + b) * controlPointImageDim.x + nodeAnte.x;

			float4 nodeCoefficientA = tex1Dfetch(controlPointTexture,index++);
			float4 nodeCoefficientB = tex1Dfetch(controlPointTexture,index++);
			float4 nodeCoefficientC = tex1Dfetch(controlPointTexture,index++);
			float4 nodeCoefficientD = tex1Dfetch(controlPointTexture,index);

			basis=yBasis[shareMemIndex+b];
			displacement.x += basis * (
						nodeCoefficientA.x * xBasis[0] +
						nodeCoefficientB.x * xBasis[1] +
						nodeCoefficientC.x * xBasis[2] +
						nodeCoefficientD.x * xBasis[3]);

			displacement.y += basis * (
						nodeCoefficientA.y * xBasis[0] +
						nodeCoefficientB.y * xBasis[1] +
						nodeCoefficientC.y * xBasis[2] +
						nodeCoefficientD.y * xBasis[3]);

		}
		positionField[tid] = displacement;
	}
	return;
}
/* *************************************************************** */
/* *************************************************************** */
__global__ void reg_spline_getApproxSecondDerivatives2D(float4 *secondDerivativeValues)
{
	__shared__ float xxbasis[9];
	__shared__ float yybasis[9];
	__shared__ float xybasis[9];

	if(threadIdx.x<9)
		GetSecondDerivativeBasisValues2D(threadIdx.x,
										 xxbasis,
										 yybasis,
										 xybasis);
	__syncthreads();

	const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		const int y =(int)(tid/gridSize.x);
		const int x = int(tid - y*gridSize.x);

		float4 XX = make_float4(0.0f,0.0f,0.0f,0.0f);
		float4 YY = make_float4(0.0f,0.0f,0.0f,0.0f);
		float4 XY = make_float4(0.0f,0.0f,0.0f,0.0f);

		int tempIndex;
		if(0<x && x<gridSize.x-1 &&
		   0<y && y<gridSize.y-1){

			tempIndex=0;
			for(int b=y-1; b<y+2; ++b){
				for(int a=x-1; a<x+2; ++a){
					int indexXY = b*gridSize.x+a;
					float4 controlPointValues = tex1Dfetch(controlPointTexture,indexXY);
					XX.x = XX.x + xxbasis[tempIndex] * controlPointValues.x;
					XX.y = XX.y + xxbasis[tempIndex] * controlPointValues.y;
					YY.x = YY.x + yybasis[tempIndex] * controlPointValues.x;
					YY.y = YY.y + yybasis[tempIndex] * controlPointValues.y;
					XY.x = XY.x + xybasis[tempIndex] * controlPointValues.x;
					XY.y = XY.y + xybasis[tempIndex] * controlPointValues.y;
					++tempIndex;
				}
			}
		}

		tempIndex=3*tid;
		secondDerivativeValues[tempIndex++]=XX;
		secondDerivativeValues[tempIndex++]=YY;
		secondDerivativeValues[tempIndex] = XY;
	}
	return;
}
/* *************************************************************** */
/* *************************************************************** */
__global__ void reg_spline_getApproxSecondDerivatives3D(float4 *secondDerivativeValues)
{
	__shared__ float xxbasis[27];
	__shared__ float yybasis[27];
	__shared__ float zzbasis[27];
	__shared__ float xybasis[27];
	__shared__ float yzbasis[27];
	__shared__ float xzbasis[27];

	if(threadIdx.x<27)
		GetSecondDerivativeBasisValues3D(threadIdx.x,
										 xxbasis,
										 yybasis,
										 zzbasis,
										 xybasis,
										 yzbasis,
										 xzbasis);
	__syncthreads();

	const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		int tempIndex=tid;
		const int z =(int)(tempIndex/(gridSize.x*gridSize.y));
		tempIndex -= int(z*gridSize.x*gridSize.y);
		const int y =(int)(tempIndex/gridSize.x);
		const int x = int(tempIndex - y*gridSize.x);

		float4 XX = make_float4(0.0f,0.0f,0.0f,0.0f);
		float4 YY = make_float4(0.0f,0.0f,0.0f,0.0f);
		float4 ZZ = make_float4(0.0f,0.0f,0.0f,0.0f);
		float4 XY = make_float4(0.0f,0.0f,0.0f,0.0f);
		float4 YZ = make_float4(0.0f,0.0f,0.0f,0.0f);
		float4 XZ = make_float4(0.0f,0.0f,0.0f,0.0f);

		if(0<x && x<gridSize.x-1 &&
		   0<y && y<gridSize.y-1 &&
		   0<z && z<gridSize.z-1){

			tempIndex=0;
			for(int c=z-1; c<z+2; ++c){
				for(int b=y-1; b<y+2; ++b){
					for(int a=x-1; a<x+2; ++a){
						int indexXYZ = (c*gridSize.y+b)*gridSize.x+a;
						float4 controlPointValues = tex1Dfetch(controlPointTexture,indexXYZ);
						XX = XX + xxbasis[tempIndex] * controlPointValues;
						YY = YY + yybasis[tempIndex] * controlPointValues;
						ZZ = ZZ + zzbasis[tempIndex] * controlPointValues;
						XY = XY + xybasis[tempIndex] * controlPointValues;
						YZ = YZ + yzbasis[tempIndex] * controlPointValues;
						XZ = XZ + xzbasis[tempIndex] * controlPointValues;
						tempIndex++;
					}
				}
			}
		}

		tempIndex=6*tid;
		secondDerivativeValues[tempIndex++]=XX;
		secondDerivativeValues[tempIndex++]=YY;
		secondDerivativeValues[tempIndex++]=ZZ;
		secondDerivativeValues[tempIndex++]=XY;
		secondDerivativeValues[tempIndex++]=YZ;
		secondDerivativeValues[tempIndex] = XZ;
	}
}
/* *************************************************************** */
__global__ void reg_spline_getApproxBendingEnergy2D_kernel(float *penaltyTerm)
{
	const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){
		int index=tid*3;
		float4 XX = tex1Dfetch(secondDerivativesTexture,index++);XX=XX*XX;
		float4 YY = tex1Dfetch(secondDerivativesTexture,index++);YY=YY*YY;
		float4 XY = tex1Dfetch(secondDerivativesTexture,index++);XY=XY*XY;

		penaltyTerm[tid]=
				XX.x + XX.y +
				YY.x + YY.y +
				2.f*(XY.x + XY.y);
	}
	return;
}
/* *************************************************************** */
__global__ void reg_spline_getApproxBendingEnergy3D_kernel(float *penaltyTerm)
{
	const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){
		int index=tid*6;
		float4 XX = tex1Dfetch(secondDerivativesTexture,index++);XX=XX*XX;
		float4 YY = tex1Dfetch(secondDerivativesTexture,index++);YY=YY*YY;
		float4 ZZ = tex1Dfetch(secondDerivativesTexture,index++);ZZ=ZZ*ZZ;
		float4 XY = tex1Dfetch(secondDerivativesTexture,index++);XY=XY*XY;
		float4 YZ = tex1Dfetch(secondDerivativesTexture,index++);YZ=YZ*YZ;
		float4 XZ = tex1Dfetch(secondDerivativesTexture,index);XZ=XZ*XZ;

		penaltyTerm[tid]=
				XX.x + XX.y + XX.z +
				YY.x + YY.y + YY.z +
				ZZ.x + ZZ.y + ZZ.z +
				2.f*(XY.x + XY.y + XY.z +
					 YZ.x + YZ.y + YZ.z +
					 XZ.x + XZ.y + XZ.z);
	}
	return;
}
/* *************************************************************** */
__global__ void reg_spline_getApproxBendingEnergyGradient2D_kernel(float4 *nodeGradientArray)
{
	__shared__ float xxbasis[9];
	__shared__ float yybasis[9];
	__shared__ float xybasis[9];

	if(threadIdx.x<9)
		GetSecondDerivativeBasisValues2D(threadIdx.x,
										 xxbasis,
										 yybasis,
										 xybasis);
	__syncthreads();

	const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		const int y = tid/gridSize.x;
		const int x = tid - y*gridSize.x;

		float2 gradientValue=make_float2(0.0f,0.0f);
		float4 secondDerivativeValues;

		int coord=0;
		for(int b=y-1; b<y+2; ++b){
			for(int a=x-1; a<x+2; ++a){
				if(-1<a && -1<b && a<gridSize.x && b<gridSize.y){
					int indexXY = 3*(b*gridSize.x+a);
					secondDerivativeValues = tex1Dfetch(secondDerivativesTexture,indexXY++); // XX
					gradientValue.x += secondDerivativeValues.x * xxbasis[coord];
					gradientValue.y += secondDerivativeValues.y * xxbasis[coord];
					secondDerivativeValues = tex1Dfetch(secondDerivativesTexture,indexXY++); // YY
					gradientValue.x += secondDerivativeValues.x * yybasis[coord];
					gradientValue.y += secondDerivativeValues.y * yybasis[coord];
					secondDerivativeValues = 2.f*tex1Dfetch(secondDerivativesTexture,indexXY); // XY
					gradientValue.x += secondDerivativeValues.x * xybasis[coord];
					gradientValue.y += secondDerivativeValues.y * xybasis[coord];
				}
				coord++;
			}
		}

		nodeGradientArray[tid].x += c_Weight*gradientValue.x;
		nodeGradientArray[tid].y += c_Weight*gradientValue.y;
	}
}
/* *************************************************************** */
__global__ void reg_spline_getApproxBendingEnergyGradient3D_kernel(float4 *nodeGradientArray)
{
	__shared__ float xxbasis[27];
	__shared__ float yybasis[27];
	__shared__ float zzbasis[27];
	__shared__ float xybasis[27];
	__shared__ float yzbasis[27];
	__shared__ float xzbasis[27];

	if(threadIdx.x<27)
		GetSecondDerivativeBasisValues3D(threadIdx.x,
										 xxbasis,
										 yybasis,
										 zzbasis,
										 xybasis,
										 yzbasis,
										 xzbasis);
	__syncthreads();

	const int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		int tempIndex=tid;
		const int z = tempIndex/(gridSize.x*gridSize.y);
		tempIndex  -= z*gridSize.x*gridSize.y;
		const int y = tempIndex/gridSize.x;
		const int x = tempIndex - y*gridSize.x;

		float3 gradientValue=make_float3(0.0f,0.0f,0.0f);
		float4 secondDerivativeValues;

		int coord=0;
		for(int c=z-1; c<z+2; ++c){
			for(int b=y-1; b<y+2; ++b){
				for(int a=x-1; a<x+2; ++a){
					if(-1<a && -1<b && -1<c && a<gridSize.x && b<gridSize.y && c<gridSize.z){
						unsigned int indexXYZ = 6*((c*gridSize.y+b)*gridSize.x+a);
						secondDerivativeValues = tex1Dfetch(secondDerivativesTexture,indexXYZ++); // XX
						gradientValue.x += secondDerivativeValues.x * xxbasis[coord];
						gradientValue.y += secondDerivativeValues.y * xxbasis[coord];
						gradientValue.z += secondDerivativeValues.z * xxbasis[coord];
						secondDerivativeValues = tex1Dfetch(secondDerivativesTexture,indexXYZ++); // YY
						gradientValue.x += secondDerivativeValues.x * yybasis[coord];
						gradientValue.y += secondDerivativeValues.y * yybasis[coord];
						gradientValue.z += secondDerivativeValues.z * yybasis[coord];
						secondDerivativeValues = tex1Dfetch(secondDerivativesTexture,indexXYZ++); //ZZ
						gradientValue.x += secondDerivativeValues.x * zzbasis[coord];
						gradientValue.y += secondDerivativeValues.y * zzbasis[coord];
						gradientValue.z += secondDerivativeValues.z * zzbasis[coord];
						secondDerivativeValues = 2.f*tex1Dfetch(secondDerivativesTexture,indexXYZ++); // XY
						gradientValue.x += secondDerivativeValues.x * xybasis[coord];
						gradientValue.y += secondDerivativeValues.y * xybasis[coord];
						gradientValue.z += secondDerivativeValues.z * xybasis[coord];
						secondDerivativeValues = 2.f*tex1Dfetch(secondDerivativesTexture,indexXYZ++); // YZ
						gradientValue.x += secondDerivativeValues.x * yzbasis[coord];
						gradientValue.y += secondDerivativeValues.y * yzbasis[coord];
						gradientValue.z += secondDerivativeValues.z * yzbasis[coord];
						secondDerivativeValues = 2.f*tex1Dfetch(secondDerivativesTexture,indexXYZ); //XZ
						gradientValue.x += secondDerivativeValues.x * xzbasis[coord];
						gradientValue.y += secondDerivativeValues.y * xzbasis[coord];
						gradientValue.z += secondDerivativeValues.z * xzbasis[coord];
					}
					coord++;
				}
			}
		}
		gradientValue = c_Weight * gradientValue;

		float4 metricGradientValue;
		metricGradientValue = nodeGradientArray[tid];
		metricGradientValue.x += gradientValue.x;
		metricGradientValue.y += gradientValue.y;
		metricGradientValue.z += gradientValue.z;
		nodeGradientArray[tid]=metricGradientValue;
	}
}
/* *************************************************************** */
/* *************************************************************** */
__global__ void reg_spline_getApproxJacobianValues2D_kernel(float *jacobianMatrices,
															float *jacobianDet)
{
	__shared__ float xbasis[9];
	__shared__ float ybasis[9];

	if(threadIdx.x<9)
		GetFirstDerivativeBasisValues2D(threadIdx.x,
										xbasis,
										ybasis);
	__syncthreads();

	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		int tempIndex=tid;
		const int y =tempIndex/gridSize.x;
		const int x = tempIndex - y*gridSize.x;

		if(0<x && x<gridSize.x-1 &&
		   0<y && y<gridSize.y-1){

			float Tx_x=0, Tx_y=0;
			float Ty_x=0, Ty_y=0;

			tempIndex=0;
			for(int b=y-1; b<y+2; ++b){
				for(int a=x-1; a<x+2; ++a){
					int indexXY = b * gridSize.x + a;
					float4 controlPointValues = tex1Dfetch(controlPointTexture,indexXY);
					Tx_x += xbasis[tempIndex]*controlPointValues.x;
					Tx_y += ybasis[tempIndex]*controlPointValues.x;
					Ty_x += xbasis[tempIndex]*controlPointValues.y;
					Ty_y += ybasis[tempIndex]*controlPointValues.y;
					tempIndex++;
				}
			}

			// The jacobian matrix is reoriented
			float Tx_x2=c_AffineMatrix0.x*Tx_x + c_AffineMatrix0.y*Ty_x;
			float Tx_y2=c_AffineMatrix0.x*Tx_y + c_AffineMatrix0.y*Ty_y;
			float Ty_x2=c_AffineMatrix1.x*Tx_x + c_AffineMatrix1.y*Ty_x;
			float Ty_y2=c_AffineMatrix1.x*Tx_y + c_AffineMatrix1.y*Ty_y;

			// The Jacobian matrix is stored
			tempIndex=tid*4;
			jacobianMatrices[tempIndex++]=Tx_x2;
			jacobianMatrices[tempIndex++]=Tx_y2;
			jacobianMatrices[tempIndex++]=Ty_x2;
			jacobianMatrices[tempIndex] = Ty_y2;

			// The Jacobian determinant is computed and stored
			jacobianDet[tid]= Tx_x2 * Ty_y2 - Tx_y2 * Ty_x2;
		}
		else{
			tempIndex=tid*4;
			jacobianMatrices[tempIndex++]=1.f;
			jacobianMatrices[tempIndex++]=0.f;
			jacobianMatrices[tempIndex++]=0.f;
			jacobianMatrices[tempIndex]=1.f;
			jacobianDet[tid]= 1.0f;
		}
	}
	return;
}
/* *************************************************************** */
/* *************************************************************** */
__global__ void reg_spline_getApproxJacobianValues3D_kernel(float *jacobianMatrices,
														   float *jacobianDet)
{
	__shared__ float xbasis[27];
	__shared__ float ybasis[27];
	__shared__ float zbasis[27];

	if(threadIdx.x<27)
		GetFirstDerivativeBasisValues3D(threadIdx.x,
									  xbasis,
									  ybasis,
									  zbasis);
	__syncthreads();

	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		int tempIndex=tid;
		const int z =tempIndex/(gridSize.x*gridSize.y);
		tempIndex -= z*gridSize.x*gridSize.y;
		const int y =tempIndex/gridSize.x;
		const int x = tempIndex - y*gridSize.x;

		if(0<x && x<gridSize.x-1 &&
		   0<y && y<gridSize.y-1 &&
		   0<z && z<gridSize.z-1){

			float Tx_x=0, Tx_y=0, Tx_z=0;
			float Ty_x=0, Ty_y=0, Ty_z=0;
			float Tz_x=0, Tz_y=0, Tz_z=0;

			tempIndex=0;
			for(int c=z-1; c<z+2; ++c){
				for(int b=y-1; b<y+2; ++b){
					for(int a=x-1; a<x+2; ++a){
						int indexXYZ = (c*gridSize.y+b)*gridSize.x+a;
						float4 controlPointValues = tex1Dfetch(controlPointTexture,indexXYZ);
						Tx_x += xbasis[tempIndex]*controlPointValues.x;
						Tx_y += ybasis[tempIndex]*controlPointValues.x;
						Tx_z += zbasis[tempIndex]*controlPointValues.x;
						Ty_x += xbasis[tempIndex]*controlPointValues.y;
						Ty_y += ybasis[tempIndex]*controlPointValues.y;
						Ty_z += zbasis[tempIndex]*controlPointValues.y;
						Tz_x += xbasis[tempIndex]*controlPointValues.z;
						Tz_y += ybasis[tempIndex]*controlPointValues.z;
						Tz_z += zbasis[tempIndex]*controlPointValues.z;
						tempIndex++;
					}
				}
			}

			// The jacobian matrix is reoriented
			float Tx_x2=c_AffineMatrix0.x*Tx_x + c_AffineMatrix0.y*Ty_x + c_AffineMatrix0.z*Tz_x;
			float Tx_y2=c_AffineMatrix0.x*Tx_y + c_AffineMatrix0.y*Ty_y + c_AffineMatrix0.z*Tz_y;
			float Tx_z2=c_AffineMatrix0.x*Tx_z + c_AffineMatrix0.y*Ty_z + c_AffineMatrix0.z*Tz_z;
			float Ty_x2=c_AffineMatrix1.x*Tx_x + c_AffineMatrix1.y*Ty_x + c_AffineMatrix1.z*Tz_x;
			float Ty_y2=c_AffineMatrix1.x*Tx_y + c_AffineMatrix1.y*Ty_y + c_AffineMatrix1.z*Tz_y;
			float Ty_z2=c_AffineMatrix1.x*Tx_z + c_AffineMatrix1.y*Ty_z + c_AffineMatrix1.z*Tz_z;
			float Tz_x2=c_AffineMatrix2.x*Tx_x + c_AffineMatrix2.y*Ty_x + c_AffineMatrix2.z*Tz_x;
			float Tz_y2=c_AffineMatrix2.x*Tx_y + c_AffineMatrix2.y*Ty_y + c_AffineMatrix2.z*Tz_y;
			float Tz_z2=c_AffineMatrix2.x*Tx_z + c_AffineMatrix2.y*Ty_z + c_AffineMatrix2.z*Tz_z;

			// The Jacobian matrix is stored
			tempIndex=tid*9;
			jacobianMatrices[tempIndex++]=Tx_x2;
			jacobianMatrices[tempIndex++]=Tx_y2;
			jacobianMatrices[tempIndex++]=Tx_z2;
			jacobianMatrices[tempIndex++]=Ty_x2;
			jacobianMatrices[tempIndex++]=Ty_y2;
			jacobianMatrices[tempIndex++]=Ty_z2;
			jacobianMatrices[tempIndex++]=Tz_x2;
			jacobianMatrices[tempIndex++]=Tz_y2;
			jacobianMatrices[tempIndex] = Tz_z2;

			// The Jacobian determinant is computed and stored
			jacobianDet[tid]= Tx_x2*Ty_y2*Tz_z2
							+ Tx_y2*Ty_z2*Tz_x2
							+ Tx_z2*Ty_x2*Tz_y2
							- Tx_x2*Ty_z2*Tz_y2
							- Tx_y2*Ty_x2*Tz_z2
							- Tx_z2*Ty_y2*Tz_x2;
		}
		else{
			tempIndex=tid*9;
			jacobianMatrices[tempIndex++]=1.f;
			jacobianMatrices[tempIndex++]=0.f;
			jacobianMatrices[tempIndex++]=0.f;
			jacobianMatrices[tempIndex++]=0.f;
			jacobianMatrices[tempIndex++]=1.f;
			jacobianMatrices[tempIndex++]=0.f;
			jacobianMatrices[tempIndex++]=0.f;
			jacobianMatrices[tempIndex++]=0.f;
			jacobianMatrices[tempIndex]=1.f;
			jacobianDet[tid]= 1.0f;
		}
	}
	return;
}
/* *************************************************************** */
__global__ void reg_spline_getJacobianValues2D_kernel(float *jacobianMatrices,
													 float *jacobianDet)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_VoxelNumber){

		int2 imageSize = make_int2(c_ReferenceImageDim.x,c_ReferenceImageDim.y);

		unsigned int tempIndex=tid;
		const int y = tempIndex/imageSize.x;
		const int x = tempIndex - y*imageSize.x;

		// the "nearest previous" node is determined [0,0,0]
		int2 nodeAnte;
		float2 gridVoxelSpacing = make_float2(c_ControlPointVoxelSpacing.x,c_ControlPointVoxelSpacing.y);
		nodeAnte.x = (int)floorf((float)x/gridVoxelSpacing.x);
		nodeAnte.y = (int)floorf((float)y/gridVoxelSpacing.y);

		float xBasis[4], yBasis[4], xFirst[4], yFirst[4], relative;

		relative = fabsf((float)x/gridVoxelSpacing.x-(float)nodeAnte.x);
		GetFirstBSplineValues(relative, xBasis, xFirst);

		relative = fabsf((float)y/gridVoxelSpacing.y-(float)nodeAnte.y);
		GetFirstBSplineValues(relative, yBasis, yFirst);

		int2 controlPointImageDim = make_int2(c_ControlPointImageDim.x,c_ControlPointImageDim.y);
		float2 Tx=make_float2(0.f,0.f);
		float2 Ty=make_float2(0.f,0.f);

		for(int b=0; b<4; ++b){
			int indexXY= ( nodeAnte.y + b) * controlPointImageDim.x + nodeAnte.x;

			float4 nodeCoefficient = tex1Dfetch(controlPointTexture,indexXY++);
			float2 tempBasis = make_float2(xFirst[0]*yBasis[b], xBasis[0]*yFirst[b]);
			Tx = Tx + nodeCoefficient.x * tempBasis;
			Ty = Ty + nodeCoefficient.y * tempBasis;

			nodeCoefficient = tex1Dfetch(controlPointTexture,indexXY++);
			tempBasis = make_float2(xFirst[1]*yBasis[b], xBasis[1]*yFirst[b]);
			Tx = Tx + nodeCoefficient.x * tempBasis;
			Ty = Ty + nodeCoefficient.y * tempBasis;

			nodeCoefficient = tex1Dfetch(controlPointTexture,indexXY++);
			tempBasis = make_float2(xFirst[2]*yBasis[b], xBasis[2]*yFirst[b]);
			Tx = Tx + nodeCoefficient.x * tempBasis;
			Ty = Ty + nodeCoefficient.y * tempBasis;

			nodeCoefficient = tex1Dfetch(controlPointTexture,indexXY);
			tempBasis = make_float2(xFirst[3]*yBasis[b], xBasis[3]*yFirst[b]);
			Tx = Tx + nodeCoefficient.x * tempBasis;
			Ty = Ty + nodeCoefficient.y * tempBasis;
		}

		// The jacobian matrix is reoriented
		float Tx_x2=c_AffineMatrix0.x*Tx.x + c_AffineMatrix0.y*Ty.x;
		float Tx_y2=c_AffineMatrix0.x*Tx.y + c_AffineMatrix0.y*Ty.y;
		float Ty_x2=c_AffineMatrix1.x*Tx.x + c_AffineMatrix1.y*Ty.x;
		float Ty_y2=c_AffineMatrix1.x*Tx.y + c_AffineMatrix1.y*Ty.y;

		// The Jacobian matrix is stored
		tempIndex=tid*4;
		jacobianMatrices[tempIndex++]=Tx_x2;
		jacobianMatrices[tempIndex++]=Tx_y2;
		jacobianMatrices[tempIndex++]=Ty_x2;
		jacobianMatrices[tempIndex] = Ty_y2;

		// The Jacobian determinant is computed and stored
		jacobianDet[tid]= Tx_x2 * Ty_y2 - Tx_y2 * Ty_x2;
	}
}
/* *************************************************************** */
__global__ void reg_spline_getJacobianValues3D_kernel(float *jacobianMatrices,
													 float *jacobianDet)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_VoxelNumber){

		int3 imageSize = c_ReferenceImageDim;

		unsigned int tempIndex=tid;
		const int z = tempIndex/(imageSize.x*imageSize.y);
		tempIndex  -= z*imageSize.x*imageSize.y;
		const int y = tempIndex/imageSize.x;
		const int x = tempIndex - y*imageSize.x;

		// the "nearest previous" node is determined [0,0,0]
		int3 nodeAnte;
		float3 gridVoxelSpacing = c_ControlPointVoxelSpacing;
		nodeAnte.x = (int)floorf((float)x/gridVoxelSpacing.x);
		nodeAnte.y = (int)floorf((float)y/gridVoxelSpacing.y);
		nodeAnte.z = (int)floorf((float)z/gridVoxelSpacing.z);

		extern __shared__ float yFirst[];
		float *zFirst=&yFirst[4*blockDim.x*blockDim.y*blockDim.z];

		float xBasis[4], yBasis[4], zBasis[4], xFirst[4], relative;

		const int shareMemIndex = 4*threadIdx.x;

		relative = fabsf((float)x/gridVoxelSpacing.x-(float)nodeAnte.x);
		GetFirstBSplineValues(relative, xBasis, xFirst);

		relative = fabsf((float)y/gridVoxelSpacing.y-(float)nodeAnte.y);
		GetFirstBSplineValues(relative, yBasis, &yFirst[shareMemIndex]);

		relative = fabsf((float)z/gridVoxelSpacing.z-(float)nodeAnte.z);
		GetFirstBSplineValues(relative, zBasis, &zFirst[shareMemIndex]);

		int3 controlPointImageDim = c_ControlPointImageDim;
		float3 Tx=make_float3(0.f,0.f,0.f);
		float3 Ty=make_float3(0.f,0.f,0.f);
		float3 Tz=make_float3(0.f,0.f,0.f);

		for(int c=0; c<4; ++c){
			for(int b=0; b<4; ++b){
				int indexXYZ= ( (nodeAnte.z + c) * controlPointImageDim.y + nodeAnte.y + b) * controlPointImageDim.x + nodeAnte.x;
				float3 tempBasisXY=make_float3(yBasis[b]*zBasis[c],
										yFirst[shareMemIndex+b]*zBasis[c],
										yBasis[b]*zFirst[shareMemIndex+c]);

				float4 nodeCoefficient = tex1Dfetch(controlPointTexture,indexXYZ++);
				float3 tempBasis = make_float3(xFirst[0],xBasis[0],xBasis[0])*tempBasisXY;
				Tx = Tx + nodeCoefficient.x * tempBasis;
				Ty = Ty + nodeCoefficient.y * tempBasis;
				Tz = Tz + nodeCoefficient.z * tempBasis;

				nodeCoefficient = tex1Dfetch(controlPointTexture,indexXYZ++);
				tempBasis = make_float3(xFirst[1],xBasis[1],xBasis[1])*tempBasisXY;
				Tx = Tx + nodeCoefficient.x * tempBasis;
				Ty = Ty + nodeCoefficient.y * tempBasis;
				Tz = Tz + nodeCoefficient.z * tempBasis;

				nodeCoefficient = tex1Dfetch(controlPointTexture,indexXYZ++);
				tempBasis = make_float3(xFirst[2],xBasis[2],xBasis[2])*tempBasisXY;
				Tx = Tx + nodeCoefficient.x * tempBasis;
				Ty = Ty + nodeCoefficient.y * tempBasis;
				Tz = Tz + nodeCoefficient.z * tempBasis;

				nodeCoefficient = tex1Dfetch(controlPointTexture,indexXYZ);
				tempBasis = make_float3(xFirst[3],xBasis[3],xBasis[3])*tempBasisXY;
				Tx = Tx + nodeCoefficient.x * tempBasis;
				Ty = Ty + nodeCoefficient.y * tempBasis;
				Tz = Tz + nodeCoefficient.z * tempBasis;
			}
		}

		// The jacobian matrix is reoriented
		float Tx_x2=c_AffineMatrix0.x*Tx.x + c_AffineMatrix0.y*Ty.x + c_AffineMatrix0.z*Tz.x;
		float Tx_y2=c_AffineMatrix0.x*Tx.y + c_AffineMatrix0.y*Ty.y + c_AffineMatrix0.z*Tz.y;
		float Tx_z2=c_AffineMatrix0.x*Tx.z + c_AffineMatrix0.y*Ty.z + c_AffineMatrix0.z*Tz.z;
		float Ty_x2=c_AffineMatrix1.x*Tx.x + c_AffineMatrix1.y*Ty.x + c_AffineMatrix1.z*Tz.x;
		float Ty_y2=c_AffineMatrix1.x*Tx.y + c_AffineMatrix1.y*Ty.y + c_AffineMatrix1.z*Tz.y;
		float Ty_z2=c_AffineMatrix1.x*Tx.z + c_AffineMatrix1.y*Ty.z + c_AffineMatrix1.z*Tz.z;
		float Tz_x2=c_AffineMatrix2.x*Tx.x + c_AffineMatrix2.y*Ty.x + c_AffineMatrix2.z*Tz.x;
		float Tz_y2=c_AffineMatrix2.x*Tx.y + c_AffineMatrix2.y*Ty.y + c_AffineMatrix2.z*Tz.y;
		float Tz_z2=c_AffineMatrix2.x*Tx.z + c_AffineMatrix2.y*Ty.z + c_AffineMatrix2.z*Tz.z;

		// The Jacobian matrix is stored
		tempIndex=tid*9;
		jacobianMatrices[tempIndex++]=Tx_x2;
		jacobianMatrices[tempIndex++]=Tx_y2;
		jacobianMatrices[tempIndex++]=Tx_z2;
		jacobianMatrices[tempIndex++]=Ty_x2;
		jacobianMatrices[tempIndex++]=Ty_y2;
		jacobianMatrices[tempIndex++]=Ty_z2;
		jacobianMatrices[tempIndex++]=Tz_x2;
		jacobianMatrices[tempIndex++]=Tz_y2;
		jacobianMatrices[tempIndex] = Tz_z2;

		// The Jacobian determinant is computed and stored
		jacobianDet[tid]= Tx_x2*Ty_y2*Tz_z2
						+ Tx_y2*Ty_z2*Tz_x2
						+ Tx_z2*Ty_x2*Tz_y2
						- Tx_x2*Ty_z2*Tz_y2
						- Tx_y2*Ty_x2*Tz_z2
						- Tx_z2*Ty_y2*Tz_x2;
	}
}
/* *************************************************************** */
__global__ void reg_spline_logSquaredValues_kernel(float *det)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_VoxelNumber){
		float val = logf(det[tid]);
		det[tid]=val*val;
	}
}
/* *************************************************************** */
__device__ void getJacobianGradientValues2D(float *jacobianMatrix,
											float detJac,
											float basisX,
											float basisY,
											float2 *jacobianConstraint)
{
	jacobianConstraint->x += detJac * (
			basisX * jacobianMatrix[3] -
			basisY * jacobianMatrix[2] );
	jacobianConstraint->y += detJac * (
			basisY * jacobianMatrix[0] -
			basisX * jacobianMatrix[1] );
}
/* *************************************************************** */
__device__ void getJacobianGradientValues3D(float *jacobianMatrix,
											float detJac,
											float basisX,
											float basisY,
											float basisZ,
											float3 *jacobianConstraint)
{
	jacobianConstraint->x += detJac * (
			basisX * (jacobianMatrix[4]*jacobianMatrix[8] - jacobianMatrix[5]*jacobianMatrix[7]) +
			basisY * (jacobianMatrix[5]*jacobianMatrix[6] - jacobianMatrix[3]*jacobianMatrix[8]) +
			basisZ * (jacobianMatrix[3]*jacobianMatrix[7] - jacobianMatrix[4]*jacobianMatrix[6]) );

	jacobianConstraint->y += detJac * (
			basisX * (jacobianMatrix[2]*jacobianMatrix[7] - jacobianMatrix[1]*jacobianMatrix[8]) +
			basisY * (jacobianMatrix[0]*jacobianMatrix[8] - jacobianMatrix[2]*jacobianMatrix[6]) +
			basisZ * (jacobianMatrix[1]*jacobianMatrix[6] - jacobianMatrix[0]*jacobianMatrix[7]) );

	jacobianConstraint->z += detJac * (
			basisX * (jacobianMatrix[1]*jacobianMatrix[5] - jacobianMatrix[2]*jacobianMatrix[4]) +
			basisY * (jacobianMatrix[2]*jacobianMatrix[3] - jacobianMatrix[0]*jacobianMatrix[5]) +
			basisZ * (jacobianMatrix[0]*jacobianMatrix[4] - jacobianMatrix[1]*jacobianMatrix[3]) );
}
/* *************************************************************** */
__global__ void reg_spline_computeApproxJacGradient2D_kernel(float4 *gradient)
{
	__shared__ float xbasis[9];
	__shared__ float ybasis[9];

	if(threadIdx.x<9)
		GetFirstDerivativeBasisValues2D(threadIdx.x,
										xbasis,
										ybasis);
	__syncthreads();

	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		unsigned int tempIndex=tid;
		const int y =(int)(tempIndex/(gridSize.x));
		const int x = tempIndex - y*(gridSize.x);

		float2 jacobianGradient=make_float2(0.f,0.f);
		tempIndex=8;
		for(int pixelY=(int)(y-1); pixelY<(int)(y+2); ++pixelY){
			if(pixelY>0 && pixelY<gridSize.y-1){

				int jacIndex = pixelY*gridSize.x+x-1;
				for(int pixelX=(int)(x-1); pixelX<(int)(x+2); ++pixelX){
					if(pixelX>0 && pixelX<gridSize.x-1){

						float detJac = tex1Dfetch(jacobianDeterminantTexture,jacIndex);

						if(detJac>0.f){
							detJac = 2.f*logf(detJac) / detJac;
							float jacobianMatrix[4];
							jacobianMatrix[0] = tex1Dfetch(jacobianMatricesTexture,jacIndex*4);
							jacobianMatrix[1] = tex1Dfetch(jacobianMatricesTexture,jacIndex*4+1);
							jacobianMatrix[2] = tex1Dfetch(jacobianMatricesTexture,jacIndex*4+2);
							jacobianMatrix[3] = tex1Dfetch(jacobianMatricesTexture,jacIndex*4+3);

							getJacobianGradientValues2D(jacobianMatrix,
														detJac,
														xbasis[tempIndex],
														ybasis[tempIndex],
														&jacobianGradient);
						}
					}
					jacIndex++;
					tempIndex--;
				}
			}
			else tempIndex-=3;
		}
		gradient[tid] = gradient[tid] + make_float4(c_Weight3.x
													* (c_AffineMatrix0.x * jacobianGradient.x
													   + c_AffineMatrix0.y * jacobianGradient.y),
													c_Weight3.y
													* (c_AffineMatrix1.x * jacobianGradient.x
													   + c_AffineMatrix1.y * jacobianGradient.y),
													0.f,
													0.f);

	}
}
/* *************************************************************** */
__global__ void reg_spline_computeApproxJacGradient3D_kernel(float4 *gradient)
{
	__shared__ float xbasis[27];
	__shared__ float ybasis[27];
	__shared__ float zbasis[27];

	if(threadIdx.x<27)
		GetFirstDerivativeBasisValues3D(threadIdx.x,
									  xbasis,
									  ybasis,
									  zbasis);
	__syncthreads();

	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		unsigned int tempIndex=tid;
		const int z =(int)(tempIndex/(gridSize.x*gridSize.y));
		tempIndex -= z*(gridSize.x)*(gridSize.y);
		const int y =(int)(tempIndex/(gridSize.x));
		const int x = tempIndex - y*(gridSize.x);

		float3 jacobianGradient=make_float3(0.f,0.f,0.f);
		tempIndex=26;
		for(int pixelZ=(int)(z-1); pixelZ<(int)(z+2); ++pixelZ){
			if(pixelZ>0 && pixelZ<gridSize.z-1){

				for(int pixelY=(int)(y-1); pixelY<(int)(y+2); ++pixelY){
					if(pixelY>0 && pixelY<gridSize.y-1){

						int jacIndex = (pixelZ*gridSize.y+pixelY)*gridSize.x+x-1;
						for(int pixelX=(int)(x-1); pixelX<(int)(x+2); ++pixelX){
							if(pixelX>0 && pixelX<gridSize.x-1){

								float detJac = tex1Dfetch(jacobianDeterminantTexture,jacIndex);

								if(detJac>0.f){
									detJac = 2.f*logf(detJac) / detJac;
									float jacobianMatrix[9];
									jacobianMatrix[0] = tex1Dfetch(jacobianMatricesTexture,jacIndex*9);
									jacobianMatrix[1] = tex1Dfetch(jacobianMatricesTexture,jacIndex*9+1);
									jacobianMatrix[2] = tex1Dfetch(jacobianMatricesTexture,jacIndex*9+2);
									jacobianMatrix[3] = tex1Dfetch(jacobianMatricesTexture,jacIndex*9+3);
									jacobianMatrix[4] = tex1Dfetch(jacobianMatricesTexture,jacIndex*9+4);
									jacobianMatrix[5] = tex1Dfetch(jacobianMatricesTexture,jacIndex*9+5);
									jacobianMatrix[6] = tex1Dfetch(jacobianMatricesTexture,jacIndex*9+6);
									jacobianMatrix[7] = tex1Dfetch(jacobianMatricesTexture,jacIndex*9+7);
									jacobianMatrix[8] = tex1Dfetch(jacobianMatricesTexture,jacIndex*9+8);

									getJacobianGradientValues3D(jacobianMatrix,
															  detJac,
															  xbasis[tempIndex],
															  ybasis[tempIndex],
															  zbasis[tempIndex],
															  &jacobianGradient);
								}
							}
							jacIndex++;
							tempIndex--;
						}
					}
					else tempIndex-=3;
				}
			}
			else tempIndex-=9;
		}
		gradient[tid] = gradient[tid] + make_float4(c_Weight3.x
													* (c_AffineMatrix0.x * jacobianGradient.x
													   + c_AffineMatrix0.y * jacobianGradient.y
													   + c_AffineMatrix0.z * jacobianGradient.z),
													c_Weight3.y
													* (c_AffineMatrix1.x * jacobianGradient.x
													   + c_AffineMatrix1.y * jacobianGradient.y
													   + c_AffineMatrix1.z * jacobianGradient.z),
													c_Weight3.z
													* (c_AffineMatrix2.x * jacobianGradient.x
													   + c_AffineMatrix2.y * jacobianGradient.y
													   + c_AffineMatrix2.z * jacobianGradient.z),
													0.f);

	}
}
/* *************************************************************** */
__global__ void reg_spline_computeJacGradient2D_kernel(float4 *gradient)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		int tempIndex=tid;
		const int y = tempIndex/gridSize.x;
		const int x = tempIndex - y*gridSize.x;

		float2 jacobianGradient=make_float2(0.f,0.f);

		float3 spacingVoxel = c_ControlPointVoxelSpacing;

		for(int pixelY=(int)ceilf((y-3)*spacingVoxel.y);
			pixelY<=(int)ceilf((y+1)*spacingVoxel.y);
			++pixelY){
			if(pixelY>-1 && pixelY<c_ReferenceImageDim.y){

				int yPre = (int)((float)pixelY/spacingVoxel.y);
				float basis = (float)pixelY/spacingVoxel.y - (float)yPre;
				float yBasis, yFirst;
				getBSplineBasisValue(basis,y-yPre,&yBasis,&yFirst);

				for(int pixelX=(int)ceilf((x-3)*spacingVoxel.x);
					pixelX<=(int)ceilf((x+1)*spacingVoxel.x);
					++pixelX){
					if(pixelX>-1 && pixelX<c_ReferenceImageDim.x && (yFirst!=0.f || yBasis!=0.f)){

						int xPre = (int)((float)pixelX/spacingVoxel.x);
						basis = (float)pixelX/spacingVoxel.x - (float)xPre;
						float xBasis, xFirst;
						getBSplineBasisValue(basis,x-xPre,&xBasis,&xFirst);

						int jacIndex = pixelY*c_ReferenceImageDim.x + pixelX;

						float detJac = tex1Dfetch(jacobianDeterminantTexture,jacIndex);

						if(detJac>0.f && (xFirst!=0.f || xBasis!=0.f)){
							detJac = 2.f*logf(detJac) / detJac;
							float jacobianMatrix[4];
							jacIndex *= 4;
							jacobianMatrix[0] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
							jacobianMatrix[1] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
							jacobianMatrix[2] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
							jacobianMatrix[3] = tex1Dfetch(jacobianMatricesTexture,jacIndex);

							float2 basisValues = make_float2(
										xFirst*yBasis,
										xBasis*yFirst);
							getJacobianGradientValues2D(jacobianMatrix,
														detJac,
														basisValues.x,
														basisValues.y,
														&jacobianGradient);
						}
					}
				}
			}
		}
		gradient[tid] = gradient[tid] + make_float4(
						c_Weight3.x
						* (c_AffineMatrix0.x * jacobianGradient.x
						   + c_AffineMatrix0.y * jacobianGradient.y),
						c_Weight3.y
						* (c_AffineMatrix1.x * jacobianGradient.x
						   + c_AffineMatrix1.y * jacobianGradient.y),
						0.f,
						0.f);
   }
}
/* *************************************************************** */
__global__ void reg_spline_computeJacGradient3D_kernel(float4 *gradient)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		int tempIndex=tid;
		const int z = tempIndex/(gridSize.x*gridSize.y);
		tempIndex  -= z*gridSize.x*gridSize.y;
		const int y = tempIndex/gridSize.x;
		const int x = tempIndex - y*gridSize.x;

		float3 jacobianGradient=make_float3(0.f,0.f,0.f);

		float3 spacingVoxel = c_ControlPointVoxelSpacing;

		for(int pixelZ=(int)ceilf((z-3)*spacingVoxel.z);
			pixelZ<=(int)ceilf((z+1)*spacingVoxel.z);
			++pixelZ){
			if(pixelZ>-1 && pixelZ<c_ReferenceImageDim.z){

				int zPre = (int)((float)pixelZ/spacingVoxel.z);
				float basis = (float)pixelZ/spacingVoxel.z - (float)zPre;
				float zBasis, zFirst;
				getBSplineBasisValue(basis,z-zPre,&zBasis,&zFirst);

				for(int pixelY=(int)ceilf((y-3)*spacingVoxel.y);
					pixelY<=(int)ceilf((y+1)*spacingVoxel.y);
					++pixelY){
					if(pixelY>-1 && pixelY<c_ReferenceImageDim.y && (zFirst!=0.f || zBasis!=0.f)){

						int yPre = (int)((float)pixelY/spacingVoxel.y);
						basis = (float)pixelY/spacingVoxel.y - (float)yPre;
						float yBasis, yFirst;
						getBSplineBasisValue(basis,y-yPre,&yBasis,&yFirst);

						for(int pixelX=(int)ceilf((x-3)*spacingVoxel.x);
							pixelX<=(int)ceilf((x+1)*spacingVoxel.x);
							++pixelX){
							if(pixelX>-1 && pixelX<c_ReferenceImageDim.x && (yFirst!=0.f || yBasis!=0.f)){

								int xPre = (int)((float)pixelX/spacingVoxel.x);
								basis = (float)pixelX/spacingVoxel.x - (float)xPre;
								float xBasis, xFirst;
								getBSplineBasisValue(basis,x-xPre,&xBasis,&xFirst);

								int jacIndex = (pixelZ*c_ReferenceImageDim.y+pixelY)*c_ReferenceImageDim.x + pixelX;

								float detJac = tex1Dfetch(jacobianDeterminantTexture,jacIndex);

								if(detJac>0.f && (xFirst!=0.f || xBasis!=0.f)){
									detJac = 2.f*logf(detJac) / detJac;
									float jacobianMatrix[9];
									jacIndex *= 9;
									jacobianMatrix[0] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[1] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[2] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[3] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[4] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[5] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[6] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[7] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[8] = tex1Dfetch(jacobianMatricesTexture,jacIndex);

									float3 basisValues = make_float3(
											xFirst*yBasis*zBasis,
											xBasis*yFirst*zBasis,
											xBasis*yBasis*zFirst);
									getJacobianGradientValues3D(jacobianMatrix,
															  detJac,
															  basisValues.x,
															  basisValues.y,
															  basisValues.z,
															  &jacobianGradient);
								}
							}
						}
					}
				}
			}
		}
		gradient[tid] = gradient[tid] + make_float4(
						c_Weight3.x
						* (c_AffineMatrix0.x * jacobianGradient.x
						   + c_AffineMatrix0.y * jacobianGradient.y
						   + c_AffineMatrix0.z * jacobianGradient.z),
						c_Weight3.y
						* (c_AffineMatrix1.x * jacobianGradient.x
						   + c_AffineMatrix1.y * jacobianGradient.y
						   + c_AffineMatrix1.z * jacobianGradient.z),
						c_Weight3.z
						* (c_AffineMatrix2.x * jacobianGradient.x
						   + c_AffineMatrix2.y * jacobianGradient.y
						   + c_AffineMatrix2.z * jacobianGradient.z),
						0.f);
   }
}
/* *************************************************************** */
__global__ void reg_spline_approxCorrectFolding3D_kernel(float4 *controlPointGrid_d)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		unsigned int tempIndex=tid;
		const int z = tempIndex/(gridSize.x*gridSize.y);
		tempIndex  -= z*gridSize.x*gridSize.y;
		const int y = tempIndex/gridSize.x;
		const int x = tempIndex - y*gridSize.x;

		float3 foldingCorrection=make_float3(0.f,0.f,0.f);
		for(int pixelZ=(int)(z-1); pixelZ<(int)(z+2); ++pixelZ){
			if(pixelZ>0 && pixelZ<gridSize.z-1){

				for(int pixelY=(int)(y-1); pixelY<(int)(y+2); ++pixelY){
					if(pixelY>0 && pixelY<gridSize.y-1){

						for(int pixelX=(int)(x-1); pixelX<(int)(x+2); ++pixelX){
							if(pixelX>0 && pixelX<gridSize.x-1){

								int jacIndex = (pixelZ*gridSize.y+pixelY)*gridSize.x+pixelX;
								float detJac = tex1Dfetch(jacobianDeterminantTexture,jacIndex);

								if(detJac<=0.f){

									float jacobianMatrix[9];
									jacIndex*=9;
									jacobianMatrix[0] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[1] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[2] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[3] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[4] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[5] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[6] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[7] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[8] = tex1Dfetch(jacobianMatricesTexture,jacIndex);

									float xBasis, xFirst, yBasis, yFirst, zBasis, zFirst;
									getBSplineBasisValue(0.f,x-pixelX+1,&xBasis,&xFirst);
									getBSplineBasisValue(0.f,y-pixelY+1,&yBasis,&yFirst);
									getBSplineBasisValue(0.f,z-pixelZ+1,&zBasis,&zFirst);

									float3 basisValue = make_float3(
											xFirst*yBasis*zBasis,
											xBasis*yFirst*zBasis,
											xBasis*yBasis*zFirst);

									getJacobianGradientValues3D(jacobianMatrix,
															  1.f,
															  basisValue.x,
															  basisValue.y,
															  basisValue.z,
															  &foldingCorrection);
								}
							}
						}
					}
				}
			}
		}
		if(foldingCorrection.x!=0.f && foldingCorrection.y!=0.f && foldingCorrection.z!=0.f){
			float3 gradient = make_float3(
				c_AffineMatrix0.x * foldingCorrection.x
				+ c_AffineMatrix0.y * foldingCorrection.y
				+ c_AffineMatrix0.z * foldingCorrection.z,
				c_AffineMatrix1.x * foldingCorrection.x
			   + c_AffineMatrix1.y * foldingCorrection.y
			   + c_AffineMatrix1.z * foldingCorrection.z,
			   c_AffineMatrix2.x * foldingCorrection.x
			   + c_AffineMatrix2.y * foldingCorrection.y
			   + c_AffineMatrix2.z * foldingCorrection.z);

			float norm = 5.f * sqrtf(gradient.x*gradient.x
									 + gradient.y*gradient.y
									 + gradient.z*gradient.z);
			controlPointGrid_d[tid] = controlPointGrid_d[tid] +
									  make_float4(gradient.x*c_ControlPointSpacing.x/norm,
												  gradient.y*c_ControlPointSpacing.y/norm,
												  gradient.z*c_ControlPointSpacing.z/norm,
												  0.f);
		}
	}
}
/* *************************************************************** */
__global__ void reg_spline_correctFolding3D_kernel(float4 *controlPointGrid_d)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_ControlPointNumber){

		int3 gridSize = c_ControlPointImageDim;

		unsigned int tempIndex=tid;
		const int z = tempIndex/(gridSize.x*gridSize.y);
		tempIndex  -= z*gridSize.x*gridSize.y;
		const int y = tempIndex/gridSize.x;
		const int x = tempIndex - y*gridSize.x;

		float3 spacingVoxel = c_ControlPointVoxelSpacing;
		float3 foldingCorrection=make_float3(0.f,0.f,0.f);

		for(int pixelZ=(int)ceilf((z-3)*spacingVoxel.z);
			pixelZ<(int)ceilf((z+1)*spacingVoxel.z);
			++pixelZ){
			if(pixelZ>-1 && pixelZ<c_ReferenceImageDim.z){

				for(int pixelY=(int)ceilf((y-3)*spacingVoxel.y);
					pixelY<(int)ceilf((y+1)*spacingVoxel.y);
					++pixelY){
					if(pixelY>-1 && pixelY<c_ReferenceImageDim.y){

						for(int pixelX=(int)ceilf((x-3)*spacingVoxel.x);
							pixelX<(int)ceilf((x+1)*spacingVoxel.x);
							++pixelX){
							if(pixelX>-1 && pixelX<c_ReferenceImageDim.x){

								int jacIndex = (pixelZ*c_ReferenceImageDim.y+pixelY)*c_ReferenceImageDim.x+pixelX;
								float detJac = tex1Dfetch(jacobianDeterminantTexture,jacIndex);

								if(detJac<=0.f){

									float jacobianMatrix[9];
									jacIndex*=9;
									jacobianMatrix[0] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[1] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[2] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[3] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[4] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[5] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[6] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[7] = tex1Dfetch(jacobianMatricesTexture,jacIndex++);
									jacobianMatrix[8] = tex1Dfetch(jacobianMatricesTexture,jacIndex);

									float xBasis, xFirst, yBasis, yFirst, zBasis, zFirst;
									int pre=(int)((float)pixelX/spacingVoxel.x);
									float basis=(float)pixelX/spacingVoxel.x-(float)pre;
									getBSplineBasisValue(basis,x-pre,&xBasis,&xFirst);
									pre=(int)((float)pixelY/spacingVoxel.y);
									basis=(float)pixelY/spacingVoxel.y-(float)pre;
									getBSplineBasisValue(basis,y-pre,&yBasis,&yFirst);
									pre=(int)((float)pixelZ/spacingVoxel.z);
									basis=(float)pixelZ/spacingVoxel.z-(float)pre;
									getBSplineBasisValue(basis,z-pre,&zBasis,&zFirst);

									float3 basisValue = make_float3(
											xFirst*yBasis*zBasis,
											xBasis*yFirst*zBasis,
											xBasis*yBasis*zFirst);

									getJacobianGradientValues3D(jacobianMatrix,
															  1.f,
															  basisValue.x,
															  basisValue.y,
															  basisValue.z,
															  &foldingCorrection);
								}
							}
						}
					}
				}
			}
		}
		if(foldingCorrection.x!=0.f && foldingCorrection.y!=0.f && foldingCorrection.z!=0.f){
			float3 gradient = make_float3(
				c_AffineMatrix0.x * foldingCorrection.x
				+ c_AffineMatrix0.y * foldingCorrection.y
				+ c_AffineMatrix0.z * foldingCorrection.z,
				c_AffineMatrix1.x * foldingCorrection.x
			   + c_AffineMatrix1.y * foldingCorrection.y
			   + c_AffineMatrix1.z * foldingCorrection.z,
			   c_AffineMatrix2.x * foldingCorrection.x
			   + c_AffineMatrix2.y * foldingCorrection.y
			   + c_AffineMatrix2.z * foldingCorrection.z);

			float norm = 5.f * sqrtf(gradient.x*gradient.x
									 + gradient.y*gradient.y
									 + gradient.z*gradient.z);
			controlPointGrid_d[tid] = controlPointGrid_d[tid] +
									  make_float4(gradient.x*c_ControlPointSpacing.x/norm,
												  gradient.y*c_ControlPointSpacing.y/norm,
												  gradient.z*c_ControlPointSpacing.z/norm,
												  0.f);
		}
	}
}
/* *************************************************************** */
__global__ void reg_getDeformationFromDisplacement3D_kernel(float4 *imageArray_d)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_VoxelNumber){

		int3 imageSize = c_ReferenceImageDim;

		unsigned int tempIndex=tid;
		const int z = tempIndex/(imageSize.x*imageSize.y);
		tempIndex  -= z*imageSize.x*imageSize.y;
		const int y = tempIndex/imageSize.x;
		const int x = tempIndex - y*imageSize.x;

		float4 initialPosition;
		initialPosition.x=x*c_AffineMatrix0b.x + y*c_AffineMatrix0b.y + z*c_AffineMatrix0b.z + c_AffineMatrix0b.w;
		initialPosition.y=x*c_AffineMatrix1b.x + y*c_AffineMatrix1b.y + z*c_AffineMatrix1b.z + c_AffineMatrix1b.w;
		initialPosition.z=x*c_AffineMatrix2b.x + y*c_AffineMatrix2b.y + z*c_AffineMatrix2b.z + c_AffineMatrix2b.w;
		initialPosition.w=0.f;

		imageArray_d[tid] = imageArray_d[tid] + initialPosition;
	}
}
/* *************************************************************** */
__global__ void reg_getDisplacementFromDeformation3D_kernel(float4 *imageArray_d)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_VoxelNumber){

		int3 imageSize = c_ReferenceImageDim;

		unsigned int tempIndex=tid;
		const int z = tempIndex/(imageSize.x*imageSize.y);
		tempIndex  -= z*imageSize.x*imageSize.y;
		const int y = tempIndex/imageSize.x;
		const int x = tempIndex - y*imageSize.x;

		float4 initialPosition;
		initialPosition.x=x*c_AffineMatrix0b.x + y*c_AffineMatrix0b.y + z*c_AffineMatrix0b.z + c_AffineMatrix0b.w;
		initialPosition.y=x*c_AffineMatrix1b.x + y*c_AffineMatrix1b.y + z*c_AffineMatrix1b.z + c_AffineMatrix1b.w;
		initialPosition.z=x*c_AffineMatrix2b.x + y*c_AffineMatrix2b.y + z*c_AffineMatrix2b.z + c_AffineMatrix2b.w;
		initialPosition.w=0.f;

		imageArray_d[tid] = imageArray_d[tid] - initialPosition;
	}
}
/* *************************************************************** */
__global__ void reg_defField_compose2D_kernel(float4 *outDef)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_VoxelNumber){

		// Extract the original voxel position
		float4 position=outDef[tid];

		// Conversion from real position to voxel coordinate
		float4 voxelPosition;
		voxelPosition.x=
				position.x*c_AffineMatrix0b.x +
				position.y*c_AffineMatrix0b.y +
				c_AffineMatrix0b.w;
		voxelPosition.y=
				position.x*c_AffineMatrix1b.x +
				position.y*c_AffineMatrix1b.y +
				c_AffineMatrix1b.w;
		voxelPosition.z=0.f;
		voxelPosition.w=0.f;

		// linear interpolation
		int2 ante=make_int2(floorf(voxelPosition.x),
							floorf(voxelPosition.y));

		float relX[2], relY[2];
		relX[1]=voxelPosition.x-(float)ante.x;relX[0]=1.f-relX[1];
		relY[1]=voxelPosition.y-(float)ante.y;relY[0]=1.f-relY[1];

		position=make_float4(0.f,0.f,0.f,0.f);

		for(int b=0;b<2;++b){
			for(int a=0;a<2;++a){
				unsigned int index=(ante.y+b)*c_ReferenceImageDim.x+ante.x+a;
				float4 deformation;
				if((ante.x+a)>-1 && (ante.y+b)>-1 &&
				   (ante.x+a)<c_ReferenceImageDim.x &&
				   (ante.y+b)<c_ReferenceImageDim.y){
					deformation=tex1Dfetch(voxelDeformationTexture,index);
				}
				else{
					deformation = get_SlidedValues_gpu((ante.x+a),
													   (ante.y+b));
				}
				float basis=relX[a]*relY[b];
				position=position+basis*deformation;
			}
		}
		outDef[tid]=position;
	}
}
/* *************************************************************** */
__global__ void reg_defField_compose3D_kernel(float4 *outDef)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_VoxelNumber){

		// Extract the original voxel position
		float4 position=outDef[tid];

		// Conversion from real position to voxel coordinate
		float4 voxelPosition;
		voxelPosition.x=
				position.x*c_AffineMatrix0b.x +
				position.y*c_AffineMatrix0b.y +
				position.z*c_AffineMatrix0b.z +
				c_AffineMatrix0b.w;
		voxelPosition.y=
				position.x*c_AffineMatrix1b.x +
				position.y*c_AffineMatrix1b.y +
				position.z*c_AffineMatrix1b.z +
				c_AffineMatrix1b.w;
		voxelPosition.z=
				position.x*c_AffineMatrix2b.x +
				position.y*c_AffineMatrix2b.y +
				position.z*c_AffineMatrix2b.z +
				c_AffineMatrix2b.w;
		voxelPosition.w=0.f;

		// linear interpolation
		int3 ante=make_int3(floorf(voxelPosition.x),
							floorf(voxelPosition.y),
							floorf(voxelPosition.z));

		float relX[2], relY[2], relZ[2];
		relX[1]=voxelPosition.x-(float)ante.x;relX[0]=1.f-relX[1];
		relY[1]=voxelPosition.y-(float)ante.y;relY[0]=1.f-relY[1];
		relZ[1]=voxelPosition.z-(float)ante.z;relZ[0]=1.f-relZ[1];

		position=make_float4(0.f,0.f,0.f,0.f);

		for(int c=0;c<2;++c){
			for(int b=0;b<2;++b){
				for(int a=0;a<2;++a){
					unsigned int index=((ante.z+c)*c_ReferenceImageDim.y+ante.y+b)*c_ReferenceImageDim.x+ante.x+a;
					float4 deformation;
					if((ante.x+a)>-1 && (ante.y+b)>-1 && (ante.z+c)>-1 &&
					   (ante.x+a)<c_ReferenceImageDim.x &&
					   (ante.y+b)<c_ReferenceImageDim.y &&
					   (ante.z+c)<c_ReferenceImageDim.z){
						deformation=tex1Dfetch(voxelDeformationTexture,index);
					}
					else{
						deformation = get_SlidedValues_gpu((ante.x+a),
														   (ante.y+b),
														   (ante.z+c));
					}
					float basis=relX[a]*relY[b]*relZ[c];
					position=position+basis*deformation;
				}
			}
		}
		outDef[tid]=position;
	}
}
/* *************************************************************** */
__global__ void reg_defField_getJacobianMatrix3D_kernel(float *jacobianMatrices)
{
	const unsigned int tid= (blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
	if(tid<c_VoxelNumber){

		int3 imageSize = c_ReferenceImageDim;

		unsigned int tempIndex=tid;
		const int z = tempIndex/(imageSize.x*imageSize.y);
		tempIndex  -= z*imageSize.x*imageSize.y;
		const int y = tempIndex/imageSize.x;
		const int x = tempIndex - y*imageSize.x;

		if(x==imageSize.x-1 ||
		   y==imageSize.y-1 ||
		   z==imageSize.z-1 ){
			int index=tid*9;
			jacobianMatrices[index++]=1.0;
			jacobianMatrices[index++]=0.0;
			jacobianMatrices[index++]=0.0;
			jacobianMatrices[index++]=0.0;
			jacobianMatrices[index++]=1.0;
			jacobianMatrices[index++]=0.0;
			jacobianMatrices[index++]=0.0;
			jacobianMatrices[index++]=0.0;
			jacobianMatrices[index]=1.0;
			return;
		}

		float matrix[9];
		int index=(z*imageSize.y+y)*imageSize.x+x;
		float4 deformation = tex1Dfetch(voxelDeformationTexture,index);
		matrix[0] = deformation.x * -1.f;
		matrix[1] = deformation.x * -1.f;
		matrix[2] = deformation.x * -1.f;
		matrix[3] = deformation.y * -1.f;
		matrix[4] = deformation.y * -1.f;
		matrix[5] = deformation.y * -1.f;
		matrix[6] = deformation.z * -1.f;
		matrix[7] = deformation.z * -1.f;
		matrix[8] = deformation.z * -1.f;
		deformation = tex1Dfetch(voxelDeformationTexture,index+1);
		matrix[0] += deformation.x * 1.f;
		matrix[3] += deformation.y * 1.f;
		matrix[6] += deformation.z * 1.f;
		index=(z*imageSize.y+y+1)*imageSize.x+x;
		deformation = tex1Dfetch(voxelDeformationTexture,index);
		matrix[1] += deformation.x * 1.f;
		matrix[4] += deformation.y * 1.f;
		matrix[7] += deformation.z * 1.f;
		index=((z+1)*imageSize.y+y)*imageSize.x+x;
		deformation = tex1Dfetch(voxelDeformationTexture,index);
		matrix[2] += deformation.x * 1.f;
		matrix[5] += deformation.y * 1.f;
		matrix[8] += deformation.z * 1.f;

		index=tid*9;
		jacobianMatrices[index++]=c_AffineMatrix0.x*matrix[0] + c_AffineMatrix0.y*matrix[3] + c_AffineMatrix0.z*matrix[6];
		jacobianMatrices[index++]=c_AffineMatrix0.x*matrix[1] + c_AffineMatrix0.y*matrix[4] + c_AffineMatrix0.z*matrix[7];
		jacobianMatrices[index++]=c_AffineMatrix0.x*matrix[2] + c_AffineMatrix0.y*matrix[5] + c_AffineMatrix0.z*matrix[8];
		jacobianMatrices[index++]=c_AffineMatrix1.x*matrix[0] + c_AffineMatrix1.y*matrix[3] + c_AffineMatrix1.z*matrix[6];
		jacobianMatrices[index++]=c_AffineMatrix1.x*matrix[1] + c_AffineMatrix1.y*matrix[4] + c_AffineMatrix1.z*matrix[7];
		jacobianMatrices[index++]=c_AffineMatrix1.x*matrix[2] + c_AffineMatrix1.y*matrix[5] + c_AffineMatrix1.z*matrix[8];
		jacobianMatrices[index++]=c_AffineMatrix2.x*matrix[0] + c_AffineMatrix2.y*matrix[3] + c_AffineMatrix2.z*matrix[6];
		jacobianMatrices[index++]=c_AffineMatrix2.x*matrix[1] + c_AffineMatrix2.y*matrix[4] + c_AffineMatrix2.z*matrix[7];
		jacobianMatrices[index] = c_AffineMatrix2.x*matrix[2] + c_AffineMatrix2.y*matrix[5] + c_AffineMatrix2.z*matrix[8];
	}
}
/* *************************************************************** */
/* *************************************************************** */
/* *************************************************************** */
#endif
